#include "hip/hip_runtime.h"
#include "algebra_cuda.h"


__global__ void lcg_set2box_cuda_device(const lcg_float *low, const lcg_float *hig, lcg_float *a, 
    int n, bool low_bound, bool hig_bound)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
		if (hig_bound && a[i] >= hig[i]) a[i] = hig[i];
		if (!hig_bound && a[i] > hig[i]) a[i] = hig[i];
		if (low_bound && a[i] <= low[i]) a[i] = low[i];
		if (!low_bound && a[i] < low[i]) a[i] = low[i];
	}
	return;
}

__global__ void smDcsr_get_diagonal_device(const int *A_ptr, const int *A_col, const lcg_float *A_val, const int A_len, lcg_float *A_diag)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < A_len)
	{
		const int num_non0_row = A_ptr[i + 1] - A_ptr[i];

		for (int j = 0; j < num_non0_row; j++)
		{
			if (A_col[j + A_ptr[i]] == i)
			{
				A_diag[i] = A_val[j + A_ptr[i]];
				break;
			}
		}
	}
	return;
}

__global__ void vecMvecD_element_wise_device(const lcg_float *a, const lcg_float *b, lcg_float *c, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
		c[i] = a[i] * b[i];
	}
	return;
}

__global__ void vecDvecD_element_wise_device(const lcg_float *a, const lcg_float *b, lcg_float *c, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
		c[i] = a[i] / b[i];
	}
	return;
}

void lcg_set2box_cuda(const lcg_float *low, const lcg_float *hig, lcg_float *a, 
    int n, bool low_bound, bool hig_bound)
{
	int blockSize = 1024;
	int numBlocks = (n+ blockSize - 1) / blockSize;
	lcg_set2box_cuda_device<<<numBlocks, blockSize>>>(low, hig, a, n, low_bound, hig_bound);
	return;
}

void smDcsr_get_diagonal(const int *A_ptr, const int *A_col, const lcg_float *A_val, const int A_len, lcg_float *A_diag, int bk_size)
{
	int blockSize = bk_size;
	int numBlocks = (A_len+ blockSize - 1) / blockSize;
	smDcsr_get_diagonal_device<<<numBlocks, blockSize>>>(A_ptr, A_col, A_val, A_len, A_diag);
	return;
}

void vecMvecD_element_wise(const lcg_float *a, const lcg_float *b, lcg_float *c, int n, int bk_size)
{
	int blockSize = bk_size;
	int numBlocks = (n + blockSize - 1) / blockSize;
	vecMvecD_element_wise_device<<<numBlocks, blockSize>>>(a, b, c, n);
	return;
}

void vecDvecD_element_wise(const lcg_float *a, const lcg_float *b, lcg_float *c, int n, int bk_size)
{
	int blockSize = bk_size;
	int numBlocks = (n + blockSize - 1) / blockSize;
	vecDvecD_element_wise_device<<<numBlocks, blockSize>>>(a, b, c, n);
	return;
}