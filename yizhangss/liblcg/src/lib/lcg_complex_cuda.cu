#include "hip/hip_runtime.h"
#include "lcg_complex_cuda.h"
#include "complex"
#include "map"

__global__ void smZcsr_get_diagonal_device(const int *A_row, const int *A_col, const hipDoubleComplex *A_val, const int A_len, hipDoubleComplex *A_diag)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < A_len)
	{
		const int num_non0_row = A_row[i + 1] - A_row[i];

		for (int j = 0; j < num_non0_row; j++)
		{
			if (A_col[j + A_row[i]] == i)
			{
				A_diag[i] = A_val[j + A_row[i]];
				break;
			}
		}
	}
	return;
}

__global__ void vecMvecZ_element_wise_device(const hipDoubleComplex *a, const hipDoubleComplex *b, hipDoubleComplex *c, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
		c[i] = hipCmul(a[i], b[i]);
	}
	return;
}

__global__ void vecDvecZ_element_wise_device(const hipDoubleComplex *a, const hipDoubleComplex *b, hipDoubleComplex *c, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
		c[i] = hipCdiv(a[i], b[i]);
	}
	return;
}

__global__ void vecC_conjugate_device(const hipComplex *a, hipComplex *ca, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
		ca[i] = a[i];
		ca[i].y *= -1.0;
	}
	return;
}

__global__ void vecZ_conjugate_device(const hipDoubleComplex *a, hipDoubleComplex *ca, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
		ca[i] = a[i];
		ca[i].y *= -1.0;
	}
	return;
}

lcg_complex cuda2lcg_complex(hipDoubleComplex a)
{
	return lcg_complex(a.x, a.y);
}

#ifdef LibLCG_STD_COMPLEX

hipDoubleComplex lcg2cuda_complex(lcg_complex a)
{
	hipDoubleComplex o;
	o.x = a.real(); o.y = a.imag();
	return o;
}

#else

hipDoubleComplex lcg2cuda_complex(lcg_complex a)
{
	hipDoubleComplex o;
	o.x = a.rel(); o.y = a.img();
	return o;
}

#endif // LibLCG_STD_COMPLEX

hipComplex Cscale(float s, hipComplex a)
{
	hipComplex o;
	o.x = s*a.x;
	o.y = s*a.y;
	return o;
}

hipComplex Csum(hipComplex a, hipComplex b)
{
	hipComplex o;
	o.x = a.x + b.x;
	o.y = a.y + b.y;
	return o;
}

hipComplex Cdiff(hipComplex a, hipComplex b)
{
	hipComplex o;
	o.x = a.x - b.x;
	o.y = a.y - b.y;
	return o;
}

hipComplex Csqrt(hipComplex a)
{
	std::complex<float> c = std::sqrt(std::complex<float>(a.x, a.y));
	hipComplex s;
	s.x = c.real(); s.y = c.imag();
	return s;
}

hipDoubleComplex Zscale(lcg_float s, hipDoubleComplex a)
{
	hipDoubleComplex o;
	o.x = s*a.x;
	o.y = s*a.y;
	return o;
}

hipDoubleComplex Zsum(hipDoubleComplex a, hipDoubleComplex b)
{
	hipDoubleComplex o;
	o.x = a.x + b.x;
	o.y = a.y + b.y;
	return o;
}

hipDoubleComplex Zdiff(hipDoubleComplex a, hipDoubleComplex b)
{
	hipDoubleComplex o;
	o.x = a.x - b.x;
	o.y = a.y - b.y;
	return o;
}

hipDoubleComplex Zsqrt(hipDoubleComplex a)
{
	std::complex<lcg_float> c = std::sqrt(std::complex<lcg_float>(a.x, a.y));
	hipDoubleComplex s;
	s.x = c.real(); s.y = c.imag();
	return s;
}

void smZcoo_row2col(const int *A_row, const int *A_col, const hipDoubleComplex *A, int N, int nz, int *Ac_row, int *Ac_col, hipDoubleComplex *Ac_val)
{
	size_t i, order;
	std::map<size_t, hipDoubleComplex> sort_map;
	std::map<size_t, hipDoubleComplex>::iterator st_iter;

	for (i = 0; i < nz; i++)
	{
		order = N*A_col[i] + A_row[i];
		sort_map[order] = A[i];
	}

	i = 0;
	for (st_iter = sort_map.begin(); st_iter != sort_map.end(); st_iter++)
	{
		order = st_iter->first;
		// exchange the row and column indice to rotate the matrix
		Ac_row[i] = order/N;
		Ac_col[i] = order%N;
		Ac_val[i] = st_iter->second;
		i++;
	}

	sort_map.clear();
	return;
}

void smZcsr_get_diagonal(const int *A_ptr, const int *A_col, const hipDoubleComplex *A_val, const int A_len, hipDoubleComplex *A_diag, int bk_size)
{
	int blockSize = bk_size;
	int numBlocks = (A_len + blockSize - 1) / blockSize;
	smZcsr_get_diagonal_device<<<numBlocks, blockSize>>>(A_ptr, A_col, A_val, A_len, A_diag);
	return;
}

void vecMvecZ_element_wise(const hipDoubleComplex *a, const hipDoubleComplex *b, hipDoubleComplex *c, int n, int bk_size)
{
	int blockSize = bk_size;
	int numBlocks = (n + blockSize - 1) / blockSize;
	vecMvecZ_element_wise_device<<<numBlocks, blockSize>>>(a, b, c, n);
	return;
}

void vecDvecZ_element_wise(const hipDoubleComplex *a, const hipDoubleComplex *b, hipDoubleComplex *c, int n, int bk_size)
{
	int blockSize = bk_size;
	int numBlocks = (n + blockSize - 1) / blockSize;
	vecDvecZ_element_wise_device<<<numBlocks, blockSize>>>(a, b, c, n);
	return;
}

void vecC_conjugate(const hipComplex *a, hipComplex *ca, int n, int bk_size)
{
	int blockSize = bk_size;
	int numBlocks = (n + blockSize - 1) / blockSize;
	vecC_conjugate_device<<<numBlocks, blockSize>>>(a, ca, n);
	return;
}

void vecZ_conjugate(const hipDoubleComplex *a, hipDoubleComplex *ca, int n, int bk_size)
{
	int blockSize = bk_size;
	int numBlocks = (n + blockSize - 1) / blockSize;
	vecZ_conjugate_device<<<numBlocks, blockSize>>>(a, ca, n);
	return;
}