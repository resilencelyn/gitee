#include "hip/hip_runtime.h"
#include "lcg_complex_cuda.h"
#include "complex"
#include "map"

__global__ void smCcsr_get_diagonal_device(const int *A_row, const int *A_col, const hipComplex *A_val, const int A_len, hipComplex *A_diag)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < A_len)
	{
		const int num_non0_row = A_row[i + 1] - A_row[i];

		for (int j = 0; j < num_non0_row; j++)
		{
			if (A_col[j + A_row[i]] == i)
			{
				A_diag[i] = A_val[j + A_row[i]];
				break;
			}
		}
	}
	return;
}

__global__ void smZcsr_get_diagonal_device(const int *A_row, const int *A_col, const hipDoubleComplex *A_val, const int A_len, hipDoubleComplex *A_diag)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < A_len)
	{
		const int num_non0_row = A_row[i + 1] - A_row[i];

		for (int j = 0; j < num_non0_row; j++)
		{
			if (A_col[j + A_row[i]] == i)
			{
				A_diag[i] = A_val[j + A_row[i]];
				break;
			}
		}
	}
	return;
}

__global__ void vecMvecC_element_wise_device(const hipComplex *a, const hipComplex *b, hipComplex *c, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
		c[i] = hipCmulf(a[i], b[i]);
	}
	return;
}

__global__ void vecMvecZ_element_wise_device(const hipDoubleComplex *a, const hipDoubleComplex *b, hipDoubleComplex *c, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
		c[i] = hipCmul(a[i], b[i]);
	}
	return;
}

__global__ void vecDvecC_element_wise_device(const hipComplex *a, const hipComplex *b, hipComplex *c, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
		c[i] = hipCdivf(a[i], b[i]);
	}
	return;
}

__global__ void vecDvecZ_element_wise_device(const hipDoubleComplex *a, const hipDoubleComplex *b, hipDoubleComplex *c, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
		c[i] = hipCdiv(a[i], b[i]);
	}
	return;
}

__global__ void vecC_conjugate_device(const hipComplex *a, hipComplex *ca, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
		ca[i] = a[i];
		ca[i].y *= -1.0;
	}
	return;
}

__global__ void vecZ_conjugate_device(const hipDoubleComplex *a, hipDoubleComplex *ca, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
		ca[i] = a[i];
		ca[i].y *= -1.0;
	}
	return;
}

lcg_complex cuda2lcg_complex(hipDoubleComplex a)
{
	return lcg_complex(a.x, a.y);
}

#ifdef LibLCG_STD_COMPLEX

hipDoubleComplex lcg2cuda_complex(lcg_complex a)
{
	hipDoubleComplex o;
	o.x = a.real(); o.y = a.imag();
	return o;
}

#else

hipDoubleComplex lcg2cuda_complex(lcg_complex a)
{
	hipDoubleComplex o;
	o.x = a.rel(); o.y = a.img();
	return o;
}

#endif // LibLCG_STD_COMPLEX

hipDoubleComplex* clcg_malloc_cuda(size_t n)
{
	hipDoubleComplex *x = new hipDoubleComplex [n];
	return x;
}

void clcg_free_cuda(hipDoubleComplex *x)
{
	if (x != nullptr)
	{
		delete[] x; x = nullptr;
	}
	return;
}

void clcg_vecset_cuda(hipDoubleComplex *a, hipDoubleComplex b, size_t size)
{
	for (size_t i = 0; i < size; i++)
	{
		a[i].x = b.x; a[i].y = b.y;
	}
	return;
}

hipComplex Cscale(float s, hipComplex a)
{
	hipComplex o;
	o.x = s*a.x;
	o.y = s*a.y;
	return o;
}

hipComplex Csum(hipComplex a, hipComplex b)
{
	hipComplex o;
	o.x = a.x + b.x;
	o.y = a.y + b.y;
	return o;
}

hipComplex Cdiff(hipComplex a, hipComplex b)
{
	hipComplex o;
	o.x = a.x - b.x;
	o.y = a.y - b.y;
	return o;
}

hipComplex Csqrt(hipComplex a)
{
	std::complex<float> c = std::sqrt(std::complex<float>(a.x, a.y));
	hipComplex s;
	s.x = c.real(); s.y = c.imag();
	return s;
}

hipDoubleComplex Zscale(lcg_float s, hipDoubleComplex a)
{
	hipDoubleComplex o;
	o.x = s*a.x;
	o.y = s*a.y;
	return o;
}

hipDoubleComplex Zsum(hipDoubleComplex a, hipDoubleComplex b)
{
	hipDoubleComplex o;
	o.x = a.x + b.x;
	o.y = a.y + b.y;
	return o;
}

hipDoubleComplex Zdiff(hipDoubleComplex a, hipDoubleComplex b)
{
	hipDoubleComplex o;
	o.x = a.x - b.x;
	o.y = a.y - b.y;
	return o;
}

hipDoubleComplex Zsqrt(hipDoubleComplex a)
{
	std::complex<lcg_float> c = std::sqrt(std::complex<lcg_float>(a.x, a.y));
	hipDoubleComplex s;
	s.x = c.real(); s.y = c.imag();
	return s;
}

void smCcoo_row2col(const int *A_row, const int *A_col, const hipComplex *A, int N, int nz, int *Ac_row, int *Ac_col, hipComplex *Ac_val)
{
	size_t i, order;
	std::map<size_t, hipComplex> sort_map;
	std::map<size_t, hipComplex>::iterator st_iter;

	for (i = 0; i < nz; i++)
	{
		order = N*A_col[i] + A_row[i];
		sort_map[order] = A[i];
	}

	i = 0;
	for (st_iter = sort_map.begin(); st_iter != sort_map.end(); st_iter++)
	{
		order = st_iter->first;
		// exchange the row and column indice to rotate the matrix
		Ac_row[i] = order/N;
		Ac_col[i] = order%N;
		Ac_val[i] = st_iter->second;
		i++;
	}

	sort_map.clear();
	return;
}

void smZcoo_row2col(const int *A_row, const int *A_col, const hipDoubleComplex *A, int N, int nz, int *Ac_row, int *Ac_col, hipDoubleComplex *Ac_val)
{
	size_t i, order;
	std::map<size_t, hipDoubleComplex> sort_map;
	std::map<size_t, hipDoubleComplex>::iterator st_iter;

	for (i = 0; i < nz; i++)
	{
		order = N*A_col[i] + A_row[i];
		sort_map[order] = A[i];
	}

	i = 0;
	for (st_iter = sort_map.begin(); st_iter != sort_map.end(); st_iter++)
	{
		order = st_iter->first;
		// exchange the row and column indice to rotate the matrix
		Ac_row[i] = order/N;
		Ac_col[i] = order%N;
		Ac_val[i] = st_iter->second;
		i++;
	}

	sort_map.clear();
	return;
}

void smCcsr_get_diagonal(const int *A_ptr, const int *A_col, const hipComplex *A_val, const int A_len, hipComplex *A_diag, int bk_size)
{
	int blockSize = bk_size;
	int numBlocks = (A_len + blockSize - 1) / blockSize;
	smCcsr_get_diagonal_device<<<numBlocks, blockSize>>>(A_ptr, A_col, A_val, A_len, A_diag);
	return;
}

void smZcsr_get_diagonal(const int *A_ptr, const int *A_col, const hipDoubleComplex *A_val, const int A_len, hipDoubleComplex *A_diag, int bk_size)
{
	int blockSize = bk_size;
	int numBlocks = (A_len + blockSize - 1) / blockSize;
	smZcsr_get_diagonal_device<<<numBlocks, blockSize>>>(A_ptr, A_col, A_val, A_len, A_diag);
	return;
}

void vecMvecC_element_wise(const hipComplex *a, const hipComplex *b, hipComplex *c, int n, int bk_size)
{
	int blockSize = bk_size;
	int numBlocks = (n + blockSize - 1) / blockSize;
	vecMvecC_element_wise_device<<<numBlocks, blockSize>>>(a, b, c, n);
	return;
}

void vecMvecZ_element_wise(const hipDoubleComplex *a, const hipDoubleComplex *b, hipDoubleComplex *c, int n, int bk_size)
{
	int blockSize = bk_size;
	int numBlocks = (n + blockSize - 1) / blockSize;
	vecMvecZ_element_wise_device<<<numBlocks, blockSize>>>(a, b, c, n);
	return;
}

void vecDvecC_element_wise(const hipComplex *a, const hipComplex *b, hipComplex *c, int n, int bk_size)
{
	int blockSize = bk_size;
	int numBlocks = (n + blockSize - 1) / blockSize;
	vecDvecC_element_wise_device<<<numBlocks, blockSize>>>(a, b, c, n);
	return;
}

void vecDvecZ_element_wise(const hipDoubleComplex *a, const hipDoubleComplex *b, hipDoubleComplex *c, int n, int bk_size)
{
	int blockSize = bk_size;
	int numBlocks = (n + blockSize - 1) / blockSize;
	vecDvecZ_element_wise_device<<<numBlocks, blockSize>>>(a, b, c, n);
	return;
}

void vecC_conjugate(const hipComplex *a, hipComplex *ca, int n, int bk_size)
{
	int blockSize = bk_size;
	int numBlocks = (n + blockSize - 1) / blockSize;
	vecC_conjugate_device<<<numBlocks, blockSize>>>(a, ca, n);
	return;
}

void vecZ_conjugate(const hipDoubleComplex *a, hipDoubleComplex *ca, int n, int bk_size)
{
	int blockSize = bk_size;
	int numBlocks = (n + blockSize - 1) / blockSize;
	vecZ_conjugate_device<<<numBlocks, blockSize>>>(a, ca, n);
	return;
}