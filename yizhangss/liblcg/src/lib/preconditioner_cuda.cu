#include "preconditioner_cuda.h"
#include "map"

void incomplete_Cholesky_cuda_half_buffsize(const int *row, const int *col, int nz_size, int *lnz_size)
{
    size_t c = 0;
    for (size_t i = 0; i < nz_size; i++)
    {
        if (row[i] >= col[i])
        {
            c++;
        }
    }
    *lnz_size = c;
    return;
}

void incomplete_Cholesky_cuda_half(const int *row, const int *col, const hipComplex *val, int N, int nz_size, 
    int lnz_size, int *IC_row, int *IC_col, hipComplex *IC_val)
{
    // We use this to store diagonal elements of the factorizated lower triangular matrix
    hipComplex *diagonal = new hipComplex [N];
    // A temporary row
    hipComplex *tmp_row = new hipComplex [N];
    // index of non-zero elements in tmp_row
    int *filled_idx = new int [N];
    // Begining index of each row in the input matrix
    int *row_st_idx = new int [N];

    size_t i, j, f;

    // Set initial values
    for (i = 0; i < N; i++)
    {
        diagonal[i].x = 0.0; diagonal[i].y = 0.0;
        tmp_row[i].x = 0.0; tmp_row[i].y = 0.0;
        filled_idx[i] = -1;
        row_st_idx[i] = -1;
    }

    // copy elements in the lower triangle to the output matrix
    j = 0;
    for (i = 0; i < nz_size; i++)
    {
        if (row[i] >= col[i])
        {
            IC_row[j] = row[i];
            IC_col[j] = col[i];
            IC_val[j] = val[i];
            j++;
        }
    }

    // Get the begining index of each row in the matrix
    j = 1;
    row_st_idx[0] = IC_row[0];
    size_t old_row = IC_row[0];
    for (i = 1; i < lnz_size; i++)
    {
        if (IC_row[i] > old_row)
        {
            row_st_idx[j] = i;
            old_row = IC_row[i];
            j++;
        }
    }

    // Calculate the first element
    IC_val[0] = Csqrt(IC_val[0]);
    diagonal[0] = IC_val[0];

    hipComplex dia_sum;
    dia_sum.x = 0.0; dia_sum.y = 0.0;
    // The first one is already calculated
    for (i = 1; i < lnz_size; i++)
    {
        // Calculate the first column if there is one
        if (IC_col[i] == 0)
        {
            IC_val[i] = hipCdivf(IC_val[i], IC_val[0]);
            dia_sum = Csum(dia_sum, hipCmulf(IC_val[i], IC_val[i]));
            continue; // Case 1 break
        }
        
        // Calculate elements in the middle of a row
        if (IC_row[i] > IC_col[i])
        {
            // Find needed values from previous elements
            f = 0;
            j = row_st_idx[IC_col[i]];
            while (IC_col[j] < IC_col[i])
            {
                tmp_row[IC_col[j]] = IC_val[j];
                filled_idx[f]  = IC_col[j];
                f++;
                j++;
            }

            j = row_st_idx[IC_row[i]];
            while (IC_col[j] < IC_col[i])
            {
                IC_val[i] = Cdiff(IC_val[i], hipCmulf(IC_val[j], tmp_row[IC_col[j]]));
                j++;
            }
            
            IC_val[i] = hipCdivf(IC_val[i], diagonal[IC_col[i]]);
            dia_sum = Csum(dia_sum, hipCmulf(IC_val[i], IC_val[i]));

            // reset tmp variables
            for (j = 0; j < f; j++)
            {
                tmp_row[filled_idx[j]].x = 0.0; tmp_row[filled_idx[j]].y = 0.0;
            }

            continue; // Case 2 break
        }
        
        // We have rearched the diagonal position
        if (IC_row[i] == IC_col[i])
        {
            IC_val[i] = Csqrt(Cdiff(IC_val[i], dia_sum));
            diagonal[IC_col[i]] = IC_val[i];
            dia_sum.x = 0.0; dia_sum.y = 0.0;
        }
    }

    delete[] diagonal;
    delete[] tmp_row;
    delete[] row_st_idx;
    delete[] filled_idx;
    return;
}

void incomplete_Cholesky_cuda_half(const int *row, const int *col, const hipDoubleComplex *val, int N, int nz_size, 
    int lnz_size, int *IC_row, int *IC_col, hipDoubleComplex *IC_val)
{
    // We use this to store diagonal elements of the factorizated lower triangular matrix
    hipDoubleComplex *diagonal = new hipDoubleComplex [N];
    // A temporary row
    hipDoubleComplex *tmp_row = new hipDoubleComplex [N];
    // index of non-zero elements in tmp_row
    int *filled_idx = new int [N];
    // Begining index of each row in the input matrix
    int *row_st_idx = new int [N];

    size_t i, j, f;

    // Set initial values
    for (i = 0; i < N; i++)
    {
        diagonal[i].x = 0.0; diagonal[i].y = 0.0;
        tmp_row[i].x = 0.0; tmp_row[i].y = 0.0;
        filled_idx[i] = -1;
        row_st_idx[i] = -1;
    }

    // copy elements in the lower triangle to the output matrix
    j = 0;
    for (i = 0; i < nz_size; i++)
    {
        if (row[i] >= col[i])
        {
            IC_row[j] = row[i];
            IC_col[j] = col[i];
            IC_val[j] = val[i];
            j++;
        }
    }

    // Get the begining index of each row in the matrix
    j = 1;
    row_st_idx[0] = IC_row[0];
    size_t old_row = IC_row[0];
    for (i = 1; i < lnz_size; i++)
    {
        if (IC_row[i] > old_row)
        {
            row_st_idx[j] = i;
            old_row = IC_row[i];
            j++;
        }
    }

    // Calculate the first element
    IC_val[0] = Zsqrt(IC_val[0]);
    diagonal[0] = IC_val[0];

    hipDoubleComplex dia_sum;
    dia_sum.x = 0.0; dia_sum.y = 0.0;
    // The first one is already calculated
    for (i = 1; i < lnz_size; i++)
    {
        // Calculate the first column if there is one
        if (IC_col[i] == 0)
        {
            IC_val[i] = hipCdiv(IC_val[i], IC_val[0]);
            dia_sum = Zsum(dia_sum, hipCmul(IC_val[i], IC_val[i]));
            continue; // Case 1 break
        }
        
        // Calculate elements in the middle of a row
        if (IC_row[i] > IC_col[i])
        {
            // Find needed values from previous elements
            f = 0;
            j = row_st_idx[IC_col[i]];
            while (IC_col[j] < IC_col[i])
            {
                tmp_row[IC_col[j]] = IC_val[j];
                filled_idx[f]  = IC_col[j];
                f++;
                j++;
            }

            j = row_st_idx[IC_row[i]];
            while (IC_col[j] < IC_col[i])
            {
                IC_val[i] = Zdiff(IC_val[i], hipCmul(IC_val[j], tmp_row[IC_col[j]]));
                j++;
            }
            
            IC_val[i] = hipCdiv(IC_val[i], diagonal[IC_col[i]]);
            dia_sum = Zsum(dia_sum, hipCmul(IC_val[i], IC_val[i]));

            // reset tmp variables
            for (j = 0; j < f; j++)
            {
                tmp_row[filled_idx[j]].x = 0.0; tmp_row[filled_idx[j]].y = 0.0;
            }

            continue; // Case 2 break
        }
        
        // We have rearched the diagonal position
        if (IC_row[i] == IC_col[i])
        {
            IC_val[i] = Zsqrt(Zdiff(IC_val[i], dia_sum));
            diagonal[IC_col[i]] = IC_val[i];
            dia_sum.x = 0.0; dia_sum.y = 0.0;
        }
    }

    delete[] diagonal;
    delete[] tmp_row;
    delete[] row_st_idx;
    delete[] filled_idx;
    return;
}

void incomplete_Cholesky_cuda_full(const int *row, const int *col, const hipDoubleComplex *val, int N, int nz_size, int *IC_row, int *IC_col, hipDoubleComplex *IC_val)
{
    // We use this to store diagonal elements of the factorizated lower triangular matrix
    hipDoubleComplex *diagonal = new hipDoubleComplex [N];
    // A temporary row
    hipDoubleComplex *tmp_row = new hipDoubleComplex [N];
    // index of non-zero elements in tmp_row
    int *filled_idx = new int [N];
    // Begining index of each row in the input matrix
    int *row_st_idx = new int [N];

    size_t i, j, f, l;

    // Set initial values
    for (i = 0; i < N; i++)
    {
        diagonal[i].x = 0.0; diagonal[i].y = 0.0;
        tmp_row[i].x = 0.0; tmp_row[i].y = 0.0;
        filled_idx[i] = -1;
        row_st_idx[i] = -1;
    }

    // copy elements to the output matrix
    for (i = 0; i < nz_size; i++)
    {
        IC_row[i] = row[i];
        IC_col[i] = col[i];
        IC_val[i] = val[i];
    }

    // count element number in the lower triangular part (including the diagonal) and the upper triangular part (excluding the diagonal)
    // build map from elements' cooridnate to their index in the array
    size_t order, L_nz = 0;
    std::map<size_t, size_t> index_map;

    for (i = 0; i < nz_size; i++)
    {
        if (row[i] >= col[i]) // Count number for thr lower triangular part
        {
            L_nz++;
        }
        else // Only need to build the map for the upper triangular part
        {
            order = N*row[i] + col[i];
            index_map[order] = i;
        }
    }

    // We use to store element index in the lower triangle
    j = 0;
    size_t *low_idx = new size_t [L_nz];
    for (i = 0; i < nz_size; i++)
    {
        if (row[i] >= col[i])
        {
            low_idx[j] = i;
            j++;
        }
    }

    // Get the begining index of each row in the matrix
    j = 1;
    row_st_idx[0] = IC_row[0];
    size_t old_row = IC_row[0];
    for (i = 1; i < nz_size; i++)
    {
        if (IC_row[i] > old_row)
        {
            row_st_idx[j] = i;
            old_row = IC_row[i];
            j++;
        }
    }

    // Calculate the first element
    IC_val[0] = Zsqrt(IC_val[0]);
    diagonal[0] = IC_val[0];

    hipDoubleComplex dia_sum;
    dia_sum.x = 0.0; dia_sum.y = 0.0;
    // The first one is already calculated
    for (i = 1; i < L_nz; i++)
    {
        l = low_idx[i];

        // Calculate the first column if there is one
        if (IC_col[l] == 0)
        {
            IC_val[l] = hipCdiv(IC_val[l], IC_val[0]);
            dia_sum = Zsum(dia_sum, hipCmul(IC_val[l], IC_val[l]));
            // Set value at the upper triangle
            order = IC_row[l];
            IC_val[index_map[order]] = IC_val[l];
            continue; // Case 1 break
        }
        
        // Calculate elements in the middle of a row
        if (IC_row[l] > IC_col[l])
        {
            // Find needed values from previous elements
            f = 0;
            j = row_st_idx[IC_col[l]];
            while (IC_col[j] < IC_col[l])
            {
                tmp_row[IC_col[j]] = IC_val[j];
                filled_idx[f]  = IC_col[j];
                f++;
                j++;
            }

            j = row_st_idx[IC_row[l]];
            while (IC_col[j] < IC_col[l])
            {
                IC_val[l] = Zdiff(IC_val[l], hipCmul(IC_val[j], tmp_row[IC_col[j]]));
                j++;
            }
            
            IC_val[l] = hipCdiv(IC_val[l], diagonal[IC_col[l]]);
            dia_sum = Zsum(dia_sum, hipCmul(IC_val[l], IC_val[l]));

            // Set value at the upper triangle
            order = N*IC_col[l] + IC_row[l];
            IC_val[index_map[order]] = IC_val[l];

            // reset tmp variables
            for (j = 0; j < f; j++)
            {
                tmp_row[filled_idx[j]].x = 0.0; tmp_row[filled_idx[j]].y = 0.0;
            }

            continue; // Case 2 break
        }
        
        // We have rearched the diagonal position
        if (IC_row[l] == IC_col[l])
        {
            IC_val[l] = Zsqrt(Zdiff(IC_val[l], dia_sum));
            diagonal[IC_col[l]] = IC_val[l];
            dia_sum.x = 0.0; dia_sum.y = 0.0;
        }
    }

    delete[] diagonal;
    delete[] tmp_row;
    delete[] row_st_idx;
    delete[] filled_idx;
    delete[] low_idx;
    index_map.clear();
    return;
}