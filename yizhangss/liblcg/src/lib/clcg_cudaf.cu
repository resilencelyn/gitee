#include "hip/hip_runtime.h"
#include "cmath"
#include "ctime"
#include "iostream"

#include "clcg_cudaf.h"


typedef int (*cuda_solver_ptr)(clcg_axfunc_cudaf_ptr Afp, clcg_progress_cudaf_ptr Pfp, hipComplex* m, 
    const hipComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

int clbicg(clcg_axfunc_cudaf_ptr Afp, clcg_progress_cudaf_ptr Pfp, hipComplex* m, 
    const hipComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

int clbicg_symmetric(clcg_axfunc_cudaf_ptr Afp, clcg_progress_cudaf_ptr Pfp, hipComplex* m, 
    const hipComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

int clcg_solver_cuda(clcg_axfunc_cudaf_ptr Afp, clcg_progress_cudaf_ptr Pfp, hipComplex* m, const hipComplex* B, 
    const int n_size, const int nz_size, const clcg_para* param, void* instance, hipblasHandle_t cub_handle, 
    hipsparseHandle_t cus_handle, clcg_solver_enum solver_id)
{
    cuda_solver_ptr cg_solver;
    switch (solver_id)
	{
		case CLCG_BICG:
			cg_solver = clbicg;
			break;
		case CLCG_BICG_SYM:
			cg_solver = clbicg_symmetric;
			break;
		default:
			return CLCG_UNKNOWN_SOLVER;
	}

	return cg_solver(Afp, Pfp, m, B, n_size, nz_size, param, instance, cub_handle, cus_handle);
}

typedef int (*cuda_precondtioned_solver_ptr)(clcg_axfunc_cudaf_ptr Afp, clcg_axfunc_cudaf_ptr Mfp, clcg_progress_cudaf_ptr Pfp, 
    hipComplex* m, const hipComplex* B, const int n_size, const int nz_size, const clcg_para* param, 
    void* instance, hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

int clpcg(clcg_axfunc_cudaf_ptr Afp, clcg_axfunc_cudaf_ptr Mfp, clcg_progress_cudaf_ptr Pfp, hipComplex* m, 
    const hipComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

int clcg_solver_preconditioned_cuda(clcg_axfunc_cudaf_ptr Afp, clcg_axfunc_cudaf_ptr Mfp, clcg_progress_cudaf_ptr Pfp, 
    hipComplex* m, const hipComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, clcg_solver_enum solver_id)
{
    cuda_precondtioned_solver_ptr cgp_solver;
    switch (solver_id)
	{
		case CLCG_PCG:
			cgp_solver = clpcg; break;
		default:
			return CLCG_UNKNOWN_SOLVER;
	}

	return cgp_solver(Afp, Mfp, Pfp, m, B, n_size, nz_size, param, instance, cub_handle, cus_handle);
}

int clbicg(clcg_axfunc_cudaf_ptr Afp, clcg_progress_cudaf_ptr Pfp, hipComplex* m, 
    const hipComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle)
{
    // set CGS parameters
	clcg_para para = (param != nullptr) ? (*param) : defparam2;

	//check parameters
	if (n_size <= 0) return CLCG_INVILAD_VARIABLE_SIZE;
	if (para.max_iterations < 0) return CLCG_INVILAD_MAX_ITERATIONS;
	if (para.epsilon <= 0.0 || para.epsilon >= 1.0) return CLCG_INVILAD_EPSILON;

	if (m == nullptr) return CLCG_INVALID_POINTER;
	if (B == nullptr) return CLCG_INVALID_POINTER;
    if (cub_handle == nullptr) return LCG_INVALID_POINTER;
    if (cus_handle == nullptr) return LCG_INVALID_POINTER;

	hipComplex *r1k = nullptr, *r2k = nullptr;
	hipComplex *d1k = nullptr, *d2k = nullptr, *Ax = nullptr;
    hipMalloc(&r1k, n_size * sizeof(hipComplex));
	hipMalloc(&r2k, n_size * sizeof(hipComplex));
    hipMalloc(&d1k, n_size * sizeof(hipComplex));
	hipMalloc(&d2k, n_size * sizeof(hipComplex));
    hipMalloc(&Ax, n_size * sizeof(hipComplex));

    hipsparseDnVecDescr_t dvec_m, dvec_d1k, dvec_d2k, dvec_Ax;
	hipsparseCreateDnVec(&dvec_m, n_size, m, HIP_C_32F);
	hipsparseCreateDnVec(&dvec_d1k, n_size, d1k, HIP_C_32F);
	hipsparseCreateDnVec(&dvec_d2k, n_size, d2k, HIP_C_32F);
	hipsparseCreateDnVec(&dvec_Ax, n_size, Ax, HIP_C_32F);

    hipComplex one, none;
    one.x = 1.0; one.y = 0.0;
    none.x = -1.0; none.y = 0.0;
	hipComplex ak, nak, conj_ak, Ad1d2, r1r2_next, betak, conj_betak;

	Afp(instance, cub_handle, cus_handle, dvec_m, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);

    // r0 = B - Ax
    hipMemcpy(r1k, B, n_size * sizeof(hipComplex), hipMemcpyDeviceToDevice); // r0 = B
    hipblasCaxpy(cub_handle, n_size, &none, Ax, 1, r1k, 1); // r0 -= Ax
    hipMemcpy(d1k, r1k, n_size * sizeof(hipComplex), hipMemcpyDeviceToDevice); // d0 = r0

	vecC_conjugate(r1k, r2k, n_size);
	hipMemcpy(d2k, r2k, n_size * sizeof(hipComplex), hipMemcpyDeviceToDevice);

	hipComplex r1r2;
    hipblasCdotc(cub_handle, n_size, r2k, 1, r1k, 1, &r1r2);

	float m_mod;
    if (!para.abs_diff)
    {
        hipblasScnrm2(cub_handle, n_size, m, 1, &m_mod);
        if (m_mod < 1.0) m_mod = 1.0;
    }

	float rk_mod;
	hipblasScnrm2(cub_handle, n_size, r1k, 1, &rk_mod);

	int ret, t = 0;
	if (para.abs_diff && rk_mod/n_size <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, m, rk_mod/n_size, &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}	
	else if (rk_mod*rk_mod/(m_mod*m_mod) <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, m, rk_mod*rk_mod/(m_mod*m_mod), &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}

	float residual;
	while(1)
	{
		if (para.abs_diff) residual = rk_mod/n_size;
		else residual = rk_mod*rk_mod/(m_mod*m_mod);

		if (Pfp != nullptr)
		{
			if (Pfp(instance, m, residual, &para, n_size, nz_size, t))
			{
				ret = CLCG_STOP; goto func_ends;
			}
		}

		if (residual <= para.epsilon)
		{
			ret = CLCG_CONVERGENCE; goto func_ends;
		}

		if (para.max_iterations > 0 && t+1 > para.max_iterations)
		{
			ret = LCG_REACHED_MAX_ITERATIONS;
			break;
		}
		
		t++;

        Afp(instance, cub_handle, cus_handle, dvec_d1k, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);
        
        hipblasCdotc(cub_handle, n_size, d2k, 1, Ax, 1, &Ad1d2);
        ak = hipCdivf(r1r2, Ad1d2);
        nak = hipCmulf(none, ak);
		conj_ak = hipConjf(nak);

        hipblasCaxpy(cub_handle, n_size, &ak, d1k, 1, m, 1);
        hipblasCaxpy(cub_handle, n_size, &nak, Ax, 1, r1k, 1);

        if (!para.abs_diff)
        {
            hipblasScnrm2(cub_handle, n_size, m, 1, &m_mod);
            if (m_mod < 1.0) m_mod = 1.0;
        }

        hipblasScnrm2(cub_handle, n_size, r1k, 1, &rk_mod);

		Afp(instance, cub_handle, cus_handle, dvec_d2k, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE);

		hipblasCaxpy(cub_handle, n_size, &conj_ak, Ax, 1, r2k, 1);

		hipblasCdotc(cub_handle, n_size, r2k, 1, r1k, 1, &r1r2_next);
		betak = hipCdivf(r1r2_next, r1r2);
		conj_betak = hipConjf(betak);
		r1r2 = r1r2_next;

        hipblasCscal(cub_handle, n_size, &betak, d1k, 1);
        hipblasCaxpy(cub_handle, n_size, &one, r1k, 1, d1k, 1);

		hipblasCscal(cub_handle, n_size, &conj_betak, d2k, 1);
        hipblasCaxpy(cub_handle, n_size, &one, r2k, 1, d2k, 1);
	}

	func_ends:
	{
		hipFree(r1k);
		hipFree(r2k);
		hipFree(d1k);
		hipFree(d2k);	
		hipFree(Ax);
        hipsparseDestroyDnVec(dvec_m);
        hipsparseDestroyDnVec(dvec_d1k);
		hipsparseDestroyDnVec(dvec_d2k);
        hipsparseDestroyDnVec(dvec_Ax);
	}

	return ret;
}

int clbicg_symmetric(clcg_axfunc_cudaf_ptr Afp, clcg_progress_cudaf_ptr Pfp, hipComplex* m, 
    const hipComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle)
{
    // set CGS parameters
	clcg_para para = (param != nullptr) ? (*param) : defparam2;

	//check parameters
	if (n_size <= 0) return CLCG_INVILAD_VARIABLE_SIZE;
	if (para.max_iterations < 0) return CLCG_INVILAD_MAX_ITERATIONS;
	if (para.epsilon <= 0.0 || para.epsilon >= 1.0) return CLCG_INVILAD_EPSILON;

	if (m == nullptr) return CLCG_INVALID_POINTER;
	if (B == nullptr) return CLCG_INVALID_POINTER;
    if (cub_handle == nullptr) return LCG_INVALID_POINTER;
    if (cus_handle == nullptr) return LCG_INVALID_POINTER;

	hipComplex *rk = nullptr, *dk = nullptr, *Ax = nullptr;
    hipMalloc(&rk, n_size * sizeof(hipComplex));
    hipMalloc(&dk, n_size * sizeof(hipComplex));
    hipMalloc(&Ax, n_size * sizeof(hipComplex));

    hipsparseDnVecDescr_t dvec_m, dvec_dk, dvec_Ax;
	hipsparseCreateDnVec(&dvec_m, n_size, m, HIP_C_32F);
	hipsparseCreateDnVec(&dvec_dk, n_size, dk, HIP_C_32F);
	hipsparseCreateDnVec(&dvec_Ax, n_size, Ax, HIP_C_32F);

    hipComplex one, none;
    one.x = 1.0; one.y = 0.0;
    none.x = -1.0; none.y = 0.0;
	hipComplex ak, nak, rkrk2, betak, dkAx;

	Afp(instance, cub_handle, cus_handle, dvec_m, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);

    // r0 = B - Ax
    hipMemcpy(rk, B, n_size * sizeof(hipComplex), hipMemcpyDeviceToDevice); // r0 = B
    hipblasCaxpy(cub_handle, n_size, &none, Ax, 1, rk, 1); // r0 -= Ax
    hipMemcpy(dk, rk, n_size * sizeof(hipComplex), hipMemcpyDeviceToDevice); // d0 = r0

	hipComplex rkrk;
    hipblasCdotu(cub_handle, n_size, rk, 1, rk, 1, &rkrk);

	float m_mod;
    if (!para.abs_diff)
    {
        hipblasScnrm2(cub_handle, n_size, m, 1, &m_mod);
        if (m_mod < 1.0) m_mod = 1.0;
    }

	float rk_mod;
	hipblasScnrm2(cub_handle, n_size, rk, 1, &rk_mod);

	int ret, t = 0;
	if (para.abs_diff && rk_mod/n_size <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, m, rk_mod/n_size, &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}	
	else if (rk_mod*rk_mod/(m_mod*m_mod) <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, m, rk_mod*rk_mod/(m_mod*m_mod), &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}

	float residual;
	while(1)
	{
		if (para.abs_diff) residual = rk_mod/n_size;
		else residual = rk_mod*rk_mod/(m_mod*m_mod);

		if (Pfp != nullptr)
		{
			if (Pfp(instance, m, residual, &para, n_size, nz_size, t))
			{
				ret = CLCG_STOP; goto func_ends;
			}
		}

		if (residual <= para.epsilon)
		{
			ret = CLCG_CONVERGENCE; goto func_ends;
		}

		if (para.max_iterations > 0 && t+1 > para.max_iterations)
		{
			ret = LCG_REACHED_MAX_ITERATIONS;
			break;
		}
		
		t++;

        Afp(instance, cub_handle, cus_handle, dvec_dk, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);
        
        hipblasCdotu(cub_handle, n_size, dk, 1, Ax, 1, &dkAx);
        ak = hipCdivf(rkrk, dkAx);
        nak = hipCmulf(none, ak);

        hipblasCaxpy(cub_handle, n_size, &ak, dk, 1, m, 1);
        hipblasCaxpy(cub_handle, n_size, &nak, Ax, 1, rk, 1);

        if (!para.abs_diff)
        {
            hipblasScnrm2(cub_handle, n_size, m, 1, &m_mod);
            if (m_mod < 1.0) m_mod = 1.0;
        }

        hipblasScnrm2(cub_handle, n_size, rk, 1, &rk_mod);

		hipblasCdotu(cub_handle, n_size, rk, 1, rk, 1, &rkrk2);
		betak = hipCdivf(rkrk2, rkrk);
		rkrk = rkrk2;

        hipblasCscal(cub_handle, n_size, &betak, dk, 1);
        hipblasCaxpy(cub_handle, n_size, &one, rk, 1, dk, 1);
	}

	func_ends:
	{
		hipFree(rk);
		hipFree(dk);
		hipFree(Ax);
        hipsparseDestroyDnVec(dvec_m);
        hipsparseDestroyDnVec(dvec_dk);
        hipsparseDestroyDnVec(dvec_Ax);
	}

	return ret;
}

int clpcg(clcg_axfunc_cudaf_ptr Afp, clcg_axfunc_cudaf_ptr Mfp, clcg_progress_cudaf_ptr Pfp, hipComplex* m, 
    const hipComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle)
{
    // set CGS parameters
	clcg_para para = (param != nullptr) ? (*param) : defparam2;

	//check parameters
	if (n_size <= 0) return CLCG_INVILAD_VARIABLE_SIZE;
	if (para.max_iterations < 0) return CLCG_INVILAD_MAX_ITERATIONS;
	if (para.epsilon <= 0.0 || para.epsilon >= 1.0) return CLCG_INVILAD_EPSILON;

	if (m == nullptr) return CLCG_INVALID_POINTER;
	if (B == nullptr) return CLCG_INVALID_POINTER;
    if (cub_handle == nullptr) return LCG_INVALID_POINTER;
    if (cus_handle == nullptr) return LCG_INVALID_POINTER;

    hipComplex *rk = nullptr, *dk = nullptr, *sk = nullptr, *Ax = nullptr;
    hipMalloc(&rk, n_size * sizeof(hipComplex));
    hipMalloc(&dk, n_size * sizeof(hipComplex));
    hipMalloc(&sk, n_size * sizeof(hipComplex));
    hipMalloc(&Ax, n_size * sizeof(hipComplex));

    hipsparseDnVecDescr_t dvec_m, dvec_rk, dvec_dk, dvec_sk, dvec_Ax;
	hipsparseCreateDnVec(&dvec_m, n_size, m, HIP_C_32F);
    hipsparseCreateDnVec(&dvec_rk, n_size, rk, HIP_C_32F);
	hipsparseCreateDnVec(&dvec_dk, n_size, dk, HIP_C_32F);
    hipsparseCreateDnVec(&dvec_sk, n_size, sk, HIP_C_32F);
	hipsparseCreateDnVec(&dvec_Ax, n_size, Ax, HIP_C_32F);

    hipComplex one, none;
    one.x = 1.0; one.y = 0.0;
    none.x = -1.0; none.y = 0.0;
    hipComplex ak, nak, d_old, betak, dkAx;

    Afp(instance, cub_handle, cus_handle, dvec_m, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);

    // r0 = B - Ax
    hipMemcpy(rk, B, n_size * sizeof(hipComplex), hipMemcpyDeviceToDevice); // r0 = B
    hipblasCaxpy(cub_handle, n_size, &none, Ax, 1, rk, 1); // r0 -= Ax

	Mfp(instance, cub_handle, cus_handle, dvec_rk, dvec_dk, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);

	hipComplex d_new;
    hipblasCdotu(cub_handle, n_size, rk, 1, dk, 1, &d_new);

    float m_mod;
    if (!para.abs_diff)
    {
        hipblasScnrm2(cub_handle, n_size, m, 1, &m_mod);
        if (m_mod < 1.0) m_mod = 1.0;
    }

	float rk_mod;
	hipblasScnrm2(cub_handle, n_size, rk, 1, &rk_mod);

    int ret, t = 0;
	if (para.abs_diff && rk_mod/n_size <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, m, rk_mod/n_size, &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}	
	else if (rk_mod*rk_mod/(m_mod*m_mod) <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, m, rk_mod*rk_mod/(m_mod*m_mod), &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}

	float residual;
	while(1)
	{
		if (para.abs_diff) residual = rk_mod/n_size;
		else residual = rk_mod*rk_mod/(m_mod*m_mod);

		if (Pfp != nullptr)
		{
			if (Pfp(instance, m, residual, &para, n_size, nz_size, t))
			{
				ret = CLCG_STOP; goto func_ends;
			}
		}

		if (residual <= para.epsilon)
		{
			ret = CLCG_CONVERGENCE; goto func_ends;
		}

		if (para.max_iterations > 0 && t+1 > para.max_iterations)
		{
			ret = LCG_REACHED_MAX_ITERATIONS;
			break;
		}
		
		t++;

        Afp(instance, cub_handle, cus_handle, dvec_dk, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);
        hipblasCdotu(cub_handle, n_size, dk, 1, Ax, 1, &dkAx);
		ak = hipCdivf(d_new, dkAx);
        nak = hipCmulf(none, ak);

        hipblasCaxpy(cub_handle, n_size, &ak, dk, 1, m, 1);
        hipblasCaxpy(cub_handle, n_size, &nak, Ax, 1, rk, 1);

		if (!para.abs_diff)
        {
            hipblasScnrm2(cub_handle, n_size, m, 1, &m_mod);
            if (m_mod < 1.0) m_mod = 1.0;
        }

        hipblasScnrm2(cub_handle, n_size, rk, 1, &rk_mod);

        Mfp(instance, cub_handle, cus_handle, dvec_rk, dvec_sk, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);

		d_old = d_new;
        hipblasCdotu(cub_handle, n_size, rk, 1, sk, 1, &d_new);

		betak = hipCdivf(d_new, d_old);

        hipblasCscal(cub_handle, n_size, &betak, dk, 1);
        hipblasCaxpy(cub_handle, n_size, &one, sk, 1, dk, 1);
	}

	func_ends:
	{
		hipFree(rk);
		hipFree(dk);
		hipFree(sk);
		hipFree(Ax);
        hipsparseDestroyDnVec(dvec_m);
        hipsparseDestroyDnVec(dvec_rk);
        hipsparseDestroyDnVec(dvec_dk);
        hipsparseDestroyDnVec(dvec_sk);
        hipsparseDestroyDnVec(dvec_Ax);
	}

	return ret;
}


CLCG_CUDAF_Solver::CLCG_CUDAF_Solver()
{
	param_ = clcg_default_parameters();
	inter_ = 1;
	silent_ = false;
}

int CLCG_CUDAF_Solver::Progress(const hipComplex* m, const float converge, 
	const clcg_para* param, const int n_size, const int nz_size, const int k)
{
	if (inter_ > 0 && k%inter_ == 0)
	{
		std::clog << "\rIteration-times: " << k << "\tconvergence: " << converge;
		return 0;
	}

	if (converge <= param->epsilon)
	{
		std::clog << "\rIteration-times: " << k << "\tconvergence: " << converge;
	}
	return 0;
}

void CLCG_CUDAF_Solver::silent()
{
	silent_ = true;
	return;
}

void CLCG_CUDAF_Solver::set_report_interval(unsigned int inter)
{
	inter_ = inter;
	return;
}

void CLCG_CUDAF_Solver::set_clcg_parameter(const clcg_para &in_param)
{
	param_ = in_param;
	return;
}

void CLCG_CUDAF_Solver::Minimize(hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, hipComplex *x, hipComplex *b, 
	const int n_size, const int nz_size, clcg_solver_enum solver_id, bool verbose, bool er_throw)
{
	if (silent_)
	{
		int ret = clcg_solver_cuda(_AxProduct, nullptr, x, b, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
		if (ret < 0) lcg_error_str(ret, true);
		return;
	}
	
	// 使用lcg求解 注意当我们使用函数指针来调用求解函数时默认参数不可以省略
	clock_t start = clock();
	int ret = clcg_solver_cuda(_AxProduct, _Progress, x, b, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
	clock_t end = clock();

	float costime = 1000*(end-start)/(double)CLOCKS_PER_SEC;
	
	if (!er_throw)
	{
		std::clog << std::endl;
		switch (solver_id)
		{
			case CLCG_BICG:
				std::clog << "Solver: BI-CG. Time cost: " << costime << " ms" << std::endl;
				break;
			case CLCG_BICG_SYM:
				std::clog << "Solver: BI-CG (symmetrically accelerated). Time cost: " << costime << " ms" << std::endl;
				break;
			default:
				std::clog << "Solver: Unknown. Time cost: " << costime << " ms" << std::endl;
				break;
		}	
	}

	if (verbose) lcg_error_str(ret, er_throw);
	else if (ret < 0) lcg_error_str(ret, er_throw);
	return;
}

void CLCG_CUDAF_Solver::MinimizePreconditioned(hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, hipComplex *x, hipComplex *b, 
	const int n_size, const int nz_size, clcg_solver_enum solver_id, bool verbose, bool er_throw)
{
	if (silent_)
	{
		int ret = clcg_solver_preconditioned_cuda(_AxProduct, _MxProduct, nullptr, x, b, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
		if (ret < 0) lcg_error_str(ret, true);
		return;
	}
	
	// 使用lcg求解 注意当我们使用函数指针来调用求解函数时默认参数不可以省略
	clock_t start = clock();
	int ret = clcg_solver_preconditioned_cuda(_AxProduct, _MxProduct, _Progress, x, b, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
	clock_t end = clock();

	float costime = 1000*(end-start)/(double)CLOCKS_PER_SEC;
	
	if (!er_throw)
	{
		std::clog << std::endl;
		switch (solver_id)
		{
			case CLCG_PCG:
				std::clog << "Solver: PCG. Time cost: " << costime << " ms" << std::endl;
				break;
			default:
				std::clog << "Solver: Unknown. Time cost: " << costime << " ms" << std::endl;
				break;
		}	
	}

	if (verbose) lcg_error_str(ret, er_throw);
	else if (ret < 0) lcg_error_str(ret, er_throw);
	return;
}