#include "hip/hip_runtime.h"
#include "cmath"
#include "ctime"
#include "iostream"

#include "clcg_cuda.h"


typedef int (*cuda_solver_ptr)(clcg_axfunc_cuda_ptr Afp, clcg_progress_cuda_ptr Pfp, hipDoubleComplex* m, 
    const hipDoubleComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

int clbicg(clcg_axfunc_cuda_ptr Afp, clcg_progress_cuda_ptr Pfp, hipDoubleComplex* m, 
    const hipDoubleComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

int clbicg_symmetric(clcg_axfunc_cuda_ptr Afp, clcg_progress_cuda_ptr Pfp, hipDoubleComplex* m, 
    const hipDoubleComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

int clcg_solver_cuda(clcg_axfunc_cuda_ptr Afp, clcg_progress_cuda_ptr Pfp, hipDoubleComplex* m, const hipDoubleComplex* B, 
    const int n_size, const int nz_size, const clcg_para* param, void* instance, hipblasHandle_t cub_handle, 
    hipsparseHandle_t cus_handle, clcg_solver_enum solver_id)
{
    cuda_solver_ptr cg_solver;
    switch (solver_id)
	{
		case CLCG_BICG:
			cg_solver = clbicg;
			break;
		case CLCG_BICG_SYM:
			cg_solver = clbicg_symmetric;
			break;
		default:
			return CLCG_UNKNOWN_SOLVER;
	}

	return cg_solver(Afp, Pfp, m, B, n_size, nz_size, param, instance, cub_handle, cus_handle);
}

typedef int (*cuda_precondtioned_solver_ptr)(clcg_axfunc_cuda_ptr Afp, clcg_axfunc_cuda_ptr Mfp, clcg_progress_cuda_ptr Pfp, 
    hipDoubleComplex* m, const hipDoubleComplex* B, const int n_size, const int nz_size, const clcg_para* param, 
    void* instance, hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

int clpcg(clcg_axfunc_cuda_ptr Afp, clcg_axfunc_cuda_ptr Mfp, clcg_progress_cuda_ptr Pfp, hipDoubleComplex* m, 
    const hipDoubleComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

int clcg_solver_preconditioned_cuda(clcg_axfunc_cuda_ptr Afp, clcg_axfunc_cuda_ptr Mfp, clcg_progress_cuda_ptr Pfp, 
    hipDoubleComplex* m, const hipDoubleComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, clcg_solver_enum solver_id)
{
    cuda_precondtioned_solver_ptr cgp_solver;
    switch (solver_id)
	{
		case CLCG_PCG:
			cgp_solver = clpcg; break;
		default:
			return CLCG_UNKNOWN_SOLVER;
	}

	return cgp_solver(Afp, Mfp, Pfp, m, B, n_size, nz_size, param, instance, cub_handle, cus_handle);
}

int clbicg(clcg_axfunc_cuda_ptr Afp, clcg_progress_cuda_ptr Pfp, hipDoubleComplex* m, 
    const hipDoubleComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle)
{
    // set CGS parameters
	clcg_para para = (param != nullptr) ? (*param) : defparam2;

	//check parameters
	if (n_size <= 0) return CLCG_INVILAD_VARIABLE_SIZE;
	if (para.max_iterations < 0) return CLCG_INVILAD_MAX_ITERATIONS;
	if (para.epsilon <= 0.0 || para.epsilon >= 1.0) return CLCG_INVILAD_EPSILON;

	if (m == nullptr) return CLCG_INVALID_POINTER;
	if (B == nullptr) return CLCG_INVALID_POINTER;
    if (cub_handle == nullptr) return LCG_INVALID_POINTER;
    if (cus_handle == nullptr) return LCG_INVALID_POINTER;

	hipDoubleComplex *r1k = nullptr, *r2k = nullptr;
	hipDoubleComplex *d1k = nullptr, *d2k = nullptr, *Ax = nullptr;
    hipMalloc(&r1k, n_size * sizeof(hipDoubleComplex));
	hipMalloc(&r2k, n_size * sizeof(hipDoubleComplex));
    hipMalloc(&d1k, n_size * sizeof(hipDoubleComplex));
	hipMalloc(&d2k, n_size * sizeof(hipDoubleComplex));
    hipMalloc(&Ax, n_size * sizeof(hipDoubleComplex));

    hipsparseDnVecDescr_t dvec_m, dvec_d1k, dvec_d2k, dvec_Ax;
	hipsparseCreateDnVec(&dvec_m, n_size, m, HIP_C_64F);
	hipsparseCreateDnVec(&dvec_d1k, n_size, d1k, HIP_C_64F);
	hipsparseCreateDnVec(&dvec_d2k, n_size, d2k, HIP_C_64F);
	hipsparseCreateDnVec(&dvec_Ax, n_size, Ax, HIP_C_64F);

    hipDoubleComplex one, none;
    one.x = 1.0; one.y = 0.0;
    none.x = -1.0; none.y = 0.0;
	hipDoubleComplex ak, nak, conj_ak, Ad1d2, r1r2_next, betak, conj_betak;

	Afp(instance, cub_handle, cus_handle, dvec_m, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);

    // r0 = B - Ax
    hipMemcpy(r1k, B, n_size * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice); // r0 = B
    hipblasZaxpy(cub_handle, n_size, &none, Ax, 1, r1k, 1); // r0 -= Ax
    hipMemcpy(d1k, r1k, n_size * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice); // d0 = r0

	vecZ_conjugate(r1k, r2k, n_size);
	hipMemcpy(d2k, r2k, n_size * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);

	hipDoubleComplex r1r2;
    hipblasZdotc(cub_handle, n_size, r2k, 1, r1k, 1, &r1r2);

	lcg_float m_mod;
    if (!para.abs_diff)
    {
        hipblasDznrm2(cub_handle, n_size, m, 1, &m_mod);
        if (m_mod < 1.0) m_mod = 1.0;
    }

	lcg_float rk_mod;
	hipblasDznrm2(cub_handle, n_size, r1k, 1, &rk_mod);

	int ret, t = 0;
	if (para.abs_diff && rk_mod/n_size <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, m, rk_mod/n_size, &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}	
	else if (rk_mod*rk_mod/(m_mod*m_mod) <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, m, rk_mod*rk_mod/(m_mod*m_mod), &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}

	lcg_float residual;
	while(1)
	{
		if (para.abs_diff) residual = rk_mod/n_size;
		else residual = rk_mod*rk_mod/(m_mod*m_mod);

		if (Pfp != nullptr)
		{
			if (Pfp(instance, m, residual, &para, n_size, nz_size, t))
			{
				ret = CLCG_STOP; goto func_ends;
			}
		}

		if (residual <= para.epsilon)
		{
			ret = CLCG_CONVERGENCE; goto func_ends;
		}

		if (para.max_iterations > 0 && t+1 > para.max_iterations)
		{
			ret = LCG_REACHED_MAX_ITERATIONS;
			break;
		}
		
		t++;

        Afp(instance, cub_handle, cus_handle, dvec_d1k, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);
        
        hipblasZdotc(cub_handle, n_size, d2k, 1, Ax, 1, &Ad1d2);
        ak = hipCdiv(r1r2, Ad1d2);
        nak = hipCmul(none, ak);
		conj_ak = hipConj(nak);

        hipblasZaxpy(cub_handle, n_size, &ak, d1k, 1, m, 1);
        hipblasZaxpy(cub_handle, n_size, &nak, Ax, 1, r1k, 1);

        if (!para.abs_diff)
        {
            hipblasDznrm2(cub_handle, n_size, m, 1, &m_mod);
            if (m_mod < 1.0) m_mod = 1.0;
        }

        hipblasDznrm2(cub_handle, n_size, r1k, 1, &rk_mod);

		Afp(instance, cub_handle, cus_handle, dvec_d2k, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE);

		hipblasZaxpy(cub_handle, n_size, &conj_ak, Ax, 1, r2k, 1);

		hipblasZdotc(cub_handle, n_size, r2k, 1, r1k, 1, &r1r2_next);
		betak = hipCdiv(r1r2_next, r1r2);
		conj_betak = hipConj(betak);
		r1r2 = r1r2_next;

        hipblasZscal(cub_handle, n_size, &betak, d1k, 1);
        hipblasZaxpy(cub_handle, n_size, &one, r1k, 1, d1k, 1);

		hipblasZscal(cub_handle, n_size, &conj_betak, d2k, 1);
        hipblasZaxpy(cub_handle, n_size, &one, r2k, 1, d2k, 1);
	}

	func_ends:
	{
		hipFree(r1k);
		hipFree(r2k);
		hipFree(d1k);
		hipFree(d2k);	
		hipFree(Ax);
        hipsparseDestroyDnVec(dvec_m);
        hipsparseDestroyDnVec(dvec_d1k);
		hipsparseDestroyDnVec(dvec_d2k);
        hipsparseDestroyDnVec(dvec_Ax);
	}

	return ret;
}

int clbicg_symmetric(clcg_axfunc_cuda_ptr Afp, clcg_progress_cuda_ptr Pfp, hipDoubleComplex* m, 
    const hipDoubleComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle)
{
    // set CGS parameters
	clcg_para para = (param != nullptr) ? (*param) : defparam2;

	//check parameters
	if (n_size <= 0) return CLCG_INVILAD_VARIABLE_SIZE;
	if (para.max_iterations < 0) return CLCG_INVILAD_MAX_ITERATIONS;
	if (para.epsilon <= 0.0 || para.epsilon >= 1.0) return CLCG_INVILAD_EPSILON;

	if (m == nullptr) return CLCG_INVALID_POINTER;
	if (B == nullptr) return CLCG_INVALID_POINTER;
    if (cub_handle == nullptr) return LCG_INVALID_POINTER;
    if (cus_handle == nullptr) return LCG_INVALID_POINTER;

	hipDoubleComplex *rk = nullptr, *dk = nullptr, *Ax = nullptr;
    hipMalloc(&rk, n_size * sizeof(hipDoubleComplex));
    hipMalloc(&dk, n_size * sizeof(hipDoubleComplex));
    hipMalloc(&Ax, n_size * sizeof(hipDoubleComplex));

    hipsparseDnVecDescr_t dvec_m, dvec_dk, dvec_Ax;
	hipsparseCreateDnVec(&dvec_m, n_size, m, HIP_C_64F);
	hipsparseCreateDnVec(&dvec_dk, n_size, dk, HIP_C_64F);
	hipsparseCreateDnVec(&dvec_Ax, n_size, Ax, HIP_C_64F);

    hipDoubleComplex one, none;
    one.x = 1.0; one.y = 0.0;
    none.x = -1.0; none.y = 0.0;
	hipDoubleComplex ak, nak, rkrk2, betak, dkAx;

	Afp(instance, cub_handle, cus_handle, dvec_m, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);

    // r0 = B - Ax
    hipMemcpy(rk, B, n_size * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice); // r0 = B
    hipblasZaxpy(cub_handle, n_size, &none, Ax, 1, rk, 1); // r0 -= Ax
    hipMemcpy(dk, rk, n_size * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice); // d0 = r0

	hipDoubleComplex rkrk;
    hipblasZdotu(cub_handle, n_size, rk, 1, rk, 1, &rkrk);

	lcg_float m_mod;
    if (!para.abs_diff)
    {
        hipblasDznrm2(cub_handle, n_size, m, 1, &m_mod);
        if (m_mod < 1.0) m_mod = 1.0;
    }

	lcg_float rk_mod;
	hipblasDznrm2(cub_handle, n_size, rk, 1, &rk_mod);

	int ret, t = 0;
	if (para.abs_diff && rk_mod/n_size <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, m, rk_mod/n_size, &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}	
	else if (rk_mod*rk_mod/(m_mod*m_mod) <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, m, rk_mod*rk_mod/(m_mod*m_mod), &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}

	lcg_float residual;
	while(1)
	{
		if (para.abs_diff) residual = rk_mod/n_size;
		else residual = rk_mod*rk_mod/(m_mod*m_mod);

		if (Pfp != nullptr)
		{
			if (Pfp(instance, m, residual, &para, n_size, nz_size, t))
			{
				ret = CLCG_STOP; goto func_ends;
			}
		}

		if (residual <= para.epsilon)
		{
			ret = CLCG_CONVERGENCE; goto func_ends;
		}

		if (para.max_iterations > 0 && t+1 > para.max_iterations)
		{
			ret = LCG_REACHED_MAX_ITERATIONS;
			break;
		}
		
		t++;

        Afp(instance, cub_handle, cus_handle, dvec_dk, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);
        
        hipblasZdotu(cub_handle, n_size, dk, 1, Ax, 1, &dkAx);
        ak = hipCdiv(rkrk, dkAx);
        nak = hipCmul(none, ak);

        hipblasZaxpy(cub_handle, n_size, &ak, dk, 1, m, 1);
        hipblasZaxpy(cub_handle, n_size, &nak, Ax, 1, rk, 1);

        if (!para.abs_diff)
        {
            hipblasDznrm2(cub_handle, n_size, m, 1, &m_mod);
            if (m_mod < 1.0) m_mod = 1.0;
        }

        hipblasDznrm2(cub_handle, n_size, rk, 1, &rk_mod);

		hipblasZdotu(cub_handle, n_size, rk, 1, rk, 1, &rkrk2);
		betak = hipCdiv(rkrk2, rkrk);
		rkrk = rkrk2;

        hipblasZscal(cub_handle, n_size, &betak, dk, 1);
        hipblasZaxpy(cub_handle, n_size, &one, rk, 1, dk, 1);
	}

	func_ends:
	{
		hipFree(rk);
		hipFree(dk);
		hipFree(Ax);
        hipsparseDestroyDnVec(dvec_m);
        hipsparseDestroyDnVec(dvec_dk);
        hipsparseDestroyDnVec(dvec_Ax);
	}

	return ret;
}

int clpcg(clcg_axfunc_cuda_ptr Afp, clcg_axfunc_cuda_ptr Mfp, clcg_progress_cuda_ptr Pfp, hipDoubleComplex* m, 
    const hipDoubleComplex* B, const int n_size, const int nz_size, const clcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle)
{
    // set CGS parameters
	clcg_para para = (param != nullptr) ? (*param) : defparam2;

	//check parameters
	if (n_size <= 0) return CLCG_INVILAD_VARIABLE_SIZE;
	if (para.max_iterations < 0) return CLCG_INVILAD_MAX_ITERATIONS;
	if (para.epsilon <= 0.0 || para.epsilon >= 1.0) return CLCG_INVILAD_EPSILON;

	if (m == nullptr) return CLCG_INVALID_POINTER;
	if (B == nullptr) return CLCG_INVALID_POINTER;
    if (cub_handle == nullptr) return LCG_INVALID_POINTER;
    if (cus_handle == nullptr) return LCG_INVALID_POINTER;

    hipDoubleComplex *rk = nullptr, *dk = nullptr, *sk = nullptr, *Ax = nullptr;
    hipMalloc(&rk, n_size * sizeof(hipDoubleComplex));
    hipMalloc(&dk, n_size * sizeof(hipDoubleComplex));
    hipMalloc(&sk, n_size * sizeof(hipDoubleComplex));
    hipMalloc(&Ax, n_size * sizeof(hipDoubleComplex));

    hipsparseDnVecDescr_t dvec_m, dvec_rk, dvec_dk, dvec_sk, dvec_Ax;
	hipsparseCreateDnVec(&dvec_m, n_size, m, HIP_C_64F);
    hipsparseCreateDnVec(&dvec_rk, n_size, rk, HIP_C_64F);
	hipsparseCreateDnVec(&dvec_dk, n_size, dk, HIP_C_64F);
    hipsparseCreateDnVec(&dvec_sk, n_size, sk, HIP_C_64F);
	hipsparseCreateDnVec(&dvec_Ax, n_size, Ax, HIP_C_64F);

    hipDoubleComplex one, none;
    one.x = 1.0; one.y = 0.0;
    none.x = -1.0; none.y = 0.0;
    hipDoubleComplex ak, nak, d_old, betak, dkAx;

    Afp(instance, cub_handle, cus_handle, dvec_m, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);

    // r0 = B - Ax
    hipMemcpy(rk, B, n_size * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice); // r0 = B
    hipblasZaxpy(cub_handle, n_size, &none, Ax, 1, rk, 1); // r0 -= Ax

	Mfp(instance, cub_handle, cus_handle, dvec_rk, dvec_dk, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);

	hipDoubleComplex d_new;
    hipblasZdotu(cub_handle, n_size, rk, 1, dk, 1, &d_new);

    lcg_float m_mod;
    if (!para.abs_diff)
    {
        hipblasDznrm2(cub_handle, n_size, m, 1, &m_mod);
        if (m_mod < 1.0) m_mod = 1.0;
    }

	lcg_float rk_mod;
	hipblasDznrm2(cub_handle, n_size, rk, 1, &rk_mod);

    int ret, t = 0;
	if (para.abs_diff && rk_mod/n_size <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, m, rk_mod/n_size, &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}	
	else if (rk_mod*rk_mod/(m_mod*m_mod) <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, m, rk_mod*rk_mod/(m_mod*m_mod), &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}

	lcg_float residual;
	while(1)
	{
		if (para.abs_diff) residual = rk_mod/n_size;
		else residual = rk_mod*rk_mod/(m_mod*m_mod);

		if (Pfp != nullptr)
		{
			if (Pfp(instance, m, residual, &para, n_size, nz_size, t))
			{
				ret = CLCG_STOP; goto func_ends;
			}
		}

		if (residual <= para.epsilon)
		{
			ret = CLCG_CONVERGENCE; goto func_ends;
		}

		if (para.max_iterations > 0 && t+1 > para.max_iterations)
		{
			ret = LCG_REACHED_MAX_ITERATIONS;
			break;
		}
		
		t++;

        Afp(instance, cub_handle, cus_handle, dvec_dk, dvec_Ax, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);
        hipblasZdotu(cub_handle, n_size, dk, 1, Ax, 1, &dkAx);
		ak = hipCdiv(d_new, dkAx);
        nak = hipCmul(none, ak);

        hipblasZaxpy(cub_handle, n_size, &ak, dk, 1, m, 1);
        hipblasZaxpy(cub_handle, n_size, &nak, Ax, 1, rk, 1);

		if (!para.abs_diff)
        {
            hipblasDznrm2(cub_handle, n_size, m, 1, &m_mod);
            if (m_mod < 1.0) m_mod = 1.0;
        }

        hipblasDznrm2(cub_handle, n_size, rk, 1, &rk_mod);

        Mfp(instance, cub_handle, cus_handle, dvec_rk, dvec_sk, n_size, nz_size, HIPSPARSE_OPERATION_NON_TRANSPOSE);

		d_old = d_new;
        hipblasZdotu(cub_handle, n_size, rk, 1, sk, 1, &d_new);

		betak = hipCdiv(d_new, d_old);

        hipblasZscal(cub_handle, n_size, &betak, dk, 1);
        hipblasZaxpy(cub_handle, n_size, &one, sk, 1, dk, 1);
	}

	func_ends:
	{
		hipFree(rk);
		hipFree(dk);
		hipFree(sk);
		hipFree(Ax);
        hipsparseDestroyDnVec(dvec_m);
        hipsparseDestroyDnVec(dvec_rk);
        hipsparseDestroyDnVec(dvec_dk);
        hipsparseDestroyDnVec(dvec_sk);
        hipsparseDestroyDnVec(dvec_Ax);
	}

	return ret;
}


CLCG_CUDA_Solver::CLCG_CUDA_Solver()
{
	param_ = clcg_default_parameters();
	inter_ = 1;
	silent_ = false;
}

int CLCG_CUDA_Solver::Progress(const hipDoubleComplex* m, const lcg_float converge, 
	const clcg_para* param, const int n_size, const int nz_size, const int k)
{
	if (inter_ > 0 && k%inter_ == 0)
	{
		std::clog << "\rIteration-times: " << k << "\tconvergence: " << converge;
		return 0;
	}

	if (converge <= param->epsilon)
	{
		std::clog << "\rIteration-times: " << k << "\tconvergence: " << converge;
	}
	return 0;
}

void CLCG_CUDA_Solver::silent()
{
	silent_ = true;
	return;
}

void CLCG_CUDA_Solver::set_report_interval(unsigned int inter)
{
	inter_ = inter;
	return;
}

void CLCG_CUDA_Solver::set_clcg_parameter(const clcg_para &in_param)
{
	param_ = in_param;
	return;
}

void CLCG_CUDA_Solver::Minimize(hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, hipDoubleComplex *x, hipDoubleComplex *b, 
	const int n_size, const int nz_size, clcg_solver_enum solver_id, bool verbose, bool er_throw)
{
	if (silent_)
	{
		int ret = clcg_solver_cuda(_AxProduct, nullptr, x, b, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
		if (ret < 0) lcg_error_str(ret, true);
		return;
	}
	
	// 使用lcg求解 注意当我们使用函数指针来调用求解函数时默认参数不可以省略
	clock_t start = clock();
	int ret = clcg_solver_cuda(_AxProduct, _Progress, x, b, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
	clock_t end = clock();

	lcg_float costime = 1000*(end-start)/(double)CLOCKS_PER_SEC;
	
	if (!er_throw)
	{
		std::clog << std::endl;
		switch (solver_id)
		{
			case CLCG_BICG:
				std::clog << "Solver: BI-CG. Time cost: " << costime << " ms" << std::endl;
				break;
			case CLCG_BICG_SYM:
				std::clog << "Solver: BI-CG (symmetrically accelerated). Time cost: " << costime << " ms" << std::endl;
				break;
			default:
				std::clog << "Solver: Unknown. Time cost: " << costime << " ms" << std::endl;
				break;
		}	
	}

	if (verbose) lcg_error_str(ret, er_throw);
	else if (ret < 0) lcg_error_str(ret, er_throw);
	return;
}

void CLCG_CUDA_Solver::MinimizePreconditioned(hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, hipDoubleComplex *x, hipDoubleComplex *b, 
	const int n_size, const int nz_size, clcg_solver_enum solver_id, bool verbose, bool er_throw)
{
	if (silent_)
	{
		int ret = clcg_solver_preconditioned_cuda(_AxProduct, _MxProduct, nullptr, x, b, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
		if (ret < 0) lcg_error_str(ret, true);
		return;
	}
	
	// 使用lcg求解 注意当我们使用函数指针来调用求解函数时默认参数不可以省略
	clock_t start = clock();
	int ret = clcg_solver_preconditioned_cuda(_AxProduct, _MxProduct, _Progress, x, b, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
	clock_t end = clock();

	lcg_float costime = 1000*(end-start)/(double)CLOCKS_PER_SEC;
	
	if (!er_throw)
	{
		std::clog << std::endl;
		switch (solver_id)
		{
			case CLCG_PCG:
				std::clog << "Solver: PCG. Time cost: " << costime << " ms" << std::endl;
				break;
			default:
				std::clog << "Solver: Unknown. Time cost: " << costime << " ms" << std::endl;
				break;
		}	
	}

	if (verbose) lcg_error_str(ret, er_throw);
	else if (ret < 0) lcg_error_str(ret, er_throw);
	return;
}