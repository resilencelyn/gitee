#include "hip/hip_runtime.h"
#include "cmath"
#include "ctime"
#include "iostream"

#include "lcg_cuda.h"


typedef int (*lcg_solver_cuda_ptr)(lcg_axfunc_cuda_ptr Afp, lcg_progress_cuda_ptr Pfp, lcg_float* m, const lcg_float* B, 
    const int n_size, const int nz_size, const lcg_para* param, void* instance, hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

int lcg(lcg_axfunc_cuda_ptr Afp, lcg_progress_cuda_ptr Pfp, lcg_float* m, const lcg_float* B, const int n_size, const int nz_size, 
    const lcg_para* param, void* instance, hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

int lcgs(lcg_axfunc_cuda_ptr Afp, lcg_progress_cuda_ptr Pfp, lcg_float* m, const lcg_float* B, const int n_size, const int nz_size, 
    const lcg_para* param, void* instance, hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);


int lcg_solver_cuda(lcg_axfunc_cuda_ptr Afp, lcg_progress_cuda_ptr Pfp, lcg_float* m, const lcg_float* B, const int n_size, const int nz_size, 
    const lcg_para* param, void* instance, hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, lcg_solver_enum solver_id)
{
	lcg_solver_cuda_ptr cg_solver_cuda;
	switch (solver_id)
	{
		case LCG_CG:
			cg_solver_cuda = lcg;
			break;
		case LCG_CGS:
			cg_solver_cuda = lcgs;
			break;
		default:
			cg_solver_cuda = lcg;
            break;
	}

	return cg_solver_cuda(Afp, Pfp, m, B, n_size, nz_size, param, instance, cub_handle, cus_handle);
}


int lpcg(lcg_axfunc_cuda_ptr Afp, lcg_axfunc_cuda_ptr Mfp, lcg_progress_cuda_ptr Pfp, lcg_float* m, const lcg_float* B, 
	const int n_size, const int nz_size, const lcg_para* param, void* instance, hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

int lcg_solver_preconditioned_cuda(lcg_axfunc_cuda_ptr Afp, lcg_axfunc_cuda_ptr Mfp, lcg_progress_cuda_ptr Pfp, 
    lcg_float* m, const lcg_float* B, const int n_size, const int nz_size, const lcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, lcg_solver_enum solver_id)
{
	return lpcg(Afp, Mfp, Pfp, m, B, n_size, nz_size, param, instance, cub_handle, cus_handle);
}


int lpg(lcg_axfunc_cuda_ptr Afp, lcg_progress_cuda_ptr Pfp, lcg_float* m, const lcg_float* B, 
	const lcg_float* low, const lcg_float* hig, const int n_size, const int nz_size, const lcg_para* param, 
	void* instance, hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

int lcg_solver_constrained_cuda(lcg_axfunc_cuda_ptr Afp, lcg_progress_cuda_ptr Pfp, lcg_float* m, const lcg_float* B, 
    const lcg_float* low, const lcg_float* hig, const int n_size, const int nz_size, const lcg_para* param, void* instance, 
    hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, lcg_solver_enum solver_id)
{
	return lpg(Afp, Pfp, m, B, low, hig, n_size, nz_size, param, instance, cub_handle, cus_handle);
}


int lcg(lcg_axfunc_cuda_ptr Afp, lcg_progress_cuda_ptr Pfp, lcg_float* m, const lcg_float* B, const int n_size, 
    const int nz_size, const lcg_para* param, void* instance, hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle)
{
	// set CG parameters
	lcg_para para = (param != nullptr) ? (*param) : defparam;

	//check parameters
	if (n_size <= 0) return LCG_INVILAD_VARIABLE_SIZE;
	if (para.max_iterations < 0) return LCG_INVILAD_MAX_ITERATIONS;
	if (para.epsilon <= 0.0 || para.epsilon >= 1.0) return LCG_INVILAD_EPSILON;

	if (m == nullptr) return LCG_INVALID_POINTER;
	if (B == nullptr) return LCG_INVALID_POINTER;
    if (cub_handle == nullptr) return LCG_INVALID_POINTER;
    if (cus_handle == nullptr) return LCG_INVALID_POINTER;

	// locate memory
	lcg_float *gk = nullptr, *dk = nullptr, *Adk = nullptr;
	hipMalloc(&gk, n_size * sizeof(lcg_float));
    hipMalloc(&dk, n_size * sizeof(lcg_float));
    hipMalloc(&Adk, n_size * sizeof(lcg_float));

	hipsparseDnVecDescr_t dvec_m, dvec_dk, dvec_Adk;
	hipsparseCreateDnVec(&dvec_m, n_size, m, HIP_R_64F);
	hipsparseCreateDnVec(&dvec_dk, n_size, dk, HIP_R_64F);
	hipsparseCreateDnVec(&dvec_Adk, n_size, Adk, HIP_R_64F);

    lcg_float none = -1.0;

	Afp(instance, cub_handle, cus_handle, dvec_m, dvec_Adk, n_size, nz_size);

    // g0 = Ax - B
    hipMemcpy(gk, Adk, n_size * sizeof(lcg_float), hipMemcpyDeviceToDevice); // g0 = A*x
    hipblasDaxpy(cub_handle, n_size, &none, B, 1, gk, 1); // g0 -= B
    hipMemset(dk, 0, n_size * sizeof(lcg_float)); // d0 = 0
    hipblasDaxpy(cub_handle, n_size, &none, gk, 1, dk, 1); // d0 = -g0

	lcg_float m_mod;
	if (!para.abs_diff)
	{
		hipblasDdot(cub_handle, n_size, m, 1, m, 1, &m_mod); // m_mod = ||m||
		if (m_mod < 1.0) m_mod = 1.0;
	}

	lcg_float gk_mod;
    hipblasDdot(cub_handle, n_size, gk, 1, gk, 1, &gk_mod); // gk_mod = ||gk||

	int ret, t = 0;
	if (para.abs_diff && sqrt(gk_mod)/n_size <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, m, sqrt(gk_mod)/n_size, &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}
	else if (gk_mod/m_mod <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, m, gk_mod/m_mod, &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}

	lcg_float dTAd, ak, betak, gk1_mod, residual;
	while (1)
	{
		if (para.abs_diff) residual = sqrt(gk_mod)/n_size;
		else residual = gk_mod/m_mod;

		if (Pfp != nullptr)
		{
			if (Pfp(instance, m, residual, &para, n_size, nz_size, t))
			{
				ret = LCG_STOP; goto func_ends;
			}
		}

		if (residual <= para.epsilon)
		{
			ret = LCG_CONVERGENCE; goto func_ends;
		}

		if (para.max_iterations > 0 && t+1 > para.max_iterations)
		{
			ret = LCG_REACHED_MAX_ITERATIONS;
			break;
		}
		
		t++;

        Afp(instance, cub_handle, cus_handle, dvec_dk, dvec_Adk, n_size, nz_size);

        hipblasDdot(cub_handle, n_size, dk, 1, Adk, 1, &dTAd); // dTAd = dk^T * Adk
		ak = gk_mod/dTAd;

        hipblasDaxpy(cub_handle, n_size, &ak, dk, 1, m, 1); // m += ak*dk
        hipblasDaxpy(cub_handle, n_size, &ak, Adk, 1, gk, 1); // gk += ak*Adk

		if (!para.abs_diff)
		{
			hipblasDdot(cub_handle, n_size, m, 1, m, 1, &m_mod); // m_mod = ||m||
			if (m_mod < 1.0) m_mod = 1.0;
		}

        hipblasDdot(cub_handle, n_size, gk, 1, gk, 1, &gk1_mod); // gk1_mod = ||gk||
		betak = gk1_mod/gk_mod;
		gk_mod = gk1_mod;

        hipblasDscal(cub_handle, n_size, &betak, dk, 1); // dk *= betak
        hipblasDaxpy(cub_handle, n_size, &none, gk, 1, dk, 1); // dk -= gk
	}

	func_ends:
	{
        hipFree(dk);
        hipFree(gk);
        hipFree(Adk);
		hipsparseDestroyDnVec(dvec_m);
		hipsparseDestroyDnVec(dvec_dk);
		hipsparseDestroyDnVec(dvec_Adk);
	}

	return ret;
}

int lcgs(lcg_axfunc_cuda_ptr Afp, lcg_progress_cuda_ptr Pfp, lcg_float* m, const lcg_float* B, const int n_size, 
    const int nz_size, const lcg_para* param, void* instance, hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle)
{
	// set CG parameters
	lcg_para para = (param != nullptr) ? (*param) : defparam;

	//check parameters
	if (n_size <= 0) return LCG_INVILAD_VARIABLE_SIZE;
	if (para.max_iterations < 0) return LCG_INVILAD_MAX_ITERATIONS;
	if (para.epsilon <= 0.0 || para.epsilon >= 1.0) return LCG_INVILAD_EPSILON;

	if (m == nullptr) return LCG_INVALID_POINTER;
	if (B == nullptr) return LCG_INVALID_POINTER;
    if (cub_handle == nullptr) return LCG_INVALID_POINTER;
    if (cus_handle == nullptr) return LCG_INVALID_POINTER;

	// locate memory
	lcg_float *rk = nullptr, *r0T = nullptr, *pk = nullptr, *qpk = nullptr;
	lcg_float *Ax = nullptr, *uk = nullptr,   *qk = nullptr, *wk = nullptr;
	hipMalloc(&rk, n_size * sizeof(lcg_float));
    hipMalloc(&r0T, n_size * sizeof(lcg_float));
    hipMalloc(&pk, n_size * sizeof(lcg_float));
	hipMalloc(&qpk, n_size * sizeof(lcg_float));
	hipMalloc(&Ax, n_size * sizeof(lcg_float));
	hipMalloc(&uk, n_size * sizeof(lcg_float));
	hipMalloc(&qk, n_size * sizeof(lcg_float));
	hipMalloc(&wk, n_size * sizeof(lcg_float));

	hipsparseDnVecDescr_t dvec_m, dvec_wk, dvec_pk, dvec_Ax;
	hipsparseCreateDnVec(&dvec_m, n_size, m, HIP_R_64F);
	hipsparseCreateDnVec(&dvec_wk, n_size, wk, HIP_R_64F);
	hipsparseCreateDnVec(&dvec_pk, n_size, pk, HIP_R_64F);
	hipsparseCreateDnVec(&dvec_Ax, n_size, Ax, HIP_R_64F);

	lcg_float one = 1.0;
    lcg_float none = -1.0;

	Afp(instance, cub_handle, cus_handle, dvec_m, dvec_Ax, n_size, nz_size);

    // r0 = B - Ax
	hipMemcpy(rk, B, n_size * sizeof(lcg_float), hipMemcpyDeviceToDevice); // r0 = B
	hipblasDaxpy(cub_handle, n_size, &none, Ax, 1, rk, 1); // r0 -= Ax
    // p0 = u0 = r0T = r0
	hipMemcpy(pk, rk, n_size * sizeof(lcg_float), hipMemcpyDeviceToDevice);
	hipMemcpy(uk, rk, n_size * sizeof(lcg_float), hipMemcpyDeviceToDevice);
	hipMemcpy(r0T, rk, n_size * sizeof(lcg_float), hipMemcpyDeviceToDevice);

	lcg_float rkr0T;
	hipblasDdot(cub_handle, n_size, rk, 1, r0T, 1, &rkr0T);

	lcg_float m_mod;
    if (!para.abs_diff) 
	{
		hipblasDdot(cub_handle, n_size, m, 1, m, 1, &m_mod); // m_mod = ||m||
		if (m_mod < 1.0) m_mod = 1.0;
	}

	lcg_float rk_mod;
    hipblasDdot(cub_handle, n_size, rk, 1, rk, 1, &rk_mod); // rk_mod = ||rk||

	int ret, t = 0;
	if (para.abs_diff && sqrt(rk_mod)/n_size <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, m, sqrt(rk_mod)/n_size, &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}
	else if (rk_mod/m_mod <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, m, rk_mod/m_mod, &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}

	lcg_float ak, nak, rkr0T1, AprT, betak, residual;
	while (1)
	{
		if (para.abs_diff) residual = sqrt(rk_mod)/n_size;
		else residual = rk_mod/m_mod;

		if (Pfp != nullptr)
		{
			if (Pfp(instance, m, residual, &para, n_size, nz_size, t))
			{
				ret = LCG_STOP; goto func_ends;
			}
		}

		if (residual <= para.epsilon)
		{
			ret = LCG_CONVERGENCE; goto func_ends;
		}

		if (para.max_iterations > 0 && t+1 > para.max_iterations)
		{
			ret = LCG_REACHED_MAX_ITERATIONS;
			break;
		}
		
		t++;

        Afp(instance, cub_handle, cus_handle, dvec_pk, dvec_Ax, n_size, nz_size);

		AprT = 0.0;
		hipblasDdot(cub_handle, n_size, r0T, 1, Ax, 1, &AprT);
		ak = rkr0T/AprT;
		nak = -1.0*ak;

		hipMemcpy(qk, uk, n_size * sizeof(lcg_float), hipMemcpyDeviceToDevice);
		hipMemcpy(wk, uk, n_size * sizeof(lcg_float), hipMemcpyDeviceToDevice);
        hipblasDaxpy(cub_handle, n_size, &nak, Ax, 1, qk, 1);
        hipblasDaxpy(cub_handle, n_size, &one, qk, 1, wk, 1);

		Afp(instance, cub_handle, cus_handle, dvec_wk, dvec_Ax, n_size, nz_size);

		hipblasDaxpy(cub_handle, n_size, &ak, wk, 1, m, 1);
        hipblasDaxpy(cub_handle, n_size, &nak, Ax, 1, rk, 1);

		if (!para.abs_diff)
		{
			hipblasDdot(cub_handle, n_size, m, 1, m, 1, &m_mod);
			if (m_mod < 1.0) m_mod = 1.0;
		}

        hipblasDdot(cub_handle, n_size, rk, 1, rk, 1, &rk_mod);
		
		hipblasDdot(cub_handle, n_size, rk, 1, r0T, 1, &rkr0T1);
		betak = rkr0T1/rkr0T;
		rkr0T = rkr0T1;

		hipMemcpy(uk, rk, n_size * sizeof(lcg_float), hipMemcpyDeviceToDevice);
		hipblasDaxpy(cub_handle, n_size, &betak, qk, 1, uk, 1);

		hipMemcpy(qpk, qk, n_size * sizeof(lcg_float), hipMemcpyDeviceToDevice);
		hipblasDaxpy(cub_handle, n_size, &betak, pk, 1, qpk, 1);

		hipMemcpy(pk, uk, n_size * sizeof(lcg_float), hipMemcpyDeviceToDevice);
		hipblasDaxpy(cub_handle, n_size, &betak, qpk, 1, pk, 1);
	}

	func_ends:
	{
        hipFree(rk);
		hipFree(r0T);
		hipFree(pk);
		hipFree(qpk);
		hipFree(Ax);
		hipFree(uk);
		hipFree(qk);
		hipFree(wk);
		hipsparseDestroyDnVec(dvec_m);
		hipsparseDestroyDnVec(dvec_wk);
		hipsparseDestroyDnVec(dvec_pk);
		hipsparseDestroyDnVec(dvec_Ax);
	}

	return ret;
}

int lpcg(lcg_axfunc_cuda_ptr Afp, lcg_axfunc_cuda_ptr Mfp, lcg_progress_cuda_ptr Pfp, lcg_float* m, const lcg_float* B, 
	const int n_size, const int nz_size, const lcg_para* param, void* instance, hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle)
{
	// set CG parameters
	lcg_para para = (param != nullptr) ? (*param) : defparam;

	//check parameters
	if (n_size <= 0) return LCG_INVILAD_VARIABLE_SIZE;
	if (para.max_iterations < 0) return LCG_INVILAD_MAX_ITERATIONS;
	if (para.epsilon <= 0.0 || para.epsilon >= 1.0) return LCG_INVILAD_EPSILON;

	if (m == nullptr) return LCG_INVALID_POINTER;
	if (B == nullptr) return LCG_INVALID_POINTER;
    if (cub_handle == nullptr) return LCG_INVALID_POINTER;
    if (cus_handle == nullptr) return LCG_INVALID_POINTER;

	// locate memory
	lcg_float *rk = nullptr, *zk = nullptr, *dk = nullptr, *Adk = nullptr;
	hipMalloc(&rk, n_size * sizeof(lcg_float));
    hipMalloc(&zk, n_size * sizeof(lcg_float));
    hipMalloc(&dk, n_size * sizeof(lcg_float));
	hipMalloc(&Adk, n_size * sizeof(lcg_float));

	hipsparseDnVecDescr_t dvec_m, dvec_rk, dvec_zk, dvec_dk, dvec_Adk;
	hipsparseCreateDnVec(&dvec_m, n_size, m, HIP_R_64F);
	hipsparseCreateDnVec(&dvec_rk, n_size, rk, HIP_R_64F);
	hipsparseCreateDnVec(&dvec_zk, n_size, zk, HIP_R_64F);
	hipsparseCreateDnVec(&dvec_dk, n_size, dk, HIP_R_64F);
	hipsparseCreateDnVec(&dvec_Adk, n_size, Adk, HIP_R_64F);

	lcg_float one = 1.0;
    lcg_float none = -1.0;

	Afp(instance, cub_handle, cus_handle, dvec_m, dvec_Adk, n_size, nz_size);

    // r0 = B - Ax
	hipMemcpy(rk, B, n_size * sizeof(lcg_float), hipMemcpyDeviceToDevice); // r0 = B
	hipblasDaxpy(cub_handle, n_size, &none, Adk, 1, rk, 1); // r0 -= Ax

	Mfp(instance, cub_handle, cus_handle, dvec_rk, dvec_zk, n_size, nz_size);

    // d0 = z0
	hipMemcpy(dk, zk, n_size * sizeof(lcg_float), hipMemcpyDeviceToDevice);

	lcg_float m_mod;
    if (!para.abs_diff) 
	{
		hipblasDdot(cub_handle, n_size, m, 1, m, 1, &m_mod); // m_mod = ||m||
		if (m_mod < 1.0) m_mod = 1.0;
	}

	lcg_float rk_mod;
    hipblasDdot(cub_handle, n_size, rk, 1, rk, 1, &rk_mod); // rk_mod = ||rk||

	lcg_float zTr;
	hipblasDdot(cub_handle, n_size, zk, 1, rk, 1, &zTr);

	int ret, t = 0;
	if (para.abs_diff && sqrt(rk_mod)/n_size <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, m, sqrt(rk_mod)/n_size, &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}
	else if (rk_mod/m_mod <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, m, rk_mod/m_mod, &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}

	lcg_float dTAd, ak, nak, betak, zTr1, residual;
	while (1)
	{
		if (para.abs_diff) residual = sqrt(rk_mod)/n_size;
		else residual = rk_mod/m_mod;

		if (Pfp != nullptr)
		{
			if (Pfp(instance, m, residual, &para, n_size, nz_size, t))
			{
				ret = LCG_STOP; goto func_ends;
			}
		}

		if (residual <= para.epsilon)
		{
			ret = LCG_CONVERGENCE; goto func_ends;
		}

		if (para.max_iterations > 0 && t+1 > para.max_iterations)
		{
			ret = LCG_REACHED_MAX_ITERATIONS;
			break;
		}
		
		t++;

        Afp(instance, cub_handle, cus_handle, dvec_dk, dvec_Adk, n_size, nz_size);

		hipblasDdot(cub_handle, n_size, dk, 1, Adk, 1, &dTAd);
		ak = zTr/dTAd;
		nak = -1.0*ak;

        hipblasDaxpy(cub_handle, n_size, &ak, dk, 1, m, 1);
        hipblasDaxpy(cub_handle, n_size, &nak, Adk, 1, rk, 1);

		Mfp(instance, cub_handle, cus_handle, dvec_rk, dvec_zk, n_size, nz_size);

		if (!para.abs_diff)
		{
			hipblasDdot(cub_handle, n_size, m, 1, m, 1, &m_mod);
			if (m_mod < 1.0) m_mod = 1.0;
		}

        hipblasDdot(cub_handle, n_size, rk, 1, rk, 1, &rk_mod);
		
		hipblasDdot(cub_handle, n_size, zk, 1, rk, 1, &zTr1);
		betak = zTr1/zTr;
		zTr = zTr1;

		hipblasDscal(cub_handle, n_size, &betak, dk, 1); // dk *= betak
		hipblasDaxpy(cub_handle, n_size, &one, zk, 1, dk, 1);
	}

	func_ends:
	{
        hipFree(rk);
		hipFree(zk);
		hipFree(dk);
		hipFree(Adk);
		hipsparseDestroyDnVec(dvec_m);
		hipsparseDestroyDnVec(dvec_rk);
		hipsparseDestroyDnVec(dvec_zk);
		hipsparseDestroyDnVec(dvec_dk);
		hipsparseDestroyDnVec(dvec_Adk);
	}

	return ret;
}


int lpg(lcg_axfunc_cuda_ptr Afp, lcg_progress_cuda_ptr Pfp, lcg_float* m, const lcg_float* B, 
	const lcg_float* low, const lcg_float* hig, const int n_size, const int nz_size, const lcg_para* param, 
	void* instance, hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle)
{
	// set CG parameters
	lcg_para para = (param != nullptr) ? (*param) : defparam;

	// check parameters
	if (n_size <= 0) return LCG_INVILAD_VARIABLE_SIZE;
	if (para.max_iterations < 0) return LCG_INVILAD_MAX_ITERATIONS;
	if (para.epsilon <= 0.0 || para.epsilon >= 1.0) return LCG_INVILAD_EPSILON;
	if (para.lambda <= 0.0) return LCG_INVALID_LAMBDA;

	if (m == nullptr) return LCG_INVALID_POINTER;
	if (B == nullptr) return LCG_INVALID_POINTER;
	if (low == nullptr) return LCG_INVALID_POINTER;
	if (hig == nullptr) return LCG_INVALID_POINTER;
	if (cub_handle == nullptr) return LCG_INVALID_POINTER;
    if (cus_handle == nullptr) return LCG_INVALID_POINTER;

	// locate memory
	lcg_float *gk = nullptr, *Adk = nullptr;
	lcg_float *m_new = nullptr, *gk_new = nullptr;
	lcg_float *sk = nullptr, *yk = nullptr;
	hipMalloc(&gk, n_size *sizeof(lcg_float));
	hipMalloc(&Adk, n_size *sizeof(lcg_float));
	hipMalloc(&m_new, n_size *sizeof(lcg_float));
	hipMalloc(&gk_new, n_size *sizeof(lcg_float));
	hipMalloc(&sk, n_size *sizeof(lcg_float));
	hipMalloc(&yk, n_size *sizeof(lcg_float));

	hipsparseDnVecDescr_t dvec_m, dvec_mnew, dvec_Adk;
	hipsparseCreateDnVec(&dvec_m, n_size, m, HIP_R_64F);
	hipsparseCreateDnVec(&dvec_mnew, n_size, m_new, HIP_R_64F);
	hipsparseCreateDnVec(&dvec_Adk, n_size, Adk, HIP_R_64F);

	lcg_float none = -1.0;
	lcg_float nalpha_k, alpha_k = para.lambda;

	lcg_set2box_cuda(low, hig, m, n_size);
	Afp(instance, cub_handle, cus_handle, dvec_m, dvec_Adk, n_size, nz_size);

	// g0 = Ax - B
    hipMemcpy(gk, Adk, n_size * sizeof(lcg_float), hipMemcpyDeviceToDevice); // g0 = A*x
    hipblasDaxpy(cub_handle, n_size, &none, B, 1, gk, 1); // g0 -= B

	lcg_float m_mod;
	if (!para.abs_diff)
	{
		hipblasDdot(cub_handle, n_size, m, 1, m, 1, &m_mod); // m_mod = ||m||
		if (m_mod < 1.0) m_mod = 1.0;
	}

	lcg_float gk_mod;
    hipblasDdot(cub_handle, n_size, gk, 1, gk, 1, &gk_mod); // gk_mod = ||gk||

	int ret, t = 0;
	if (para.abs_diff && sqrt(gk_mod)/n_size <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, m, sqrt(gk_mod)/n_size, &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}
	else if (gk_mod/m_mod <= para.epsilon)
	{
		ret = LCG_ALREADY_OPTIMIZIED;
		if (Pfp != nullptr)
		{
			Pfp(instance, m, gk_mod/m_mod, &para, n_size, nz_size, 0);
		}
		goto func_ends;
	}


	lcg_float sk_mod, syk_mod, residual;
	while(1)
	{
		if (para.abs_diff) residual = sqrt(gk_mod)/n_size;
		else residual = gk_mod/m_mod;

		if (Pfp != nullptr)
		{
			if (Pfp(instance, m, residual, &para, n_size, nz_size, t))
			{
				ret = LCG_STOP; goto func_ends;
			}
		}

		if (residual <= para.epsilon)
		{
			ret = LCG_CONVERGENCE; goto func_ends;
		}

		if (para.max_iterations > 0 && t+1 > para.max_iterations)
		{
			ret = LCG_REACHED_MAX_ITERATIONS;
			break;
		}
		
		t++;

		nalpha_k = -1.0*alpha_k;
		hipMemcpy(m_new, m, n_size * sizeof(lcg_float), hipMemcpyDeviceToDevice);
		hipblasDaxpy(cub_handle, n_size, &nalpha_k, gk, 1, m_new, 1);

		lcg_set2box_cuda(low, hig, m_new, n_size);
		Afp(instance, cub_handle, cus_handle, dvec_mnew, dvec_Adk, n_size, nz_size);
		
		hipMemcpy(gk_new, Adk, n_size * sizeof(lcg_float), hipMemcpyDeviceToDevice); // g0 = A*x
    	hipblasDaxpy(cub_handle, n_size, &none, B, 1, gk, 1); // g0 -= B

		hipMemcpy(sk, m_new, n_size * sizeof(lcg_float), hipMemcpyDeviceToDevice);
		hipblasDaxpy(cub_handle, n_size, &none, m, 1, sk, 1);

		hipMemcpy(yk, gk_new, n_size * sizeof(lcg_float), hipMemcpyDeviceToDevice);
		hipblasDaxpy(cub_handle, n_size, &none, gk, 1, sk, 1);

		hipblasDdot(cub_handle, n_size, sk, 1, sk, 1, &sk_mod);
		hipblasDdot(cub_handle, n_size, sk, 1, yk, 1, &syk_mod);
		alpha_k = sk_mod/syk_mod;

		hipMemcpy(m, m_new, n_size * sizeof(lcg_float), hipMemcpyDeviceToDevice);
		hipMemcpy(gk, gk_new, n_size * sizeof(lcg_float), hipMemcpyDeviceToDevice);

		if (!para.abs_diff)
		{
			hipblasDdot(cub_handle, n_size, m, 1, m, 1, &m_mod); // m_mod = ||m||
			if (m_mod < 1.0) m_mod = 1.0;
		}

		lcg_float gk_mod;
		hipblasDdot(cub_handle, n_size, gk, 1, gk, 1, &gk_mod); // gk_mod = ||gk||
	}

	func_ends:
	{
		hipFree(gk);
		hipFree(gk_new);
		hipFree(m_new);
		hipFree(sk);
		hipFree(yk);
		hipFree(Adk);
		hipsparseDestroyDnVec(dvec_m);
		hipsparseDestroyDnVec(dvec_mnew);
		hipsparseDestroyDnVec(dvec_Adk);
	}

	return ret;
}


LCG_CUDA_Solver::LCG_CUDA_Solver()
{
	param_ = lcg_default_parameters();
	inter_ = 1;
	silent_ = false;
}

int LCG_CUDA_Solver::Progress(const lcg_float* m, const lcg_float converge, 
	const lcg_para* param, const int n_size, const int nz_size, const int k)
{
	if (inter_ > 0 && k%inter_ == 0)
	{
		std::clog << "\rIteration-times: " << k << "\tconvergence: " << converge;
		return 0;
	}

	if (converge <= param->epsilon)
	{
		std::clog << "\rIteration-times: " << k << "\tconvergence: " << converge;
	}
	return 0;
}

void LCG_CUDA_Solver::silent()
{
	silent_ = true;
	return;
}

void LCG_CUDA_Solver::set_report_interval(unsigned int inter)
{
	inter_ = inter;
	return;
}

void LCG_CUDA_Solver::set_lcg_parameter(const lcg_para &in_param)
{
	param_ = in_param;
	return;
}

void LCG_CUDA_Solver::Minimize(hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, lcg_float *x, lcg_float *b, 
	const int n_size, const int nz_size, lcg_solver_enum solver_id, bool verbose, bool er_throw)
{
	if (silent_)
	{
		int ret = lcg_solver_cuda(_AxProduct, nullptr, x, b, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
		if (ret < 0) lcg_error_str(ret, true);
		return;
	}
	
	// 使用lcg求解 注意当我们使用函数指针来调用求解函数时默认参数不可以省略
	clock_t start = clock();
	int ret = lcg_solver_cuda(_AxProduct, _Progress, x, b, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
	clock_t end = clock();

	lcg_float costime = 1000*(end-start)/(double)CLOCKS_PER_SEC;
	
	if (!er_throw)
	{
		std::clog << std::endl;
		switch (solver_id)
		{
			case LCG_CG:
				std::clog << "Solver: CG. Time cost: " << costime << " ms" << std::endl;
				break;
			case LCG_CGS:
				std::clog << "Solver: CGS. Time cost: " << costime << " ms" << std::endl;
				break;
			default:
				std::clog << "Solver: Unknown. Time cost: " << costime << " ms" << std::endl;
				break;
		}	
	}

	if (verbose) lcg_error_str(ret, er_throw);
	else if (ret < 0) lcg_error_str(ret, er_throw);
	return;
}

void LCG_CUDA_Solver::MinimizePreconditioned(hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, lcg_float *x, lcg_float *b, 
    const int n_size, const int nz_size, lcg_solver_enum solver_id, bool verbose, bool er_throw)
{
	if (silent_)
	{
		int ret = lcg_solver_preconditioned_cuda(_AxProduct, _MxProduct, nullptr, x, b, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
		if (ret < 0) lcg_error_str(ret, true);
		return;
	}
	
	// 使用lcg求解 注意当我们使用函数指针来调用求解函数时默认参数不可以省略
	clock_t start = clock();
	int ret = lcg_solver_preconditioned_cuda(_AxProduct, _MxProduct, _Progress, x, b, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
	clock_t end = clock();

	lcg_float costime = 1000*(end-start)/(double)CLOCKS_PER_SEC;
	
	if (!er_throw)
	{
		std::clog << std::endl;
		switch (solver_id)
		{
			case LCG_PCG:
				std::clog << "Solver: PCG. Time cost: " << costime << " ms" << std::endl;
				break;
			default:
				std::clog << "Solver: Unknown. Time cost: " << costime << " ms" << std::endl;
				break;
		}	
	}

	if (verbose) lcg_error_str(ret, er_throw);
	else if (ret < 0) lcg_error_str(ret, er_throw);
	return;
}

void LCG_CUDA_Solver::MinimizeConstrained(hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, lcg_float *x, const lcg_float *b, 
    const lcg_float* low, const lcg_float *hig, const int n_size, const int nz_size, lcg_solver_enum solver_id, 
    bool verbose, bool er_throw)
{
	if (silent_)
	{
		int ret = lcg_solver_constrained_cuda(_AxProduct, nullptr, x, b, low, hig, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
		if (ret < 0) lcg_error_str(ret, true);
		return;
	}
	
	// 使用lcg求解 注意当我们使用函数指针来调用求解函数时默认参数不可以省略
	clock_t start = clock();
	int ret = lcg_solver_constrained_cuda(_AxProduct, _Progress, x, b, low, hig, n_size, nz_size, &param_, this, cub_handle, cus_handle, solver_id);
	clock_t end = clock();

	lcg_float costime = 1000*(end-start)/(double)CLOCKS_PER_SEC;
	
	if (!er_throw)
	{
		std::clog << std::endl;
		switch (solver_id)
		{
			case LCG_PG:
				std::clog << "Solver: PG. Time cost: " << costime << " ms" << std::endl;
				break;
			default:
				std::clog << "Solver: Unknown. Time cost: " << costime << " ms" << std::endl;
				break;
		}	
	}

	if (verbose) lcg_error_str(ret, er_throw);
	else if (ret < 0) lcg_error_str(ret, er_throw);
	return;
}