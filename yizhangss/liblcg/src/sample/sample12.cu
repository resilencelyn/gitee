#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cmath>

#include "../lib/clcg_cuda.h"
#include "../lib/preconditioner_cuda.h"

// Declare as global variables
hipDoubleComplex one = {1.0, 0.0};
hipDoubleComplex zero = {0.0, 0.0};

void read(std::string filePath, int *pN, int *pnz, hipDoubleComplex **cooVal,
	int **cooRowIdx, int **cooColIdx, hipDoubleComplex **b)
{
	std::ifstream in(filePath, std::ios::binary);

	in.read((char*)pN, sizeof(int));
	in.read((char*)pnz, sizeof(int));

	*cooVal = new hipDoubleComplex[*pnz]{};
	*cooRowIdx = new int[*pnz]{};
	*cooColIdx = new int[*pnz]{};
	*b = new hipDoubleComplex[*pN]{};

	for (int i = 0; i < *pnz; ++i)
	{
		in.read((char*)&(*cooRowIdx)[i], sizeof(int));
		in.read((char*)&(*cooColIdx)[i], sizeof(int));
		in.read((char*)&(*cooVal)[i], sizeof(hipDoubleComplex));
	}

	in.read((char*)(*b), sizeof(hipDoubleComplex)*(*pN));
    return;
}

void readAnswer(std::string filePath, int *pN, hipDoubleComplex **x)
{
	std::ifstream in(filePath, std::ios::binary);

	in.read((char*)pN, sizeof(int));

	*x = new hipDoubleComplex[*pN]{};

	in.read((char*)(*x), sizeof(hipDoubleComplex)*(*pN));
    return;
}

lcg_float avg_error(hipDoubleComplex *a, hipDoubleComplex *b, int n)
{
	lcg_float avg = 0.0;
	hipDoubleComplex tmp;
	for (size_t i = 0; i < n; i++)
	{
		tmp = Zdiff(a[i], b[i]);
		avg += (tmp.x*tmp.x + tmp.y*tmp.y);
	}
	return sqrt(avg)/n;
}

class sample12 : public CLCG_CUDA_Solver
{
public:
	sample12(){}
	virtual ~sample12(){}

	void solve(std::string inputPath, std::string answerPath, hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

	void AxProduct(hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, hipsparseDnVecDescr_t x, hipsparseDnVecDescr_t prod_Ax, 
		const int n_size, const int nz_size, hipsparseOperation_t oper_t)
	{
		// Calculate the product of A*x
		hipsparseSpMV(cus_handle, oper_t, &one, smat_A, x, &zero, prod_Ax, HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT, d_buf);
		return;
	}

	void MxProduct(hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, hipsparseDnVecDescr_t x, hipsparseDnVecDescr_t prod_Ax, 
		const int n_size, const int nz_size, hipsparseOperation_t oper_t)
	{
		hipsparseSpSV_solve(cus_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, smat_IC, x, dvec_p, 
			HIP_C_64F, HIPSPARSE_SPSV_ALG_DEFAULT, descr_L);
		
		hipsparseSpSV_solve(cus_handle, HIPSPARSE_OPERATION_TRANSPOSE, &one, smat_IC, dvec_p, prod_Ax, 
			HIP_C_64F, HIPSPARSE_SPSV_ALG_DEFAULT, descr_LT);
		return;
	}

private:
	int N, nz;
	int *rowIdxA, *colIdxA;
	hipDoubleComplex *A, *b;
	hipDoubleComplex *ans_x;

	int *IC_row, *IC_col;
    hipDoubleComplex *IC_val;

	void *d_buf, *d_buf2;
	hipsparseSpMatDescr_t smat_A;
	hipsparseSpMatDescr_t smat_IC;
	hipsparseSpSVDescr_t descr_L, descr_LT;

	int *d_rowIdxA; // COO
	int *d_rowPtrA; // CSR
	int *d_colIdxA;
	hipDoubleComplex *d_A;
	hipDoubleComplex *d_p;
	hipsparseDnVecDescr_t dvec_p;

	int *d_rowIdxIC; // COO
	int *d_rowPtrIC; // CSR
	int *d_colIdxIC;
	hipDoubleComplex *d_IC;

	hipDoubleComplex *host_m;
	hipDoubleComplex *d_t;
	hipsparseDnVecDescr_t dvec_tmp;
};

void sample12::solve(std::string inputPath, std::string answerPath, hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle)
{
	read(inputPath, &N, &nz, &A, &rowIdxA, &colIdxA, &b);
	readAnswer(answerPath, &N, &ans_x);

	std::clog << "N = " << N << std::endl;
	std::clog << "nz = " << nz << std::endl;

	IC_row = new int [nz];
    IC_col = new int [nz];
    IC_val = new hipDoubleComplex [nz];

    incomplete_Cholesky_cuda_full(rowIdxA, colIdxA, A, N, nz, IC_row, IC_col, IC_val);
/*
	for (size_t i = 0; i < nz; i++)
	{
		if (IC_row[i] >= IC_col[i])
		{
			std::cout << IC_row[i] << " " << IC_col[i] << " (" << IC_val[i].x << "," << IC_val[i].y << ")\n";	
		}
	}
*/
    // Allocate GPU memory & copy matrix/vector to device
	hipMalloc(&d_A, nz * sizeof(hipDoubleComplex));
	hipMalloc(&d_rowIdxA, nz * sizeof(int));
	hipMalloc(&d_rowPtrA, (N + 1) * sizeof(int));
	hipMalloc(&d_colIdxA, nz * sizeof(int));
	hipMalloc(&d_p, N * sizeof(hipDoubleComplex));
    hipsparseCreateDnVec(&dvec_p, N, d_p, HIP_C_64F);

	hipMemcpy(d_A, A, nz * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	hipMemcpy(d_rowIdxA, rowIdxA, nz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_colIdxA, colIdxA, nz * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_IC, nz * sizeof(hipDoubleComplex));
	hipMalloc(&d_rowIdxIC, nz * sizeof(int));
	hipMalloc(&d_rowPtrIC, (N + 1) * sizeof(int));
	hipMalloc(&d_colIdxIC, nz * sizeof(int));

    hipMemcpy(d_IC, IC_val, nz * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	hipMemcpy(d_rowIdxIC, IC_row, nz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_colIdxIC, IC_col, nz * sizeof(int), hipMemcpyHostToDevice);

	// Convert matrix A from COO format to CSR format
	hipsparseXcoo2csr(cus_handle, d_rowIdxA, nz, N, d_rowPtrA, HIPSPARSE_INDEX_BASE_ZERO);

	// Create sparse matrix
	hipsparseCreateCsr(&smat_A, N, N, nz, d_rowPtrA, d_colIdxA, d_A, HIPSPARSE_INDEX_32I,
		HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F);

	// Convert matrix L from COO format to CSR format
    hipsparseXcoo2csr(cus_handle, d_rowIdxIC, nz, N, d_rowPtrIC, HIPSPARSE_INDEX_BASE_ZERO);

	// Create sparse matrix
    hipsparseCreateCsr(&smat_IC, N, N, nz, d_rowPtrIC, d_colIdxIC, d_IC, HIPSPARSE_INDEX_32I,
		HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F);

	// Specify Non-Unit diagonal type.
    //hipsparseDiagType_t diagtype = HIPSPARSE_DIAG_TYPE_NON_UNIT;
	//hipsparseSpMatSetAttribute(smat_IC, HIPSPARSE_SPMAT_DIAG_TYPE, &diagtype, sizeof(diagtype));

    // This is just used to get bufferSize;
	hipMalloc(&d_t, N * sizeof(hipDoubleComplex));
	hipsparseCreateDnVec(&dvec_tmp, N, d_t, HIP_C_64F);

	size_t bufferSize_B;
	hipsparseSpMV_bufferSize(cus_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, smat_A,
		dvec_tmp, &zero, dvec_tmp, HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize_B);

    // --- Start of the preconditioning part ---
    hipsparseSpSV_createDescr(&descr_L);
    hipsparseSpSV_createDescr(&descr_LT);

    size_t bufferSize, bufferSize_L, bufferSize_LT;
	bufferSize = bufferSize_B;

    hipsparseSpSV_bufferSize(cus_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, smat_IC, dvec_p, 
        dvec_tmp, HIP_C_64F, HIPSPARSE_SPSV_ALG_DEFAULT, descr_L, &bufferSize_L);
    hipsparseSpSV_bufferSize(cus_handle, HIPSPARSE_OPERATION_TRANSPOSE, &one, smat_IC, dvec_p, 
        dvec_tmp, HIP_C_64F, HIPSPARSE_SPSV_ALG_DEFAULT, descr_LT, &bufferSize_LT);

    bufferSize = max(max(bufferSize, bufferSize_L), bufferSize_LT);
	hipMalloc(&d_buf, bufferSize);
	hipMalloc(&d_buf2, bufferSize);

	hipsparseSpSV_analysis(cus_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, smat_IC, dvec_tmp, dvec_p, 
		HIP_C_64F, HIPSPARSE_SPSV_ALG_DEFAULT, descr_L, d_buf);

	hipsparseSpSV_analysis(cus_handle, HIPSPARSE_OPERATION_TRANSPOSE, &one, smat_IC, dvec_p, dvec_tmp, 
		HIP_C_64F, HIPSPARSE_SPSV_ALG_DEFAULT, descr_LT, d_buf2);

	// --- End of the preconditioning part ---

	// Declare an initial solution
    clcg_para self_para = clcg_default_parameters();
	self_para.epsilon = 1e-6;
	self_para.abs_diff = 0;

	host_m = new hipDoubleComplex[N];

	// Preconditioning with incomplete-chelosky factorization
	for (size_t i = 0; i < N; i++)
	{
		host_m[i].x = 0.0; host_m[i].y = 0.0;	
	}

	MinimizePreconditioned(cub_handle, cus_handle, host_m, b, N, nz, CLCG_PCG);

	std::clog << "Averaged error (compared with ans_x): " << avg_error(host_m, ans_x, N) << std::endl;

	// Free Host memory
	if (rowIdxA != nullptr) delete[] rowIdxA;
	if (colIdxA != nullptr) delete[] colIdxA;
    if (A != nullptr) delete[] A;
	if (b != nullptr) delete[] b;
	if (ans_x != nullptr) delete[] ans_x;

    if (IC_row != nullptr) delete[] IC_row;
    if (IC_col != nullptr) delete[] IC_col;
    if (IC_val != nullptr) delete[] IC_val;

    if (host_m != nullptr) delete[] host_m;

	hipsparseDestroyDnVec(dvec_tmp);
    hipsparseDestroyDnVec(dvec_p);

	hipFree(d_buf);
	hipFree(d_buf2);
	hipFree(d_rowIdxA);
	hipFree(d_rowPtrA);
	hipFree(d_colIdxA);
    hipFree(d_A);
	hipFree(d_p);
	hipFree(d_t);

    hipFree(d_rowIdxIC);
	hipFree(d_rowPtrIC);
	hipFree(d_colIdxIC);
    hipFree(d_IC);

	hipsparseDestroySpMat(smat_A);
	hipsparseDestroySpMat(smat_IC);
    hipsparseSpSV_destroyDescr(descr_L);
    hipsparseSpSV_destroyDescr(descr_LT);
	return;
}

int main(int argc, char **argv)
{
	std::string inputPath = "data/case_1M_cA";
	std::string answerPath = "data/case_1M_cB";

	hipblasHandle_t cubHandle;
	hipsparseHandle_t cusHandle;

	hipblasCreate(&cubHandle);
	hipsparseCreate(&cusHandle);

	sample12 sp;
	sp.set_report_interval(0);
	sp.solve(inputPath, answerPath, cubHandle, cusHandle);

	hipblasDestroy(cubHandle);
	hipsparseDestroy(cusHandle);
	return 0;
}