#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cmath>

#include "../lib/lcg_cuda.h"

void read(std::string filePath, int *pN, int *pnz, double **cooVal,
	int **cooRowIdx, int **cooColIdx, double **b)
{
	std::ifstream in(filePath, std::ios::binary);

	in.read((char*)pN, sizeof(int));
	in.read((char*)pnz, sizeof(int));

	*cooVal = new double[*pnz]{};
	*cooRowIdx = new int[*pnz]{};
	*cooColIdx = new int[*pnz]{};
	*b = new double[*pN]{};

	for (int i = 0; i < *pnz; ++i)
	{
		in.read((char*)&(*cooRowIdx)[i], sizeof(int));
		in.read((char*)&(*cooColIdx)[i], sizeof(int));
		in.read((char*)&(*cooVal)[i], sizeof(double));
	}

	in.read((char*)(*b), sizeof(double)*(*pN));
    return;
}

void readAnswer(std::string filePath, int *pN, double **x)
{
	std::ifstream in(filePath, std::ios::binary);

	in.read((char*)pN, sizeof(int));

	*x = new double[*pN]{};

	in.read((char*)(*x), sizeof(double)*(*pN));
    return;
}

lcg_float avg_error(lcg_float *a, lcg_float *b, int n)
{
	lcg_float avg = 0.0;
	for (size_t i = 0; i < n; i++)
	{
		avg += (a[i] - b[i])*(a[i] - b[i]);
	}
	return sqrt(avg)/n;
}

// Declare as global variables
lcg_float one = 1.0;
lcg_float zero = 0.0;

void *d_buf;
hipsparseSpMatDescr_t smat_A;

int *d_rowIdxA; // COO
int *d_rowPtrA; // CSR
int *d_colIdxA;
double *d_A;
double *d_B;
double *d_pd;
double *d_ic;

hipsparseMatDescr_t descr_A = 0;
hipsparseMatDescr_t descr_L = 0;
csric02Info_t icinfo_A = 0;
csrsv2Info_t info_L = 0;
csrsv2Info_t info_LT = 0;

void cudaAx(void* instance, hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, 
    hipsparseDnVecDescr_t x, hipsparseDnVecDescr_t prod_Ax, const int n_size, const int nz_size)
{
	// Calculate the product of A*x
	hipsparseSpMV(cus_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, smat_A,
		x, &zero, prod_Ax, HIP_R_64F, HIPSPARSE_MV_ALG_DEFAULT, d_buf);
    return;
}

void cudaMx(void* instance, hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, 
    hipsparseDnVecDescr_t x, hipsparseDnVecDescr_t prod_Ax, const int n_size, const int nz_size)
{
	void *d_x, *d_Ax;
	hipsparseDnVecGetValues(x, &d_x);
	hipsparseDnVecGetValues(prod_Ax, &d_Ax);

	hipsparseDcsrsv2_solve(cus_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
		n_size, nz_size, &one, descr_L, d_ic, d_rowPtrA, d_colIdxA, info_L, (double*) d_x, (double*) d_pd, 
		HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_buf);

	hipsparseDcsrsv2_solve(cus_handle, HIPSPARSE_OPERATION_TRANSPOSE, 
		n_size, nz_size, &one, descr_L, d_ic, d_rowPtrA, d_colIdxA, info_LT, (double*) d_pd, (double*) d_Ax, 
		HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_buf);
    return;
}

int cudaProgress(void* instance, const lcg_float* m, const lcg_float converge, 
	const lcg_para* param, const int n_size, const int nz_size, const int k)
{
    if (converge <= param->epsilon) {
		std::clog << "Iteration-times: " << k << "\tconvergence: " << converge << std::endl;
	}
	return 0;
}

int main(int argc, char **argv)
{
	std::string inputPath = "data/case_10K_A";
	std::string answerPath = "data/case_10K_B";

	int N;
	int nz;
	double *A;
	int *rowIdxA;
	int *colIdxA;
	double *b;
	read(inputPath, &N, &nz, &A, &rowIdxA, &colIdxA, &b);

	double *ans_x;
	readAnswer(answerPath, &N, &ans_x);

	std::clog << "N = " << N << std::endl;
	std::clog << "nz = " << nz << std::endl;
	
	// Create handles
	hipblasHandle_t cubHandle;
	hipsparseHandle_t cusHandle;

	hipblasCreate(&cubHandle);
	hipsparseCreate(&cusHandle);

	// Allocate GPU memory & copy matrix/vector to device
	hipMalloc(&d_A, nz * sizeof(double));
	hipMalloc(&d_rowIdxA, nz * sizeof(int));
	hipMalloc(&d_rowPtrA, (N + 1) * sizeof(int));
	hipMalloc(&d_colIdxA, nz * sizeof(int));
	hipMalloc(&d_B, N * sizeof(double));
	hipMalloc(&d_pd, N * sizeof(double));

	hipMemcpy(d_A, A, nz * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_rowIdxA, rowIdxA, nz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_colIdxA, colIdxA, nz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_B, b, N * sizeof(double), hipMemcpyHostToDevice);

	// Convert matrix A from COO format to CSR format
	hipsparseXcoo2csr(cusHandle, d_rowIdxA, nz, N, d_rowPtrA, HIPSPARSE_INDEX_BASE_ZERO);

	// Create sparse matrix
	hipsparseCreateCsr(&smat_A, N, N, nz, d_rowPtrA, d_colIdxA, d_A, HIPSPARSE_INDEX_32I,
		HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

	// This is just used to get bufferSize;
	hipsparseDnVecDescr_t dvec_b;
	hipsparseCreateDnVec(&dvec_b, N, d_B, HIP_R_64F);

	size_t bufferSize_B;
	hipsparseSpMV_bufferSize(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, smat_A,
		dvec_b, &zero, dvec_b, HIP_R_64F, HIPSPARSE_MV_ALG_DEFAULT, &bufferSize_B);

	// --- Start of the preconditioning part ---

	// Copy A
	hipMalloc(&d_ic, nz * sizeof(lcg_float));
	hipMemcpy(d_ic, d_A, nz * sizeof(lcg_float), hipMemcpyDeviceToDevice);

	int bufferSize, bufferSize_A, bufferSize_L, bufferSize_LT;
	bufferSize = bufferSize_B;

	// create descriptor for matrix A
	hipsparseCreateMatDescr(&descr_A);

	// initialize properties of matrix A
	hipsparseSetMatType(descr_A, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatFillMode(descr_A, HIPSPARSE_FILL_MODE_LOWER);
	hipsparseSetMatDiagType(descr_A, HIPSPARSE_DIAG_TYPE_NON_UNIT);
	hipsparseSetMatIndexBase(descr_A, HIPSPARSE_INDEX_BASE_ZERO);

	// create descriptor for matrix L
	hipsparseCreateMatDescr(&descr_L);

	// initialize properties of matrix L
	hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
	hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_NON_UNIT);
	hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ZERO);

	// Create empty info objects for incomplete-cholesky factorization
	hipsparseCreateCsric02Info(&icinfo_A);
	hipsparseCreateCsrsv2Info(&info_L);
	hipsparseCreateCsrsv2Info(&info_LT);

	// Compute buffer size in computing ic factorization
	hipsparseDcsric02_bufferSize(cusHandle, N, nz, descr_A, d_A, d_rowPtrA, 
		d_colIdxA, icinfo_A, &bufferSize_A);
	hipsparseDcsrsv2_bufferSize(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
		N, nz, descr_L, d_ic, d_rowPtrA, d_colIdxA, info_L, &bufferSize_L);
	hipsparseDcsrsv2_bufferSize(cusHandle, HIPSPARSE_OPERATION_TRANSPOSE, 
		N, nz, descr_L, d_ic, d_rowPtrA, d_colIdxA, info_LT, &bufferSize_LT);
	
	bufferSize = max(max(max(bufferSize, bufferSize_A), bufferSize_L), bufferSize_LT);
	hipMalloc(&d_buf, bufferSize);

	// Perform incomplete-choleskey factorization: analysis phase
	hipsparseDcsric02_analysis(cusHandle, N, nz, descr_A, d_ic, d_rowPtrA, 
		d_colIdxA, icinfo_A, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_buf);
	hipsparseDcsrsv2_analysis(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
		N, nz, descr_L, d_ic, d_rowPtrA, d_colIdxA, info_L, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_buf);
	hipsparseDcsrsv2_analysis(cusHandle, HIPSPARSE_OPERATION_TRANSPOSE, 
		N, nz, descr_L, d_ic, d_rowPtrA, d_colIdxA, info_LT, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_buf);

	// Perform incomplete-choleskey factorization: solve phase
	hipsparseDcsric02(cusHandle, N, nz, descr_A, d_ic, d_rowPtrA, d_colIdxA, 
		icinfo_A, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_buf);

	// --- End of the preconditioning part ---

	// Declare an initial solution
    lcg_float *d_m;
    hipMalloc(&d_m, N * sizeof(lcg_float));

    lcg_para self_para = lcg_default_parameters();
	self_para.epsilon = 1e-6;
	self_para.abs_diff = 0;

	int ret;
	double *host_m = new double[N];

	// Solve with CG
	hipMemset(d_m, 0.0, N * sizeof(lcg_float));

    ret = lcg_solver_cuda(cudaAx, cudaProgress, d_m, d_B, N, nz, &self_para, nullptr, cubHandle, cusHandle, LCG_CG);
    lcg_error_str(ret);

	hipMemcpy(host_m, d_m, N * sizeof(double), hipMemcpyDeviceToHost);
	std::clog << "Averaged error (compared with ans_x): " << avg_error(host_m, ans_x, N) << std::endl;

	// Solve with CGS
	hipMemset(d_m, 0.0, N * sizeof(lcg_float));

	ret = lcg_solver_cuda(cudaAx, cudaProgress, d_m, d_B, N, nz, &self_para, nullptr, cubHandle, cusHandle, LCG_CGS);
    lcg_error_str(ret);

	hipMemcpy(host_m, d_m, N * sizeof(double), hipMemcpyDeviceToHost);
	std::clog << "Averaged error (compared with ans_x): " << avg_error(host_m, ans_x, N) << std::endl;

	// Solve with PCG
	hipMemset(d_m, 0.0, N * sizeof(lcg_float));

	ret = lcg_solver_preconditioned_cuda(cudaAx, cudaMx, cudaProgress, d_m, d_B, N, nz, &self_para, nullptr, cubHandle, cusHandle, LCG_PCG);
    lcg_error_str(ret);

	hipMemcpy(host_m, d_m, N * sizeof(double), hipMemcpyDeviceToHost);
	std::clog << "Averaged error (compared with ans_x): " << avg_error(host_m, ans_x, N) << std::endl;

	// Free Host memory
	delete[] A;
	delete[] rowIdxA;
	delete[] colIdxA;
	delete[] b;
	delete[] ans_x;
	delete[] host_m;

	// Free Device memory
	hipFree(d_A);
	hipFree(d_rowIdxA);
	hipFree(d_rowPtrA);
	hipFree(d_colIdxA);
	hipFree(d_B);
	hipFree(d_pd);
	hipFree(d_m);
	hipFree(d_ic);

	hipsparseDestroyDnVec(dvec_b);
	hipsparseDestroySpMat(smat_A);
	hipFree(d_buf);

	hipsparseDestroyMatDescr(descr_A);
	hipsparseDestroyMatDescr(descr_L);
	hipsparseDestroyCsric02Info(icinfo_A);
	hipsparseDestroyCsrsv2Info(info_L);
	hipsparseDestroyCsrsv2Info(info_LT);

	// Free handles
	hipblasDestroy(cubHandle);
	hipsparseDestroy(cusHandle);

	return 0;
}