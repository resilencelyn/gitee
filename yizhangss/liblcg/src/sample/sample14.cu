#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cmath>

#include "../lib/clcg_cudaf.h"
#include "../lib/preconditioner_cuda.h"

// Declare as global variables
hipComplex one = {1.0, 0.0};
hipComplex zero = {0.0, 0.0};

void read(std::string filePath, int *pN, int *pnz, hipDoubleComplex **cooVal,
	int **cooRowIdx, int **cooColIdx, hipDoubleComplex **b)
{
	std::ifstream in(filePath, std::ios::binary);

	in.read((char*)pN, sizeof(int));
	in.read((char*)pnz, sizeof(int));

	*cooVal = new hipDoubleComplex[*pnz]{};
	*cooRowIdx = new int[*pnz]{};
	*cooColIdx = new int[*pnz]{};
	*b = new hipDoubleComplex[*pN]{};

	for (int i = 0; i < *pnz; ++i)
	{
		in.read((char*)&(*cooRowIdx)[i], sizeof(int));
		in.read((char*)&(*cooColIdx)[i], sizeof(int));
		in.read((char*)&(*cooVal)[i], sizeof(hipDoubleComplex));
	}

	in.read((char*)(*b), sizeof(hipDoubleComplex)*(*pN));
    return;
}

void readAnswer(std::string filePath, int *pN, hipDoubleComplex **x)
{
	std::ifstream in(filePath, std::ios::binary);

	in.read((char*)pN, sizeof(int));

	*x = new hipDoubleComplex[*pN]{};

	in.read((char*)(*x), sizeof(hipDoubleComplex)*(*pN));
    return;
}

float avg_error(hipComplex *a, hipComplex *b, int n)
{
	float avg = 0.0;
	hipComplex tmp;
	for (size_t i = 0; i < n; i++)
	{
		tmp = Cdiff(a[i], b[i]);
		avg += (tmp.x*tmp.x + tmp.y*tmp.y);
	}
	return sqrt(avg)/n;
}

class sample14 : public CLCG_CUDAF_Solver
{
public:
	sample14(){}
	virtual ~sample14(){}

	void solve(std::string inputPath, std::string answerPath, hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

	void AxProduct(hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, hipsparseDnVecDescr_t x, hipsparseDnVecDescr_t prod_Ax, 
		const int n_size, const int nz_size, hipsparseOperation_t oper_t)
	{
		// Calculate the product of A*x
		hipsparseSpMV(cus_handle, oper_t, &one, smat_A, x, &zero, prod_Ax, HIP_C_32F, HIPSPARSE_SPMV_ALG_DEFAULT, d_buf);
		return;
	}

	void MxProduct(hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, hipsparseDnVecDescr_t x, hipsparseDnVecDescr_t prod_Ax, 
		const int n_size, const int nz_size, hipsparseOperation_t oper_t)
	{
		hipsparseSpSV_solve(cus_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, smat_L, x, dvec_p, 
			HIP_C_32F, HIPSPARSE_SPSV_ALG_DEFAULT, descr_L);
		
		hipsparseSpSV_solve(cus_handle, HIPSPARSE_OPERATION_TRANSPOSE, &one, smat_L, dvec_p, prod_Ax, 
			HIP_C_32F, HIPSPARSE_SPSV_ALG_DEFAULT, descr_LT);
		return;
	}

private:
	int N, nz, lnz;
	int *rowIdxA, *colIdxA;
	hipDoubleComplex *A, *b;
	hipDoubleComplex *ans_x;
    hipComplex *Af, *bf;
	hipComplex *ans_xf;

	int *L_row, *L_col;
    hipComplex *L_val;

	void *d_buf, *d_buf2;
	hipsparseSpMatDescr_t smat_A;
	hipsparseSpMatDescr_t smat_L;
	hipsparseSpSVDescr_t descr_L, descr_LT;

	int *d_rowIdxA; // COO
	int *d_rowPtrA; // CSR
	int *d_colIdxA;
	hipComplex *d_A;
	hipComplex *d_t;
	hipComplex *d_p;
	hipsparseDnVecDescr_t dvec_p;

	int *d_rowIdxL; // COO
	int *d_rowPtrL; // CSR
	int *d_colIdxL;
	hipComplex *d_L;

	hipComplex *host_m;
	hipsparseDnVecDescr_t dvec_tmp;
};

void sample14::solve(std::string inputPath, std::string answerPath, hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle)
{
	read(inputPath, &N, &nz, &A, &rowIdxA, &colIdxA, &b);
	readAnswer(answerPath, &N, &ans_x);

    incomplete_Cholesky_cuda_half_buffsize(rowIdxA, colIdxA, nz, &lnz);

	std::clog << "N = " << N << std::endl;
	std::clog << "nz = " << nz << std::endl;
    std::clog << "lnz = " << lnz << std::endl;

    Af = new hipComplex [nz];
    bf = new hipComplex [nz];
    ans_xf = new hipComplex [nz];

	// Note that converting complex numbers from double to single precisions may case stack overflow
    for (size_t i = 0; i < nz; i++)
    {
        Af[i].x = A[i].x; Af[i].y = A[i].y;
        bf[i].x = b[i].x; bf[i].y = b[i].y;
        ans_xf[i].x = ans_x[i].x; ans_xf[i].y = ans_x[i].y;
    }
    
	L_row = new int [lnz];
    L_col = new int [lnz];
    L_val = new hipComplex [lnz];

    incomplete_Cholesky_cuda_half(rowIdxA, colIdxA, Af, N, nz, lnz, L_row, L_col, L_val);
/*
    for (size_t i = 0; i < lnz; i++)
    {
        std::cout << L_row[i] << " " << L_col[i] << " (" << L_val[i].x << "," << L_val[i].y << ")\n";
    }
*/
    // Allocate GPU memory & copy matrix/vector to device
	hipMalloc(&d_A, nz * sizeof(hipComplex));
	hipMalloc(&d_rowIdxA, nz * sizeof(int));
	hipMalloc(&d_rowPtrA, (N + 1) * sizeof(int));
	hipMalloc(&d_colIdxA, nz * sizeof(int));
	hipMalloc(&d_t, N * sizeof(hipComplex));
	hipMalloc(&d_p, N * sizeof(hipComplex));
    hipsparseCreateDnVec(&dvec_p, N, d_p, HIP_C_32F);

	hipMemcpy(d_A, Af, nz * sizeof(hipComplex), hipMemcpyHostToDevice);
	hipMemcpy(d_rowIdxA, rowIdxA, nz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_colIdxA, colIdxA, nz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_t, bf, N * sizeof(hipComplex), hipMemcpyHostToDevice);

    hipMalloc(&d_L, lnz * sizeof(hipComplex));
	hipMalloc(&d_rowIdxL, lnz * sizeof(int));
	hipMalloc(&d_rowPtrL, (N + 1) * sizeof(int));
	hipMalloc(&d_colIdxL, lnz * sizeof(int));

    hipMemcpy(d_L, L_val, lnz * sizeof(hipComplex), hipMemcpyHostToDevice);
	hipMemcpy(d_rowIdxL, L_row, lnz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_colIdxL, L_col, lnz * sizeof(int), hipMemcpyHostToDevice);

	// Convert matrix A from COO format to CSR format
	hipsparseXcoo2csr(cus_handle, d_rowIdxA, nz, N, d_rowPtrA, HIPSPARSE_INDEX_BASE_ZERO);

	// Create sparse matrix
	hipsparseCreateCsr(&smat_A, N, N, nz, d_rowPtrA, d_colIdxA, d_A, HIPSPARSE_INDEX_32I,
		HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_32F);

	// Convert matrix L from COO format to CSR format
    hipsparseXcoo2csr(cus_handle, d_rowIdxL, lnz, N, d_rowPtrL, HIPSPARSE_INDEX_BASE_ZERO);

	// Create sparse matrix
    hipsparseCreateCsr(&smat_L, N, N, lnz, d_rowPtrL, d_colIdxL, d_L, HIPSPARSE_INDEX_32I,
		HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_32F);

    // Specify Lower fill mode.
    hipsparseFillMode_t fillmode = HIPSPARSE_FILL_MODE_LOWER;
	hipsparseSpMatSetAttribute(smat_L, HIPSPARSE_SPMAT_FILL_MODE, &fillmode, sizeof(fillmode));

	// Specify Non-Unit diagonal type.
    hipsparseDiagType_t diagtype = HIPSPARSE_DIAG_TYPE_NON_UNIT;
	hipsparseSpMatSetAttribute(smat_L, HIPSPARSE_SPMAT_DIAG_TYPE, &diagtype, sizeof(diagtype));

    // This is just used to get bufferSize;
	hipsparseCreateDnVec(&dvec_tmp, N, d_t, HIP_C_32F);

	size_t bufferSize_B;
	hipsparseSpMV_bufferSize(cus_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, smat_A,
		dvec_tmp, &zero, dvec_tmp, HIP_C_32F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize_B);

    // --- Start of the preconditioning part ---
    hipsparseSpSV_createDescr(&descr_L);
    hipsparseSpSV_createDescr(&descr_LT);

    size_t bufferSize, bufferSize_L, bufferSize_LT;
	bufferSize = bufferSize_B;

    hipsparseSpSV_bufferSize(cus_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, smat_L, dvec_p, 
        dvec_tmp, HIP_C_32F, HIPSPARSE_SPSV_ALG_DEFAULT, descr_L, &bufferSize_L);
    hipsparseSpSV_bufferSize(cus_handle, HIPSPARSE_OPERATION_TRANSPOSE, &one, smat_L, dvec_p, 
        dvec_tmp, HIP_C_32F, HIPSPARSE_SPSV_ALG_DEFAULT, descr_LT, &bufferSize_LT);

    bufferSize = max(max(bufferSize, bufferSize_L), bufferSize_LT);
	hipMalloc(&d_buf, bufferSize);
	hipMalloc(&d_buf2, bufferSize);

	hipsparseSpSV_analysis(cus_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, smat_L, dvec_tmp, dvec_p, 
		HIP_C_32F, HIPSPARSE_SPSV_ALG_DEFAULT, descr_L, d_buf);

	hipsparseSpSV_analysis(cus_handle, HIPSPARSE_OPERATION_TRANSPOSE, &one, smat_L, dvec_p, dvec_tmp, 
		HIP_C_32F, HIPSPARSE_SPSV_ALG_DEFAULT, descr_LT, d_buf2);
	// --- End of the preconditioning part ---

	// Declare an initial solution
    clcg_para self_para = clcg_default_parameters();
	self_para.epsilon = 1e-6;
	self_para.abs_diff = 0;

	host_m = new hipComplex[N];

	// Preconditioning with incomplete-chelosky factorization
	for (size_t i = 0; i < N; i++)
	{
		host_m[i].x = 0.0; host_m[i].y = 0.0;	
	}

	MinimizePreconditioned(cub_handle, cus_handle, host_m, bf, N, nz, CLCG_PCG);

	std::clog << "Averaged error (compared with ans_x): " << avg_error(host_m, ans_xf, N) << std::endl;

	// Free Host memory
	if (rowIdxA != nullptr) delete[] rowIdxA;
	if (colIdxA != nullptr) delete[] colIdxA;
    if (A != nullptr) delete[] A;
	if (b != nullptr) delete[] b;
	if (ans_x != nullptr) delete[] ans_x;
    if (Af != nullptr) delete[] Af;
	if (bf != nullptr) delete[] bf;
	if (ans_xf != nullptr) delete[] ans_xf;

    if (L_row != nullptr) delete[] L_row;
    if (L_col != nullptr) delete[] L_col;
    if (L_val != nullptr) delete[] L_val;

    if (host_m != nullptr) delete[] host_m;

	hipsparseDestroyDnVec(dvec_tmp);
    hipsparseDestroyDnVec(dvec_p);

	hipFree(d_buf);
	hipFree(d_buf2);
	hipFree(d_rowIdxA);
	hipFree(d_rowPtrA);
	hipFree(d_colIdxA);
    hipFree(d_A);
	hipFree(d_t);
	hipFree(d_p);

    hipFree(d_rowIdxL);
	hipFree(d_rowPtrL);
	hipFree(d_colIdxL);
    hipFree(d_L);

	hipsparseDestroySpMat(smat_A);
	hipsparseDestroySpMat(smat_L);
    hipsparseSpSV_destroyDescr(descr_L);
    hipsparseSpSV_destroyDescr(descr_LT);
	return;
}

int main(int argc, char **argv)
{
	std::string inputPath = "data/case_1K_cA";
	std::string answerPath = "data/case_1K_cB";

	hipblasHandle_t cubHandle;
	hipsparseHandle_t cusHandle;

	hipblasCreate(&cubHandle);
	hipsparseCreate(&cusHandle);

	sample14 sp;
	sp.set_report_interval(100);
	sp.solve(inputPath, answerPath, cubHandle, cusHandle);

	hipblasDestroy(cubHandle);
	hipsparseDestroy(cusHandle);
	return 0;
}