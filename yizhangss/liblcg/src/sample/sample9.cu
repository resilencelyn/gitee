#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cmath>

#include "../lib/clcg_cuda.h"

void read(std::string filePath, int *pN, int *pnz, hipDoubleComplex **cooVal,
	int **cooRowIdx, int **cooColIdx, hipDoubleComplex **b)
{
	std::ifstream in(filePath, std::ios::binary);

	in.read((char*)pN, sizeof(int));
	in.read((char*)pnz, sizeof(int));

	*cooVal = new hipDoubleComplex[*pnz]{};
	*cooRowIdx = new int[*pnz]{};
	*cooColIdx = new int[*pnz]{};
	*b = new hipDoubleComplex[*pN]{};

	for (int i = 0; i < *pnz; ++i)
	{
		in.read((char*)&(*cooRowIdx)[i], sizeof(int));
		in.read((char*)&(*cooColIdx)[i], sizeof(int));
		in.read((char*)&(*cooVal)[i], sizeof(hipDoubleComplex));
	}

	in.read((char*)(*b), sizeof(hipDoubleComplex)*(*pN));
    return;
}

void readAnswer(std::string filePath, int *pN, hipDoubleComplex **x)
{
	std::ifstream in(filePath, std::ios::binary);

	in.read((char*)pN, sizeof(int));

	*x = new hipDoubleComplex[*pN]{};

	in.read((char*)(*x), sizeof(hipDoubleComplex)*(*pN));
    return;
}

lcg_float avg_error(hipDoubleComplex *a, hipDoubleComplex *b, int n)
{
	lcg_float avg = 0.0;
	hipDoubleComplex tmp;
	for (size_t i = 0; i < n; i++)
	{
		tmp = Zdiff(a[i], b[i]);
		avg += (tmp.x*tmp.x + tmp.y*tmp.y);
	}
	return sqrt(avg)/n;
}

// Declare as global variables
hipDoubleComplex one, zero;

void *d_buf;
hipsparseSpMatDescr_t smat_A;

int *d_rowIdxA; // COO
int *d_rowPtrA; // CSR
int *d_colIdxA;
hipDoubleComplex *d_A;
hipDoubleComplex *d_B;

void cudaAx(void* instance, hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, 
    hipsparseDnVecDescr_t x, hipsparseDnVecDescr_t prod_Ax, const int n_size, const int nz_size, 
	hipsparseOperation_t oper_t)
{
	one.x = 1.0; one.y = 0.0;
	zero.x = 0.0; zero.y = 0.0;
	// Calculate the product of A*x
	hipsparseSpMV(cus_handle, oper_t, &one, smat_A, x, &zero, prod_Ax, HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT, d_buf);
    return;
}

int cudaProgress(void* instance, const hipDoubleComplex* m, const lcg_float converge, 
	const clcg_para* param, const int n_size, const int nz_size, const int k)
{
    if (converge <= param->epsilon) {
		std::clog << "Iteration-times: " << k << "\tconvergence: " << converge << std::endl;
	}
	return 0;
}

int main(int argc, char **argv)
{
	std::string inputPath = "data/case_1K_cA";
	std::string answerPath = "data/case_1K_cB";

	int N, nz;
	int *rowIdxA, *colIdxA;
	hipDoubleComplex *A, *b;

	read(inputPath, &N, &nz, &A, &rowIdxA, &colIdxA, &b);

	hipDoubleComplex *ans_x;
	readAnswer(answerPath, &N, &ans_x);

	std::clog << "N = " << N << std::endl;
	std::clog << "nz = " << nz << std::endl;

	// Create handles
	hipblasHandle_t cubHandle;
	hipsparseHandle_t cusHandle;

	hipblasCreate(&cubHandle);
	hipsparseCreate(&cusHandle);

	// Allocate GPU memory & copy matrix/vector to device
	hipMalloc(&d_A, nz * sizeof(hipDoubleComplex));
	hipMalloc(&d_rowIdxA, nz * sizeof(int));
	hipMalloc(&d_rowPtrA, (N + 1) * sizeof(int));
	hipMalloc(&d_colIdxA, nz * sizeof(int));
	hipMalloc(&d_B, N * sizeof(hipDoubleComplex));

	hipMemcpy(d_A, A, nz * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	hipMemcpy(d_rowIdxA, rowIdxA, nz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_colIdxA, colIdxA, nz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_B, b, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

	// Convert matrix A from COO format to CSR format
	hipsparseXcoo2csr(cusHandle, d_rowIdxA, nz, N, d_rowPtrA, HIPSPARSE_INDEX_BASE_ZERO);

	// Create sparse matrix
	hipsparseCreateCsr(&smat_A, N, N, nz, d_rowPtrA, d_colIdxA, d_A, HIPSPARSE_INDEX_32I,
		HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F);

	// This is just used to get bufferSize;
	hipsparseDnVecDescr_t dvec_b;
	hipsparseCreateDnVec(&dvec_b, N, d_B, HIP_C_64F);

	size_t bufferSize_B, bufferSize_B2;

	hipsparseSpMV_bufferSize(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, smat_A,
		dvec_b, &zero, dvec_b, HIP_C_64F, HIPSPARSE_MV_ALG_DEFAULT, &bufferSize_B);
	
	hipsparseSpMV_bufferSize(cusHandle, HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE, &one, smat_A,
		dvec_b, &zero, dvec_b, HIP_C_64F, HIPSPARSE_MV_ALG_DEFAULT, &bufferSize_B2);
	
	if (bufferSize_B2 > bufferSize_B) bufferSize_B = bufferSize_B2;
	hipMalloc(&d_buf, bufferSize_B);

	// Declare an initial solution
    hipDoubleComplex *d_m;
    hipMalloc(&d_m, N * sizeof(hipDoubleComplex));

    clcg_para self_para = clcg_default_parameters();
	self_para.epsilon = 1e-6;
	self_para.abs_diff = 0;

	int ret;
	hipDoubleComplex *host_m = new hipDoubleComplex[N];

	// Solve with BICG
	for (size_t i = 0; i < N; i++)
	{
		host_m[i].x = 0.0; host_m[i].y = 0.0;	
	}
	hipMemcpy(d_m, host_m, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    ret = clcg_solver_cuda(cudaAx, cudaProgress, d_m, d_B, N, nz, &self_para, nullptr, cubHandle, cusHandle, CLCG_BICG);
    lcg_error_str(ret);

	hipMemcpy(host_m, d_m, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
	std::clog << "Averaged error (compared with ans_x): " << avg_error(host_m, ans_x, N) << std::endl;

	// Solve with BICG_SYM
	for (size_t i = 0; i < N; i++)
	{
		host_m[i].x = 0.0; host_m[i].y = 0.0;	
	}
	hipMemcpy(d_m, host_m, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    ret = clcg_solver_cuda(cudaAx, cudaProgress, d_m, d_B, N, nz, &self_para, nullptr, cubHandle, cusHandle, CLCG_BICG_SYM);
    lcg_error_str(ret);

	hipMemcpy(host_m, d_m, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
	std::clog << "Averaged error (compared with ans_x): " << avg_error(host_m, ans_x, N) << std::endl;

	// Free Host memory
	delete[] A;
	delete[] rowIdxA;
	delete[] colIdxA;
	delete[] b;
	delete[] ans_x;
	delete[] host_m;

	// Free Device memory
	hipFree(d_A);
	hipFree(d_rowIdxA);
	hipFree(d_rowPtrA);
	hipFree(d_colIdxA);
	hipFree(d_B);
	hipFree(d_m);

	hipsparseDestroyDnVec(dvec_b);
	hipsparseDestroySpMat(smat_A);
	hipFree(d_buf);

	// Free handles
	hipblasDestroy(cubHandle);
	hipsparseDestroy(cusHandle);

	return 0;
}