#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cmath>

#include "../lib/clcg_cuda.h"
#include "../lib/preconditioner_cuda.h"

// Declare as global variables
hipDoubleComplex one = {1.0, 0.0};
hipDoubleComplex zero = {0.0, 0.0};

void read(std::string filePath, int *pN, int *pnz, hipDoubleComplex **cooVal,
	int **cooRowIdx, int **cooColIdx, hipDoubleComplex **b)
{
	std::ifstream in(filePath, std::ios::binary);

	in.read((char*)pN, sizeof(int));
	in.read((char*)pnz, sizeof(int));

	*cooVal = new hipDoubleComplex[*pnz]{};
	*cooRowIdx = new int[*pnz]{};
	*cooColIdx = new int[*pnz]{};
	*b = new hipDoubleComplex[*pN]{};

	for (int i = 0; i < *pnz; ++i)
	{
		in.read((char*)&(*cooRowIdx)[i], sizeof(int));
		in.read((char*)&(*cooColIdx)[i], sizeof(int));
		in.read((char*)&(*cooVal)[i], sizeof(hipDoubleComplex));
	}

	in.read((char*)(*b), sizeof(hipDoubleComplex)*(*pN));
    return;
}

void readAnswer(std::string filePath, int *pN, hipDoubleComplex **x)
{
	std::ifstream in(filePath, std::ios::binary);

	in.read((char*)pN, sizeof(int));

	*x = new hipDoubleComplex[*pN]{};

	in.read((char*)(*x), sizeof(hipDoubleComplex)*(*pN));
    return;
}

lcg_float avg_error(hipDoubleComplex *a, hipDoubleComplex *b, int n)
{
	lcg_float avg = 0.0;
	hipDoubleComplex tmp;
	for (size_t i = 0; i < n; i++)
	{
		tmp = Zdiff(a[i], b[i]);
		avg += (tmp.x*tmp.x + tmp.y*tmp.y);
	}
	return sqrt(avg)/n;
}

class sample13 : public CLCG_CUDA_Solver
{
public:
	sample13(){}
	virtual ~sample13(){}

	void solve(std::string inputPath, std::string answerPath, hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle);

	void AxProduct(hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, hipsparseDnVecDescr_t x, hipsparseDnVecDescr_t prod_Ax, 
		const int n_size, const int nz_size, hipsparseOperation_t oper_t)
	{
		// Calculate the product of A*x
		hipsparseSpMV(cus_handle, oper_t, &one, smat_A, x, &zero, prod_Ax, HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT, d_tuf);
		return;
	}

	void MxProduct(hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, hipsparseDnVecDescr_t x, hipsparseDnVecDescr_t prod_Ax, 
		const int n_size, const int nz_size, hipsparseOperation_t oper_t)
	{
		hipsparseSpSV_solve(cus_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, smat_L, x, dvec_p, 
			HIP_C_64F, HIPSPARSE_SPSV_ALG_DEFAULT, descr_L);
		
		hipsparseSpSV_solve(cus_handle, HIPSPARSE_OPERATION_TRANSPOSE, &one, smat_L, dvec_p, prod_Ax, 
			HIP_C_64F, HIPSPARSE_SPSV_ALG_DEFAULT, descr_LT);
		return;
	}

private:
	int N, nz, lnz;
	int *rowIdxA, *colIdxA;
	hipDoubleComplex *A, *b;
	hipDoubleComplex *ans_x;

	int *L_row, *L_col;
    hipDoubleComplex *L_val;

	void *d_tuf, *d_tuf2;
	hipsparseSpMatDescr_t smat_A;
	hipsparseSpMatDescr_t smat_L;
	hipsparseSpSVDescr_t descr_L, descr_LT;

	int *d_rowIdxA; // COO
	int *d_rowPtrA; // CSR
	int *d_colIdxA;
	hipDoubleComplex *d_A;
	hipDoubleComplex *d_t;
	hipDoubleComplex *d_p;
	hipsparseDnVecDescr_t dvec_p;

	int *d_rowIdxL; // COO
	int *d_rowPtrL; // CSR
	int *d_colIdxL;
	hipDoubleComplex *d_L;

	hipDoubleComplex *host_m;
	hipsparseDnVecDescr_t dvec_tmp;
};

void sample13::solve(std::string inputPath, std::string answerPath, hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle)
{
	read(inputPath, &N, &nz, &A, &rowIdxA, &colIdxA, &b);
	readAnswer(answerPath, &N, &ans_x);

    incomplete_Cholesky_cuda_half_buffsize(rowIdxA, colIdxA, nz, &lnz);

	std::clog << "N = " << N << std::endl;
	std::clog << "nz = " << nz << std::endl;
    std::clog << "lnz = " << lnz << std::endl;

	L_row = new int [lnz];
    L_col = new int [lnz];
    L_val = new hipDoubleComplex [lnz];

    incomplete_Cholesky_cuda_half(rowIdxA, colIdxA, A, N, nz, lnz, L_row, L_col, L_val);
/*
    for (size_t i = 0; i < lnz; i++)
    {
        std::cout << L_row[i] << " " << L_col[i] << " (" << L_val[i].x << "," << L_val[i].y << ")\n";
    }
*/
    // Allocate GPU memory & copy matrix/vector to device
	hipMalloc(&d_A, nz * sizeof(hipDoubleComplex));
	hipMalloc(&d_rowIdxA, nz * sizeof(int));
	hipMalloc(&d_rowPtrA, (N + 1) * sizeof(int));
	hipMalloc(&d_colIdxA, nz * sizeof(int));
	hipMalloc(&d_t, N * sizeof(hipDoubleComplex));
	hipMalloc(&d_p, N * sizeof(hipDoubleComplex));
    hipsparseCreateDnVec(&dvec_p, N, d_p, HIP_C_64F);

	hipMemcpy(d_A, A, nz * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	hipMemcpy(d_rowIdxA, rowIdxA, nz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_colIdxA, colIdxA, nz * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_L, lnz * sizeof(hipDoubleComplex));
	hipMalloc(&d_rowIdxL, lnz * sizeof(int));
	hipMalloc(&d_rowPtrL, (N + 1) * sizeof(int));
	hipMalloc(&d_colIdxL, lnz * sizeof(int));

    hipMemcpy(d_L, L_val, lnz * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	hipMemcpy(d_rowIdxL, L_row, lnz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_colIdxL, L_col, lnz * sizeof(int), hipMemcpyHostToDevice);

	// Convert matrix A from COO format to CSR format
	hipsparseXcoo2csr(cus_handle, d_rowIdxA, nz, N, d_rowPtrA, HIPSPARSE_INDEX_BASE_ZERO);

	// Create sparse matrix
	hipsparseCreateCsr(&smat_A, N, N, nz, d_rowPtrA, d_colIdxA, d_A, HIPSPARSE_INDEX_32I,
		HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F);

	// Convert matrix L from COO format to CSR format
    hipsparseXcoo2csr(cus_handle, d_rowIdxL, lnz, N, d_rowPtrL, HIPSPARSE_INDEX_BASE_ZERO);

	// Create sparse matrix
    hipsparseCreateCsr(&smat_L, N, N, lnz, d_rowPtrL, d_colIdxL, d_L, HIPSPARSE_INDEX_32I,
		HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F);

    // Specify Lower fill mode.
    hipsparseFillMode_t fillmode = HIPSPARSE_FILL_MODE_LOWER;
	hipsparseSpMatSetAttribute(smat_L, HIPSPARSE_SPMAT_FILL_MODE, &fillmode, sizeof(fillmode));

	// Specify Non-Unit diagonal type.
    hipsparseDiagType_t diagtype = HIPSPARSE_DIAG_TYPE_NON_UNIT;
	hipsparseSpMatSetAttribute(smat_L, HIPSPARSE_SPMAT_DIAG_TYPE, &diagtype, sizeof(diagtype));

    // This is just used to get bufferSize;
	hipsparseCreateDnVec(&dvec_tmp, N, d_t, HIP_C_64F);

	size_t bufferSize_B;
	hipsparseSpMV_bufferSize(cus_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, smat_A,
		dvec_tmp, &zero, dvec_tmp, HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize_B);

    // --- Start of the preconditioning part ---
    hipsparseSpSV_createDescr(&descr_L);
    hipsparseSpSV_createDescr(&descr_LT);

    size_t bufferSize, bufferSize_L, bufferSize_LT;
	bufferSize = bufferSize_B;

    hipsparseSpSV_bufferSize(cus_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, smat_L, dvec_p, 
        dvec_tmp, HIP_C_64F, HIPSPARSE_SPSV_ALG_DEFAULT, descr_L, &bufferSize_L);
    hipsparseSpSV_bufferSize(cus_handle, HIPSPARSE_OPERATION_TRANSPOSE, &one, smat_L, dvec_p, 
        dvec_tmp, HIP_C_64F, HIPSPARSE_SPSV_ALG_DEFAULT, descr_LT, &bufferSize_LT);

    bufferSize = max(max(bufferSize, bufferSize_L), bufferSize_LT);
	hipMalloc(&d_tuf, bufferSize);
	hipMalloc(&d_tuf2, bufferSize);

	hipsparseSpSV_analysis(cus_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, smat_L, dvec_tmp, dvec_p, 
		HIP_C_64F, HIPSPARSE_SPSV_ALG_DEFAULT, descr_L, d_tuf);

	hipsparseSpSV_analysis(cus_handle, HIPSPARSE_OPERATION_TRANSPOSE, &one, smat_L, dvec_p, dvec_tmp, 
		HIP_C_64F, HIPSPARSE_SPSV_ALG_DEFAULT, descr_LT, d_tuf2);
	// --- End of the preconditioning part ---

	// Declare an initial solution
    clcg_para self_para = clcg_default_parameters();
	self_para.epsilon = 1e-6;
	self_para.abs_diff = 0;

	// Preconditioning with incomplete-chelosky factorization
	host_m = clcg_malloc_cuda(N);
	clcg_vecset_cuda(host_m, zero, N);

	MinimizePreconditioned(cub_handle, cus_handle, host_m, b, N, nz, CLCG_PCG);

	std::clog << "Averaged error (compared with ans_x): " << avg_error(host_m, ans_x, N) << std::endl;

	// Free Host memory
	if (rowIdxA != nullptr) delete[] rowIdxA;
	if (colIdxA != nullptr) delete[] colIdxA;
    if (A != nullptr) delete[] A;
	if (b != nullptr) delete[] b;
	if (ans_x != nullptr) delete[] ans_x;

    if (L_row != nullptr) delete[] L_row;
    if (L_col != nullptr) delete[] L_col;
    if (L_val != nullptr) delete[] L_val;

	clcg_free_cuda(host_m);

	hipsparseDestroyDnVec(dvec_tmp);
    hipsparseDestroyDnVec(dvec_p);

	hipFree(d_tuf);
	hipFree(d_tuf2);
	hipFree(d_rowIdxA);
	hipFree(d_rowPtrA);
	hipFree(d_colIdxA);
    hipFree(d_A);
	hipFree(d_t);
	hipFree(d_p);

    hipFree(d_rowIdxL);
	hipFree(d_rowPtrL);
	hipFree(d_colIdxL);
    hipFree(d_L);

	hipsparseDestroySpMat(smat_A);
	hipsparseDestroySpMat(smat_L);
    hipsparseSpSV_destroyDescr(descr_L);
    hipsparseSpSV_destroyDescr(descr_LT);
	return;
}

int main(int argc, char **argv)
{
	std::string inputPath = "data/case_10K_cA";
	std::string answerPath = "data/case_10K_cB";

	hipblasHandle_t cubHandle;
	hipsparseHandle_t cusHandle;

	hipblasCreate(&cubHandle);
	hipsparseCreate(&cusHandle);

	sample13 sp;
	sp.set_report_interval(0);
	sp.solve(inputPath, answerPath, cubHandle, cusHandle);

	hipblasDestroy(cubHandle);
	hipsparseDestroy(cusHandle);
	return 0;
}