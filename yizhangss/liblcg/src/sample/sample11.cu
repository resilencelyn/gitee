#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cmath>

#include "../lib/clcg_cuda.h"

void read(std::string filePath, int *pN, int *pnz, hipDoubleComplex **cooVal,
	int **cooRowIdx, int **cooColIdx, hipDoubleComplex **b)
{
	std::ifstream in(filePath, std::ios::binary);

	in.read((char*)pN, sizeof(int));
	in.read((char*)pnz, sizeof(int));

	*cooVal = new hipDoubleComplex[*pnz]{};
	*cooRowIdx = new int[*pnz]{};
	*cooColIdx = new int[*pnz]{};
	*b = new hipDoubleComplex[*pN]{};

	for (int i = 0; i < *pnz; ++i)
	{
		in.read((char*)&(*cooRowIdx)[i], sizeof(int));
		in.read((char*)&(*cooColIdx)[i], sizeof(int));
		in.read((char*)&(*cooVal)[i], sizeof(hipDoubleComplex));
	}

	in.read((char*)(*b), sizeof(hipDoubleComplex)*(*pN));
    return;
}

void readAnswer(std::string filePath, int *pN, hipDoubleComplex **x)
{
	std::ifstream in(filePath, std::ios::binary);

	in.read((char*)pN, sizeof(int));

	*x = new hipDoubleComplex[*pN]{};

	in.read((char*)(*x), sizeof(hipDoubleComplex)*(*pN));
    return;
}

lcg_float avg_error(hipDoubleComplex *a, hipDoubleComplex *b, int n)
{
	lcg_float avg = 0.0;
	hipDoubleComplex tmp;
	for (size_t i = 0; i < n; i++)
	{
		tmp = Zdiff(a[i], b[i]);
		avg += (tmp.x*tmp.x + tmp.y*tmp.y);
	}
	return sqrt(avg)/n;
}

// Declare as global variables
hipDoubleComplex one, zero;

void *d_buf;
hipsparseSpMatDescr_t smat_A;

int *d_rowIdxA; // COO
int *d_rowPtrA; // CSR
int *d_colIdxA;
hipDoubleComplex *d_A;
hipDoubleComplex *d_pd;
hipDoubleComplex *d_iu;

hipsparseMatDescr_t descr_A = 0;
hipsparseMatDescr_t descr_L = 0;
hipsparseMatDescr_t descr_U = 0;
csrilu02Info_t info_ILU = 0;
csrsv2Info_t info_L = 0;
csrsv2Info_t info_U = 0;

void cudaAx(void* instance, hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, 
    hipsparseDnVecDescr_t x, hipsparseDnVecDescr_t prod_Ax, const int n_size, const int nz_size, 
	hipsparseOperation_t oper_t)
{
	one.x = 1.0; one.y = 0.0;
	zero.x = 0.0; zero.y = 0.0;
	// Calculate the product of A*x
	hipsparseSpMV(cus_handle, oper_t, &one, smat_A, x, &zero, prod_Ax, HIP_C_64F, HIPSPARSE_MV_ALG_DEFAULT, d_buf);
    return;
}

void cudaMx_ILU(void* instance, hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, 
    hipsparseDnVecDescr_t x, hipsparseDnVecDescr_t prod_Ax, const int n_size, const int nz_size, 
	hipsparseOperation_t oper_t)
{
	void *d_x, *d_Ax;
	hipsparseDnVecGetValues(x, &d_x);
	hipsparseDnVecGetValues(prod_Ax, &d_Ax);

	one.x = 1.0; one.y = 0.0;
	hipsparseZcsrsv2_solve(cus_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n_size, nz_size, &one, descr_L, d_iu, d_rowPtrA, d_colIdxA, info_L, (hipDoubleComplex*) d_x, (hipDoubleComplex*) d_pd, 
		HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_buf);

	hipsparseZcsrsv2_solve(cus_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n_size, nz_size, &one, descr_U, d_iu, d_rowPtrA, d_colIdxA, info_U, (hipDoubleComplex*) d_pd, (hipDoubleComplex*) d_Ax, 
		HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_buf);
    return;
}

int cudaProgress(void* instance, const hipDoubleComplex* m, const lcg_float converge, 
	const clcg_para* param, const int n_size, const int nz_size, const int k)
{
    if (converge <= param->epsilon) {
		std::clog << "Iteration-times: " << k << "\tconvergence: " << converge << std::endl;
	}
	return 0;
}

int main(int argc, char **argv)
{
	std::string inputPath = "data/case_1M_cA";
	std::string answerPath = "data/case_1M_cB";

	int N;
	int nz;
	hipDoubleComplex *A;
	int *rowIdxA;
	int *colIdxA;
	hipDoubleComplex *b;
	read(inputPath, &N, &nz, &A, &rowIdxA, &colIdxA, &b);

	hipDoubleComplex *ans_x;
	readAnswer(answerPath, &N, &ans_x);

	std::clog << "N = " << N << std::endl;
	std::clog << "nz = " << nz << std::endl;

	// Create handles
	hipblasHandle_t cubHandle;
	hipsparseHandle_t cusHandle;

	hipblasCreate(&cubHandle);
	hipsparseCreate(&cusHandle);

	// Allocate GPU memory & copy matrix/vector to device
	hipMalloc(&d_A, nz * sizeof(hipDoubleComplex));
	hipMalloc(&d_rowIdxA, nz * sizeof(int));
	hipMalloc(&d_rowPtrA, (N + 1) * sizeof(int));
	hipMalloc(&d_colIdxA, nz * sizeof(int));
	hipMalloc(&d_pd, N * sizeof(hipDoubleComplex));

	hipMemcpy(d_A, A, nz * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	hipMemcpy(d_rowIdxA, rowIdxA, nz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_colIdxA, colIdxA, nz * sizeof(int), hipMemcpyHostToDevice);

	// Convert matrix A from COO format to CSR format
	hipsparseXcoo2csr(cusHandle, d_rowIdxA, nz, N, d_rowPtrA, HIPSPARSE_INDEX_BASE_ZERO);

	// Create sparse matrix
	hipsparseCreateCsr(&smat_A, N, N, nz, d_rowPtrA, d_colIdxA, d_A, HIPSPARSE_INDEX_32I,
		HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F);

	// This is just used to get bufferSize;
	hipsparseDnVecDescr_t dvec_tmp;
	hipsparseCreateDnVec(&dvec_tmp, N, d_pd, HIP_C_64F);

	size_t bufferSize_B;
	hipsparseSpMV_bufferSize(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, smat_A,
		dvec_tmp, &zero, dvec_tmp, HIP_C_64F, HIPSPARSE_MV_ALG_DEFAULT, &bufferSize_B);

	// --- Start of the preconditioning part ---
	// Copy A
	hipMalloc(&d_iu, nz * sizeof(hipDoubleComplex));
	hipMemcpy(d_iu, d_A, nz * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);

	int bufferSize, bufferSize_A, bufferSize_L, bufferSize_U;
	bufferSize = bufferSize_B;

	// create descriptor for matrix A
	hipsparseCreateMatDescr(&descr_A);

	hipsparseSetMatType(descr_A, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr_A, HIPSPARSE_INDEX_BASE_ZERO);

	// create descriptor for matrix L
	hipsparseCreateMatDescr(&descr_L);

	// initialize properties of matrix L
	hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
	hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_UNIT);
	hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ZERO);

	// create descriptor for matrix U
	hipsparseCreateMatDescr(&descr_U);

	hipsparseSetMatType(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatFillMode(descr_U, HIPSPARSE_FILL_MODE_UPPER);
	hipsparseSetMatDiagType(descr_U, HIPSPARSE_DIAG_TYPE_NON_UNIT);
	hipsparseSetMatIndexBase(descr_U, HIPSPARSE_INDEX_BASE_ZERO);

	// Create empty info objects for incomplete-cholesky factorization
	hipsparseCreateCsrilu02Info(&info_ILU);
	hipsparseCreateCsrsv2Info(&info_L);
	hipsparseCreateCsrsv2Info(&info_U);

	// Compute buffer size in computing ic factorization
	hipsparseZcsrilu02_bufferSize(cusHandle, N, nz, descr_A, d_A, d_rowPtrA, 
		d_colIdxA, info_ILU, &bufferSize_A);
	hipsparseZcsrsv2_bufferSize(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
		N, nz, descr_L, d_iu, d_rowPtrA, d_colIdxA, info_L, &bufferSize_L);
	hipsparseZcsrsv2_bufferSize(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
		N, nz, descr_U, d_iu, d_rowPtrA, d_colIdxA, info_U, &bufferSize_U);
	
	bufferSize = max(max(max(bufferSize, bufferSize_A), bufferSize_L), bufferSize_U);
	hipMalloc(&d_buf, bufferSize);

	// Perform incomplete-choleskey factorization: analysis phase
	hipsparseZcsrilu02_analysis(cusHandle, N, nz, descr_A, d_iu, d_rowPtrA, 
		d_colIdxA, info_ILU, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_buf);
	hipsparseZcsrsv2_analysis(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
		N, nz, descr_L, d_iu, d_rowPtrA, d_colIdxA, info_L, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_buf);
	hipsparseZcsrsv2_analysis(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
		N, nz, descr_U, d_iu, d_rowPtrA, d_colIdxA, info_U, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_buf);

	// Perform incomplete-choleskey factorization: solve phase
	hipsparseZcsrilu02(cusHandle, N, nz, descr_A, d_iu, d_rowPtrA, d_colIdxA, 
		info_ILU, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_buf);
	// --- End of the preconditioning part ---

	// Declare an initial solution
    clcg_para self_para = clcg_default_parameters();
	self_para.epsilon = 1e-6;
	self_para.abs_diff = 0;

	int ret;
	hipDoubleComplex *host_m = new hipDoubleComplex[N];

	// Preconditioning with incomplete-LU factorization
	for (size_t i = 0; i < N; i++)
	{
		host_m[i].x = 0.0; host_m[i].y = 0.0;	
	}

	ret = clcg_solver_preconditioned_cuda(cudaAx, cudaMx_ILU, cudaProgress, host_m, b, N, nz, &self_para, nullptr, cubHandle, cusHandle, CLCG_PCG);
    lcg_error_str(ret);

	std::clog << "Averaged error (compared with ans_x): " << avg_error(host_m, ans_x, N) << std::endl;

	// Free Host memory
	delete[] A;
	delete[] rowIdxA;
	delete[] colIdxA;
	delete[] b;
	delete[] ans_x;
	delete[] host_m;

	// Free Device memory
	hipFree(d_A);
	hipFree(d_rowIdxA);
	hipFree(d_rowPtrA);
	hipFree(d_colIdxA);
	hipFree(d_pd);
	hipFree(d_iu);

	hipsparseDestroyDnVec(dvec_tmp);
	hipsparseDestroySpMat(smat_A);
	hipFree(d_buf);

	hipsparseDestroyMatDescr(descr_A);
	hipsparseDestroyMatDescr(descr_L);
	hipsparseDestroyMatDescr(descr_U);
	hipsparseDestroyCsrilu02Info(info_ILU);
	hipsparseDestroyCsrsv2Info(info_L);
	hipsparseDestroyCsrsv2Info(info_U);

	// Free handles
	hipblasDestroy(cubHandle);
	hipsparseDestroy(cusHandle);

	return 0;
}