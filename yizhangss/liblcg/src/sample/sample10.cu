#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cmath>

#include "../lib/clcg_cuda.h"

// Declare as global variables
hipDoubleComplex one = {1.0, 0.0};
hipDoubleComplex zero = {0.0, 0.0};

void read(std::string filePath, int *pN, int *pnz, hipDoubleComplex **cooVal,
	int **cooRowIdx, int **cooColIdx, hipDoubleComplex **b)
{
	std::ifstream in(filePath, std::ios::binary);

	in.read((char*)pN, sizeof(int));
	in.read((char*)pnz, sizeof(int));

	*cooVal = new hipDoubleComplex[*pnz]{};
	*cooRowIdx = new int[*pnz]{};
	*cooColIdx = new int[*pnz]{};
	*b = new hipDoubleComplex[*pN]{};

	for (int i = 0; i < *pnz; ++i)
	{
		in.read((char*)&(*cooRowIdx)[i], sizeof(int));
		in.read((char*)&(*cooColIdx)[i], sizeof(int));
		in.read((char*)&(*cooVal)[i], sizeof(hipDoubleComplex));
	}

	in.read((char*)(*b), sizeof(hipDoubleComplex)*(*pN));
    return;
}

void readAnswer(std::string filePath, int *pN, hipDoubleComplex **x)
{
	std::ifstream in(filePath, std::ios::binary);

	in.read((char*)pN, sizeof(int));

	*x = new hipDoubleComplex[*pN]{};

	in.read((char*)(*x), sizeof(hipDoubleComplex)*(*pN));
    return;
}

lcg_float avg_error(hipDoubleComplex *a, hipDoubleComplex *b, int n)
{
	lcg_float avg = 0.0;
	hipDoubleComplex tmp;
	for (size_t i = 0; i < n; i++)
	{
		tmp = Zdiff(a[i], b[i]);
		avg += (tmp.x*tmp.x + tmp.y*tmp.y);
	}
	return sqrt(avg)/n;
}

class sample10 : public CLCG_CUDA_Solver
{
public:
	sample10(){}
	virtual ~sample10(){}

	void solve(std::string inputPath, std::string answerPath);

	void AxProduct(hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, 
    hipsparseDnVecDescr_t x, hipsparseDnVecDescr_t prod_Ax, const int n_size, const int nz_size, 
	hipsparseOperation_t oper_t)
	{
		// Calculate the product of A*x
		hipsparseSpMV(cus_handle, oper_t, &one, smat_A, x, &zero, prod_Ax, HIP_C_64F, HIPSPARSE_MV_ALG_DEFAULT, d_buf);
		return;
	}

	void MxProduct(hipblasHandle_t cub_handle, hipsparseHandle_t cus_handle, 
		hipsparseDnVecDescr_t x, hipsparseDnVecDescr_t prod_Ax, const int n_size, const int nz_size, 
		hipsparseOperation_t oper_t)
	{
		void *d_x, *d_Ax;
		hipsparseDnVecGetValues(x, &d_x);
		hipsparseDnVecGetValues(prod_Ax, &d_Ax);

		if (use_incomplete_cholesky)
		{
			hipsparseZcsrsv2_solve(cus_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n_size, nz_size, &one, descr_L, d_ic, d_rowPtrA, d_colIdxA, info_L, (hipDoubleComplex*) d_x, (hipDoubleComplex*) d_pd, 
				HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_buf);

			hipsparseZcsrsv2_solve(cus_handle, HIPSPARSE_OPERATION_TRANSPOSE, n_size, nz_size, &one, descr_L, d_ic, d_rowPtrA, d_colIdxA, info_LT, (hipDoubleComplex*) d_pd, (hipDoubleComplex*) d_Ax, 
				HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_buf);
		}
		else
		{
			vecDvecZ_element_wise((hipDoubleComplex*) d_x, d_pd, (hipDoubleComplex*) d_Ax, n_size);
		}	
		return;
	}

private:
	bool use_incomplete_cholesky;

	int N, nz;
	int *rowIdxA, *colIdxA;
	hipDoubleComplex *A, *b;
	hipDoubleComplex *ans_x;

	void *d_buf;
	hipsparseSpMatDescr_t smat_A;

	int *d_rowIdxA; // COO
	int *d_rowPtrA; // CSR
	int *d_colIdxA;
	hipDoubleComplex *d_A;
	hipDoubleComplex *d_B;
	hipDoubleComplex *d_pd;
	hipDoubleComplex *d_ic;

	hipsparseMatDescr_t descr_A;
	hipsparseMatDescr_t descr_L;
	csric02Info_t icinfo_A;
	csrsv2Info_t info_L;
	csrsv2Info_t info_LT;

	hipDoubleComplex *host_m;
	hipDoubleComplex *d_m;
	hipsparseDnVecDescr_t dvec_b;
};

void sample10::solve(std::string inputPath, std::string answerPath)
{
	read(inputPath, &N, &nz, &A, &rowIdxA, &colIdxA, &b);
	readAnswer(answerPath, &N, &ans_x);

	std::clog << "N = " << N << std::endl;
	std::clog << "nz = " << nz << std::endl;

	// Create handles
	hipblasHandle_t cubHandle;
	hipsparseHandle_t cusHandle;

	hipblasCreate(&cubHandle);
	hipsparseCreate(&cusHandle);

	// Allocate GPU memory & copy matrix/vector to device
	hipMalloc(&d_A, nz * sizeof(hipDoubleComplex));
	hipMalloc(&d_rowIdxA, nz * sizeof(int));
	hipMalloc(&d_rowPtrA, (N + 1) * sizeof(int));
	hipMalloc(&d_colIdxA, nz * sizeof(int));
	hipMalloc(&d_B, N * sizeof(hipDoubleComplex));
	hipMalloc(&d_pd, N * sizeof(hipDoubleComplex));

	hipMemcpy(d_A, A, nz * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	hipMemcpy(d_rowIdxA, rowIdxA, nz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_colIdxA, colIdxA, nz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_B, b, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

	// Convert matrix A from COO format to CSR format
	hipsparseXcoo2csr(cusHandle, d_rowIdxA, nz, N, d_rowPtrA, HIPSPARSE_INDEX_BASE_ZERO);

	// Create sparse matrix
	hipsparseCreateCsr(&smat_A, N, N, nz, d_rowPtrA, d_colIdxA, d_A, HIPSPARSE_INDEX_32I,
		HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F);

	// This is just used to get bufferSize;
	hipsparseDnVecDescr_t dvec_b;
	hipsparseCreateDnVec(&dvec_b, N, d_B, HIP_C_64F);

	size_t bufferSize_B;
	hipsparseSpMV_bufferSize(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, smat_A,
		dvec_b, &zero, dvec_b, HIP_C_64F, HIPSPARSE_MV_ALG_DEFAULT, &bufferSize_B);

	// --- Start of the preconditioning part ---
	// Get the diagonal elemenets
	smZcsr_get_diagonal(d_rowPtrA, d_colIdxA, d_A, N, d_pd);

	// Copy A
	hipMalloc(&d_ic, nz * sizeof(hipDoubleComplex));
	hipMemcpy(d_ic, d_A, nz * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);

	// create descriptor for matrix A
	hipsparseCreateMatDescr(&descr_A);

	// initialize properties of matrix A
	hipsparseSetMatType(descr_A, HIPSPARSE_MATRIX_TYPE_SYMMETRIC);
	hipsparseSetMatFillMode(descr_A, HIPSPARSE_FILL_MODE_LOWER);
	hipsparseSetMatDiagType(descr_A, HIPSPARSE_DIAG_TYPE_NON_UNIT);
	hipsparseSetMatIndexBase(descr_A, HIPSPARSE_INDEX_BASE_ZERO);

	// create descriptor for matrix L
	hipsparseCreateMatDescr(&descr_L);

	// initialize properties of matrix L
	hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
	hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_NON_UNIT);
	hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ZERO);

	// Create empty info objects for incomplete-cholesky factorization
	hipsparseCreateCsric02Info(&icinfo_A);
	hipsparseCreateCsrsv2Info(&info_L);
	hipsparseCreateCsrsv2Info(&info_LT);

	int bufferSize, bufferSize_A, bufferSize_L, bufferSize_LT;
	bufferSize = bufferSize_B;

	// Compute buffer size in computing ic factorization
	hipsparseZcsric02_bufferSize(cusHandle, N, nz, descr_A, d_A, d_rowPtrA, 
		d_colIdxA, icinfo_A, &bufferSize_A);
	hipsparseZcsrsv2_bufferSize(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
		N, nz, descr_L, d_ic, d_rowPtrA, d_colIdxA, info_L, &bufferSize_L);
	hipsparseZcsrsv2_bufferSize(cusHandle, HIPSPARSE_OPERATION_TRANSPOSE, 
		N, nz, descr_L, d_ic, d_rowPtrA, d_colIdxA, info_LT, &bufferSize_LT);
	
	bufferSize = max(max(max(bufferSize, bufferSize_A), bufferSize_L), bufferSize_LT);
	hipMalloc(&d_buf, bufferSize);

	// Perform incomplete-choleskey factorization: analysis phase
	hipsparseZcsric02_analysis(cusHandle, N, nz, descr_A, d_ic, d_rowPtrA, 
		d_colIdxA, icinfo_A, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_buf);
	hipsparseZcsrsv2_analysis(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
		N, nz, descr_L, d_ic, d_rowPtrA, d_colIdxA, info_L, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_buf);
	hipsparseZcsrsv2_analysis(cusHandle, HIPSPARSE_OPERATION_TRANSPOSE, 
		N, nz, descr_L, d_ic, d_rowPtrA, d_colIdxA, info_LT, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_buf);

	// Perform incomplete-choleskey factorization: solve phase
	hipsparseZcsric02(cusHandle, N, nz, descr_A, d_ic, d_rowPtrA, d_colIdxA, 
		icinfo_A, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_buf);
	// --- End of the preconditioning part ---

	// Declare an initial solution
	host_m = new hipDoubleComplex[N];
    hipMalloc(&d_m, N * sizeof(hipDoubleComplex));

    clcg_para self_para = clcg_default_parameters();
	self_para.epsilon = 1e-6;

	// Preconditioning with Diagonal elements
	for (size_t i = 0; i < N; i++)
	{
		host_m[i].x = 0.0; host_m[i].y = 0.0;	
	}
	hipMemcpy(d_m, host_m, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

	use_incomplete_cholesky = false;
	MinimizePreconditioned(cubHandle, cusHandle, d_m, d_B, N, nz, CLCG_PCG);

	hipMemcpy(host_m, d_m, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
	std::clog << "Averaged error (compared with ans_x): " << avg_error(host_m, ans_x, N) << std::endl;
	
	// Preconditioning with incomplete-Cholesky factorization
	for (size_t i = 0; i < N; i++)
	{
		host_m[i].x = 0.0; host_m[i].y = 0.0;	
	}
	hipMemcpy(d_m, host_m, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

	use_incomplete_cholesky = true;
	MinimizePreconditioned(cubHandle, cusHandle, d_m, d_B, N, nz, CLCG_PCG);

	hipMemcpy(host_m, d_m, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
	std::clog << "Averaged error (compared with ans_x): " << avg_error(host_m, ans_x, N) << std::endl;

	// Free Host memory
	delete[] A;
	delete[] rowIdxA;
	delete[] colIdxA;
	delete[] b;
	delete[] ans_x;
	delete[] host_m;

	// Free Device memory
	hipFree(d_A);
	hipFree(d_rowIdxA);
	hipFree(d_rowPtrA);
	hipFree(d_colIdxA);
	hipFree(d_B);
	hipFree(d_pd);
	hipFree(d_m);
	hipFree(d_ic);

	hipsparseDestroyDnVec(dvec_b);
	hipsparseDestroySpMat(smat_A);
	hipFree(d_buf);

	hipsparseDestroyMatDescr(descr_A);
	hipsparseDestroyMatDescr(descr_L);
	hipsparseDestroyCsric02Info(icinfo_A);
	hipsparseDestroyCsrsv2Info(info_L);
	hipsparseDestroyCsrsv2Info(info_LT);

	// Free handles
	hipblasDestroy(cubHandle);
	hipsparseDestroy(cusHandle);
	return;
}

int main(int argc, char **argv)
{
	std::string inputPath = "data/case_10K_cA";
	std::string answerPath = "data/case_10K_cB";

	sample10 sp;
	sp.set_report_interval(0);
	sp.solve(inputPath, answerPath);
	return 0;
}