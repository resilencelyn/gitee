#include "hip/hip_runtime.h"
#include "NeighborPointQuery.h"
#include "Topology/GridHash.h"

namespace dyno
{
	__constant__ int offset_nq[27][3] = { 
		0, 0, 0,
		0, 0, 1,
		0, 1, 0,
		1, 0, 0,
		0, 0, -1,
		0, -1, 0,
		-1, 0, 0,
		0, 1, 1,
		0, 1, -1,
		0, -1, 1,
		0, -1, -1,
		1, 0, 1,
		1, 0, -1,
		-1, 0, 1,
		-1, 0, -1,
		1, 1, 0,
		1, -1, 0,
		-1, 1, 0,
		-1, -1, 0,
		1, 1, 1,
		1, 1, -1,
		1, -1, 1,
		-1, 1, 1,
		1, -1, -1,
		-1, 1, -1,
		-1, -1, 1,
		-1, -1, -1
	};

	IMPLEMENT_TCLASS(NeighborPointQuery, TDataType)

	template<typename TDataType>
	NeighborPointQuery<TDataType>::NeighborPointQuery()
		: ComputeModule()
	{
		this->inOther()->tagOptional(true);
	}

	template<typename TDataType>
	NeighborPointQuery<TDataType>::~NeighborPointQuery()
	{
	}

	template<typename TDataType>
	void NeighborPointQuery<TDataType>::compute()
	{
		if (this->varSizeLimit()->getData() <= 0) {
			requestDynamicNeighborIds();
		}
		else {
			requestFixedSizeNeighborIds();
		}
	}

	template<typename Real, typename Coord, typename TDataType>
	__global__ void K_CalNeighborSize(
		DArray<int> count,
		DArray<Coord> position_new,
		DArray<Coord> position, 
		GridHash<TDataType> hash, 
		Real h)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= position_new.size()) return;

		Coord pos_ijk = position_new[pId];
		int3 gId3 = hash.getIndex3(pos_ijk);

		int counter = 0;
		for (int c = 0; c < 27; c++)
		{
			int cId = hash.getIndex(gId3.x + offset_nq[c][0], gId3.y + offset_nq[c][1], gId3.z + offset_nq[c][2]);
			if (cId >= 0) {
				int totalNum = hash.getCounter(cId);
				for (int i = 0; i < totalNum; i++) {
					int nbId = hash.getParticleId(cId, i);
					Real d_ij = (pos_ijk - position[nbId]).norm();
					if (d_ij < h)
					{
						counter++;
					}
				}
			}
		}

		count[pId] = counter;
	}
	

	template<typename Real, typename Coord, typename TDataType>
	__global__ void K_GetNeighborElements(
		DArrayList<int> nbrIds,
		DArray<Coord> position_new,
		DArray<Coord> position, 
		GridHash<TDataType> hash, 
		Real h)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= position_new.size()) return;

		Coord pos_ijk = position_new[pId];
		int3 gId3 = hash.getIndex3(pos_ijk);

		List<int>& list_i = nbrIds[pId];

		int j = 0;
		for (int c = 0; c < 27; c++)
		{
			int cId = hash.getIndex(gId3.x + offset_nq[c][0], gId3.y + offset_nq[c][1], gId3.z + offset_nq[c][2]);
			if (cId >= 0) {
				int totalNum = hash.getCounter(cId);
				for (int i = 0; i < totalNum; i++) {
					int nbId = hash.getParticleId(cId, i);
					Real d_ij = (pos_ijk - position[nbId]).norm();
					if (d_ij < h)
					{
						list_i.insert(nbId);
						j++;
					}
				}
			}
		}
	}

	template<typename TDataType>
	void NeighborPointQuery<TDataType>::requestDynamicNeighborIds()
	{
		// Prepare inputs
		auto& points	= this->inPosition()->getData();
		auto& other		= this->inOther()->isEmpty() ? this->inPosition()->getData() : this->inOther()->getData();
		auto h			= this->inRadius()->getData();

		// Prepare outputs
		if (this->outNeighborIds()->isEmpty())
			this->outNeighborIds()->allocate();

		auto& nbrIds = this->outNeighborIds()->getData();

		// Construct hash grid
		Reduction<Coord> reduce;
		Coord hiBound = reduce.maximum(points.begin(), points.size());
		Coord loBound = reduce.minimum(points.begin(), points.size());

		GridHash<TDataType> hashGrid;
		hashGrid.setSpace(h, loBound - Coord(h), hiBound + Coord(h));
		hashGrid.clear();
		hashGrid.construct(points);

		DArray<int> counter(other.size());
		cuExecute(other.size(),
			K_CalNeighborSize,
			counter,
			other,
			points, 
			hashGrid, 
			h);

		nbrIds.resize(counter);

		cuExecute(other.size(),
			K_GetNeighborElements,
			nbrIds, 
			other,
			points, 
			hashGrid,
			h);

		counter.clear();
		hashGrid.release();
	}
	
	template<typename Real, typename Coord, typename TDataType>
	__global__ void K_ComputeNeighborFixed(
		DArrayList<int> nbrIds, 
		DArray<Coord> position_new,
		DArray<Coord> position, 
		GridHash<TDataType> hash, 
		Real h,
		int sizeLimit,
		DArray<int> heapIDs,
		DArray<Real> heapDistance)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= position_new.size()) return;

		int* ids(heapIDs.begin() + pId * sizeLimit);// = new int[nbrLimit];
		Real* distance(heapDistance.begin() + pId * sizeLimit);// = new Real[nbrLimit];

		Coord pos_ijk = position_new[pId];
		int3 gId3 = hash.getIndex3(pos_ijk);

		int counter = 0;
		for (int c = 0; c < 27; c++)
		{
			int cId = hash.getIndex(gId3.x + offset_nq[c][0], gId3.y + offset_nq[c][1], gId3.z + offset_nq[c][2]);
			if (cId >= 0) {
				int totalNum = hash.getCounter(cId);// min(hash.getCounter(cId), hash.npMax);
				for (int i = 0; i < totalNum; i++) {
					int nbId = hash.getParticleId(cId, i);
					float d_ij = (pos_ijk - position[nbId]).norm();
					if (d_ij < h)
					{
						if (counter < sizeLimit)
						{
							ids[counter] = nbId;
							distance[counter] = d_ij;
							counter++;
						}
						else
						{
							int maxId = 0;
							float maxDist = distance[0];
							for (int ne = 1; ne < sizeLimit; ne++)
							{
								if (maxDist < distance[ne])
								{
									maxDist = distance[ne];
									maxId = ne;
								}
							}
							if (d_ij < distance[maxId])
							{
								distance[maxId] = d_ij;
								ids[maxId] = nbId;
							}
						}
					}
				}
			}
		}

		List<int>& list_i = nbrIds[pId];
		for (int bId = 0; bId < counter; bId++)
		{
			list_i.insert(ids[bId]);
		}
	}

	template<typename TDataType>
	void NeighborPointQuery<TDataType>::requestFixedSizeNeighborIds()
	{
		// Prepare inputs
		auto& points	= this->inPosition()->getData();
		auto& other		= this->inOther()->isEmpty() ? this->inPosition()->getData() : this->inOther()->getData();
		auto h			= this->inRadius()->getData();

		// Prepare outputs
		if (this->outNeighborIds()->isEmpty())
			this->outNeighborIds()->allocate();

		auto& nbrIds = this->outNeighborIds()->getData();

		uint numPt  = this->inPosition()->getDataPtr()->size();
		uint sizeLimit = this->varSizeLimit()->getData();
		
		nbrIds.resize(numPt, sizeLimit);

		// Construct hash grid
		Reduction<Coord> reduce;
		Coord hiBound = reduce.maximum(points.begin(), points.size());
		Coord loBound = reduce.minimum(points.begin(), points.size());

		GridHash<TDataType> hashGrid;
		hashGrid.setSpace(h, loBound - Coord(h), hiBound + Coord(h));
		hashGrid.clear();
		hashGrid.construct(points);

		DArray<int> ids(numPt * sizeLimit);
		DArray<Real> distance(numPt * sizeLimit);
		cuExecute(numPt,
			K_ComputeNeighborFixed,
			nbrIds,
			other,
			points,
			hashGrid,
			h,
			sizeLimit,
			ids,
			distance);

		ids.clear();
		distance.clear();
		hashGrid.clear();
	}

	DEFINE_CLASS(NeighborPointQuery);
}