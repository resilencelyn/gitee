#include "hip/hip_runtime.h"
#include "TriangleSet.h"
#include <fstream>
#include <iostream>
#include <sstream>

#include <thrust/sort.h>

namespace dyno
{
	template<typename TDataType>
	TriangleSet<TDataType>::TriangleSet()
		: EdgeSet<TDataType>()
	{
		std::vector<Coord> positions;
		std::vector<Triangle> triangles;
		Real dx = Real(0.1);
		int Nx = 11;
		int Nz = 11;

		for (int k = 0; k < Nz; k++) {
			for (int i = 0; i < Nx; i++) {
				positions.push_back(Coord(Real(i*dx), Real(0.0), Real(k*dx)));
				if (k < Nz - 1 && i < Nx - 1)
				{
					Triangle tri1(i + k*Nx, i + 1 + k*Nx, i + 1 + (k + 1)*Nx);
					Triangle tri2(i + k*Nx, i + 1 + (k + 1)*Nx, i + (k + 1)*Nx);
					triangles.push_back(tri1);
					triangles.push_back(tri2);
				}
			}
		}
		this->setPoints(positions);
		this->setTriangles(triangles);
	}

	template<typename TDataType>
	TriangleSet<TDataType>::~TriangleSet()
	{
	}

	template<typename Triangle>
	__global__ void TS_CountTriangles(
		DArray<int> counter,
		DArray<Triangle> triangles)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= triangles.size()) return;

		Triangle t = triangles[tId];

		atomicAdd(&counter[t[0]], 1);
		atomicAdd(&counter[t[1]], 1);
		atomicAdd(&counter[t[2]], 1);
	}

	template<typename Triangle>
	__global__ void TS_SetupTriIds(
		DArrayList<int> triIds,
		DArray<Triangle> triangles)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= triangles.size()) return;

		Triangle t = triangles[tId];

		triIds[t[0]].atomicInsert(tId);
		triIds[t[1]].atomicInsert(tId);
		triIds[t[2]].atomicInsert(tId);
	}

	template<typename TDataType>
	DArrayList<int>& TriangleSet<TDataType>::getVertex2Triangles()
	{
		DArray<int> counter(m_coords.size());
		counter.reset();

		cuExecute(m_triangles.size(),
			TS_CountTriangles,
			counter,
			m_triangles);

		m_ver2Tri.resize(counter);

		counter.reset();
		cuExecute(m_triangles.size(),
			TS_SetupTriIds,
			m_ver2Tri,
			m_triangles);

		counter.clear();

		return m_ver2Tri;
	}

	template<typename EKey, typename Triangle>
	__global__ void TS_SetupKeys(
		DArray<EKey> keys,
		DArray<int> ids,
		DArray<Triangle> triangles)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= triangles.size()) return;

		Triangle tri = triangles[tId];
		keys[3 * tId] = EKey(tri[0], tri[1]);
		keys[3 * tId + 1] = EKey(tri[1], tri[2]);
		keys[3 * tId + 2] = EKey(tri[2], tri[0]);

		ids[3 * tId] = tId;
		ids[3 * tId + 1] = tId;
		ids[3 * tId + 2] = tId;
	}

	template<typename EKey>
	__global__ void TS_CountEdgeNumber(
		DArray<int> counter,
		DArray<EKey> keys)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= keys.size()) return;

		if (tId == 0 || keys[tId] != keys[tId - 1])
			counter[tId] = 1;
		else
			counter[tId] = 0;
	}

	template<typename Edge, typename Edg2Tri, typename EKey>
	__global__ void TS_SetupEdges(
		DArray<Edge> edges,
		DArray<Edg2Tri> edg2Tri,
		DArray<EKey> keys,
		DArray<int> counter,
		DArray<int> triIds)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= keys.size()) return;

		int shift = counter[tId];
		if (tId == 0 || keys[tId] != keys[tId - 1])
		{
			EKey key = keys[tId];
			edges[shift] = Edge(key[0], key[1]);

			Edg2Tri e2T(EMPTY, EMPTY);
			e2T[0] = triIds[tId];

			if (tId + 1 < keys.size() && keys[tId + 1] == key)
				e2T[1] = triIds[tId + 1];

			edg2Tri[shift] = e2T;

// 			printf("T2T %d: %d %d \n", shift, t2T[0], t2T[1]);
// 
// 			printf("Tri %d: %d %d %d; Tet: %d \n", shift, keys[tId][0], keys[tId][1], keys[tId][2], tetIds[tId]);
// 			printf("Counter: %d \n", shift, counter[tId]);
		}
	}

	template<typename TDataType>
	void TriangleSet<TDataType>::updateEdges()
	{
		uint triSize = m_triangles.size();

		DArray<EKey> keys;
		DArray<int> triIds;

		keys.resize(3 * triSize);
		triIds.resize(3 * triSize);

		cuExecute(triSize,
			TS_SetupKeys,
			keys,
			triIds,
			m_triangles);

		thrust::sort_by_key(thrust::device, keys.begin(), keys.begin() + keys.size(), triIds.begin());

		DArray<int> counter;
		counter.resize(3 * triSize);

		cuExecute(keys.size(),
			TS_CountEdgeNumber,
			counter,
			keys);

		int edgeNum = thrust::reduce(thrust::device, counter.begin(), counter.begin() + counter.size());
		thrust::exclusive_scan(thrust::device, counter.begin(), counter.begin() + counter.size(), counter.begin());

		edg2Tri.resize(edgeNum);

		auto& pEdges = this->getEdges();
		pEdges.resize(edgeNum);
		cuExecute(keys.size(),
			TS_SetupEdges,
			pEdges,
			edg2Tri,
			keys,
			counter,
			triIds);

		counter.clear();
		triIds.clear();
		keys.clear();
	}

	template<typename TDataType>
	void TriangleSet<TDataType>::setTriangles(std::vector<Triangle>& triangles)
	{
		m_triangles.resize(triangles.size());
		m_triangles.assign(triangles);
	}

	template<typename TDataType>
	void TriangleSet<TDataType>::loadObjFile(std::string filename)
	{
		if (filename.size() < 5 || filename.substr(filename.size() - 4) != std::string(".obj")) {
			std::cerr << "Error: Expected OBJ file with filename of the form <name>.obj.\n";
			exit(-1);
		}

		std::ifstream infile(filename);
		if (!infile) {
			std::cerr << "Failed to open. Terminating.\n";
			exit(-1);
		}

		int ignored_lines = 0;
		std::string line;
		std::vector<Coord> vertList;
		std::vector<Triangle> faceList;
		while (!infile.eof()) {
			std::getline(infile, line);

			//.obj files sometimes contain vertex normals indicated by "vn"
			if (line.substr(0, 1) == std::string("v") && line.substr(0, 2) != std::string("vn")) {
				std::stringstream data(line);
				char c;
				Coord point;
				data >> c >> point[0] >> point[1] >> point[2];
				vertList.push_back(point);
			}
			else if (line.substr(0, 1) == std::string("f")) {
				std::stringstream data(line);
				char c;
				int v0, v1, v2;
				data >> c >> v0 >> v1 >> v2;
				faceList.push_back(Triangle(v0 - 1, v1 - 1, v2 - 1));
			}
			else {
				++ignored_lines;
			}
		}
		infile.close();

		this->setPoints(vertList);
		setTriangles(faceList);
	}

	template<typename TDataType>
	void TriangleSet<TDataType>::copyFrom(TriangleSet<TDataType>& triangleSet)
	{
		m_ver2Tri.assign(triangleSet.m_ver2Tri);

		m_triangles.resize(triangleSet.m_triangles.size());
		m_triangles.assign(triangleSet.m_triangles);

		edg2Tri.resize(triangleSet.edg2Tri.size());
		edg2Tri.assign(triangleSet.edg2Tri);

		EdgeSet<TDataType>::copyFrom(triangleSet);
	}

	template<typename Coord, typename Triangle>
	__global__ void TS_SetupVertexNormals(
		DArray<Coord> normals,
		DArray<Coord> vertices,
		DArray<Triangle> triangles,
		DArrayList<int> triIds)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= normals.size()) return;

		List<int>& list_i = triIds[tId];
		int triSize = list_i.size();

		Coord N = Coord(0);
		for (int ne = 0; ne < triSize; ne++)
		{
			int j = list_i[ne];
			Triangle t = triangles[j];

			Coord v0 = vertices[t[0]];
			Coord v1 = vertices[t[1]];
			Coord v2 = vertices[t[2]];

			N += (v1 - v0).cross(v2 - v0);
		}

		N.normalize();

		normals[tId] = N;
	}

	template<typename TDataType>
	void TriangleSet<TDataType>::updateTopology()
	{
		if (this->outVertexNormal()->isEmpty())
			this->outVertexNormal()->allocate();

		auto& vn = this->outVertexNormal()->getData();
		
		uint vertSize = this->m_coords.size();

		if (vn.size() != vertSize) {
			vn.resize(vertSize);
		}
		
		auto& vert2Tri = getVertex2Triangles();

		cuExecute(vertSize,
			TS_SetupVertexNormals,
			vn,
			this->m_coords,
			m_triangles,
			vert2Tri);
	}

	DEFINE_CLASS(TriangleSet);
}