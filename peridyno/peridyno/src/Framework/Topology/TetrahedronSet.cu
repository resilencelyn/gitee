#include "hip/hip_runtime.h"
#include "TetrahedronSet.h"
#include <fstream>
#include <iostream>
#include <sstream>

#include <thrust/sort.h>

namespace dyno
{
	template<typename TDataType>
	TetrahedronSet<TDataType>::TetrahedronSet()
		: TriangleSet<TDataType>()
	{
		
	}

	template<typename TDataType>
	TetrahedronSet<TDataType>::~TetrahedronSet()
	{
	}

	template<typename TDataType>
	void TetrahedronSet<TDataType>::setTetrahedrons(std::vector<Tetrahedron>& tetrahedrons)
	{
		std::vector<Triangle> triangles;

		m_tethedrons.resize(tetrahedrons.size());
		m_tethedrons.assign(tetrahedrons);

		this->updateTriangles();
	}

	template<typename TDataType>
	void TetrahedronSet<TDataType>::setTetrahedrons(DArray<Tetrahedron>& tetrahedrons)
	{
		if (tetrahedrons.size() != m_tethedrons.size())
		{
			m_tethedrons.resize(tetrahedrons.size());
		}

		m_tethedrons.assign(tetrahedrons);

		this->updateTriangles();
	}

	template<typename TDataType>
	void TetrahedronSet<TDataType>::loadTetFile(std::string filename)
	{
		std::string filename_node = filename;	filename_node.append(".node");
		std::string filename_ele = filename;	filename_ele.append(".ele");

		std::ifstream infile_node(filename_node);
		std::ifstream infile_ele(filename_ele);
		if (!infile_node || !infile_ele) {
			std::cerr << "Failed to open the tetrahedron file. Terminating.\n";
			exit(-1);
		}

		std::string line;
		std::getline(infile_node, line);
		std::stringstream ss_node(line);

		int node_num;
		ss_node >> node_num;
		std::vector<Coord> nodes;
		for (int i = 0; i < node_num; i++)
		{
			std::getline(infile_node, line);
			std::stringstream data(line);
			int id;
			Coord v;
			data >> id >> v[0] >> v[1] >> v[2];
			nodes.push_back(v);
		}

		
		std::getline(infile_ele, line);
		std::stringstream ss_ele(line);

		int ele_num;
		ss_ele >> ele_num;
		std::vector<Triangle> tris;
		std::vector<Tetrahedron> tets;
		for (int i = 0; i < ele_num; i++)
		{
			std::getline(infile_ele, line);
			std::stringstream data(line);
			int id;
			Tetrahedron tet;
			data >> id >> tet[0] >> tet[1] >> tet[2] >> tet[3];
			tet[0] -= 1;
			tet[1] -= 1;
			tet[2] -= 1;
			tet[3] -= 1;
			tets.push_back(tet);

			tris.push_back(Triangle(tet[0], tet[1], tet[2]));
			tris.push_back(Triangle(tet[0], tet[3], tet[1]));
			tris.push_back(Triangle(tet[1], tet[3], tet[2]));
			tris.push_back(Triangle(tet[0], tet[2], tet[3]));
		}

		this->setPoints(nodes);

		this->setTriangles(tris);
		this->setTetrahedrons(tets);
	}

	template<typename Tetrahedron>
	__global__ void TS_CountTets(
		DArray<int> counter,
		DArray<Tetrahedron> tets)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= tets.size()) return;

		Tetrahedron t = tets[tId];

		atomicAdd(&counter[t[0]], 1);
		atomicAdd(&counter[t[1]], 1);
		atomicAdd(&counter[t[2]], 1);
		atomicAdd(&counter[t[3]], 1);
	}

	template<typename Tetrahedron>
	__global__ void TS_SetupTetIds(
		DArrayList<int> tetIds,
		DArray<Tetrahedron> tets)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= tets.size()) return;

		Tetrahedron t = tets[tId];

		tetIds[t[0]].atomicInsert(tId);
		tetIds[t[1]].atomicInsert(tId);
		tetIds[t[2]].atomicInsert(tId);
		tetIds[t[3]].atomicInsert(tId);
	}

	template<typename TDataType>
	DArrayList<int>& TetrahedronSet<TDataType>::getVer2Tet()
	{
		DArray<int> counter;
		counter.resize(m_coords.size());
		counter.reset();

		cuExecute(m_tethedrons.size(),
			TS_CountTets,
			counter,
			m_tethedrons);

		m_ver2Tet.resize(counter);

		counter.reset();
		cuExecute(m_tethedrons.size(),
			TS_SetupTetIds,
			m_ver2Tet,
			m_tethedrons);

		counter.clear();

		return m_ver2Tet;
	}

	template<typename TDataType>
	void TetrahedronSet<TDataType>::getVolume(DArray<Real>& volume)
	{

	}

	template<typename TKey, typename Tetrahedron>
	__global__ void TS_SetupKeys(
		DArray<TKey> keys,
		DArray<int> ids,
		DArray<Tetrahedron> tets)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= tets.size()) return;

		Tetrahedron tet = tets[tId];
		keys[4 * tId] = TKey(tet[0], tet[1], tet[2]);
		keys[4 * tId + 1] = TKey(tet[1], tet[2], tet[3]);
		keys[4 * tId + 2] = TKey(tet[2], tet[3], tet[0]);
		keys[4 * tId + 3] = TKey(tet[3], tet[0], tet[1]);

		ids[4 * tId] = tId;
		ids[4 * tId + 1] = tId;
		ids[4 * tId + 2] = tId;
		ids[4 * tId + 3] = tId;
	}

	template<typename TKey>
	__global__ void TS_CountTriangleNumber(
		DArray<int> counter,
		DArray<TKey> keys) 
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= keys.size()) return;

		if (tId == 0 || keys[tId] != keys[tId - 1])
			counter[tId] = 1;
		else
			counter[tId] = 0;
	}

	template<typename Triangle, typename Tri2Tet, typename TKey>
	__global__ void TS_SetupTriangles(
		DArray<Triangle> triangles,
		DArray<Tri2Tet> tri2Tet,
		DArray<TKey> keys,
		DArray<int> counter,
		DArray<int> tetIds)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= keys.size()) return;

		int shift = counter[tId];
		if (tId == 0 || keys[tId] != keys[tId - 1])
		{
			TKey key = keys[tId];
			triangles[shift] = Triangle(key[0], key[1], key[2]);

			Tri2Tet t2T(EMPTY, EMPTY);
			t2T[0] = tetIds[tId];

			if (tId + 1 < keys.size() && keys[tId + 1] == key)
				t2T[1] = tetIds[tId + 1];

			tri2Tet[shift] = t2T;
		}
	}

	template<typename TKey>
	void printTKey(DArray<TKey> keys, int maxLength) {
		CArray<TKey> h_keys;
		h_keys.resize(keys.size());
		h_keys.assign(keys);

		int psize = min((int)h_keys.size(), maxLength);
		for (int i = 0; i < psize; i++)
		{
			printf("%d: %d %d %d \n", i, h_keys[i][0], h_keys[i][1], h_keys[i][2]);
		}

		h_keys.clear();
	}

	void printCount(DArray<int> keys, int maxLength) {
		CArray<int> h_keys;
		h_keys.resize(keys.size());
		h_keys.assign(keys);

		int psize = minimum((int)h_keys.size(), maxLength);
		for (int i = 0; i < psize; i++)
		{
			printf("%d: %d \n", i, h_keys[i]);
		}

		h_keys.clear();
	}

	template<typename TDataType>
	void TetrahedronSet<TDataType>::updateTriangles()
	{
		uint tetSize = m_tethedrons.size();

		DArray<TKey> keys;
		DArray<int> tetIds;

		keys.resize(4 * tetSize);
		tetIds.resize(4 * tetSize);

		cuExecute(tetSize,
			TS_SetupKeys,
			keys,
			tetIds,
			m_tethedrons);

		thrust::sort_by_key(thrust::device, keys.begin(), keys.begin() + keys.size(), tetIds.begin());

		DArray<int> counter;
		counter.resize(4 * tetSize);

		cuExecute(keys.size(),
			TS_CountTriangleNumber,
			counter,
			keys);

		int triNum = thrust::reduce(thrust::device, counter.begin(), counter.begin() + counter.size());
		thrust::exclusive_scan(thrust::device, counter.begin(), counter.begin() + counter.size(), counter.begin());

		tri2Tet.resize(triNum);

		auto& pTri = this->getTriangles();
		pTri.resize(triNum);
		cuExecute(keys.size(),
			TS_SetupTriangles,
			pTri,
			tri2Tet,
			keys,
			counter,
			tetIds);

		counter.clear();
		tetIds.clear();
		keys.clear();

		this->updateEdges();
	}


	template<typename TDataType>
	void TetrahedronSet<TDataType>::copyFrom(TetrahedronSet<TDataType> tetSet)
	{
		m_tethedrons.resize(tetSet.m_tethedrons.size());
		m_tethedrons.assign(tetSet.m_tethedrons);

		tri2Tet.resize(tetSet.tri2Tet.size());
		tri2Tet.assign(tetSet.tri2Tet);

		m_ver2Tet.assign(tetSet.m_ver2Tet);

		TriangleSet<TDataType>::copyFrom(tetSet);
	}

	DEFINE_CLASS(TetrahedronSet);
}