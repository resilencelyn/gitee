#include "hip/hip_runtime.h"
#include "DiscreteElementsToTriangleSet.h"

namespace dyno
{
	typedef typename TOrientedBox3D<Real> Box3D;

	template<typename TDataType>
	DiscreteElementsToTriangleSet<TDataType>::DiscreteElementsToTriangleSet()
		: TopologyMapping()
	{
		mStandardSphere.loadObjFile(getAssetPath() + "standard/standard_icosahedron.obj");
		mStandardCapsule.loadObjFile(getAssetPath() + "standard/standard_capsule.obj");
	}

	template<typename Triangle>
	__global__ void SetupCubeInstances(
		DArray<Vec3f> vertices,
		DArray<Triangle> indices,
		DArray<Box3D> boxes,
		uint pointOffset,
		uint indexOffset)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= boxes.size()) return;
		
		int idx = tId;
		Box3D box = boxes[idx];

		Vec3f hx = box.u * box.extent[0];
		Vec3f hy = box.v * box.extent[1];
		Vec3f hz = box.w * box.extent[2];

		Vec3f hyz = hy + hz;
		Vec3f hxy = hx + hy;
		Vec3f hxz = hx + hz;

		Vec3f c = box.center;

		Vec3f v0 = c - hx - hyz;
		Vec3f v1 = c + hx - hyz;
		Vec3f v2 = c + hxz - hy;
		Vec3f v3 = c - hxy + hz;

		Vec3f v4 = c - hxz + hy;
		Vec3f v5 = c + hxy - hz;
		Vec3f v6 = c + hx + hyz;
		Vec3f v7 = c - hx + hyz;

		vertices[pointOffset + idx * 8] = v0;
		vertices[pointOffset + idx * 8 + 1] = v1;
		vertices[pointOffset + idx * 8 + 2] = v2;
		vertices[pointOffset + idx * 8 + 3] = v3;
		vertices[pointOffset + idx * 8 + 4] = v4;
		vertices[pointOffset + idx * 8 + 5] = v5;
		vertices[pointOffset + idx * 8 + 6] = v6;
		vertices[pointOffset + idx * 8 + 7] = v7;

		uint offset = idx * 8 + pointOffset;

		indices[indexOffset + idx * 12] = Triangle(offset + 0, offset + 1, offset + 2);
		indices[indexOffset + idx * 12 + 1] = Triangle(offset + 0, offset + 2, offset + 3);

		indices[indexOffset + idx * 12 + 2] = Triangle(offset + 0, offset + 4, offset + 5);
		indices[indexOffset + idx * 12 + 3] = Triangle(offset + 0, offset + 5, offset + 1);

		indices[indexOffset + idx * 12 + 4] = Triangle(offset + 4, offset + 7, offset + 6);
		indices[indexOffset + idx * 12 + 5] = Triangle(offset + 4, offset + 6, offset + 5);

		indices[indexOffset + idx * 12 + 6] = Triangle(offset + 1, offset + 5, offset + 6);
		indices[indexOffset + idx * 12 + 7] = Triangle(offset + 1, offset + 6, offset + 2);

		indices[indexOffset + idx * 12 + 8] = Triangle(offset + 2, offset + 6, offset + 7);
		indices[indexOffset + idx * 12 + 9] = Triangle(offset + 2, offset + 7, offset + 3);

		indices[indexOffset + idx * 12 + 10] = Triangle(offset + 0, offset + 3, offset + 7);
		indices[indexOffset + idx * 12 + 11] = Triangle(offset + 0, offset + 7, offset + 4);
	}

	template<typename Triangle>
	__global__ void SetupTetInstances(
		DArray<Vec3f> vertices,
		DArray<Triangle> indices,
		DArray<Tet3D> tets,
		uint pointOffset,
		uint indexOffset)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= tets.size()) return;
	
		int idx = tId;
		Tet3D tet = tets[idx];

		Vec3f v0 = tet.v[0];
		Vec3f v1 = tet.v[1];
		Vec3f v2 = tet.v[2];
		Vec3f v3 = tet.v[3];

		vertices[pointOffset + idx * 4] = v0;
		vertices[pointOffset + idx * 4 + 1] = v1;
		vertices[pointOffset + idx * 4 + 2] = v2;
		vertices[pointOffset + idx * 4 + 3] = v3;

		uint offset = idx * 4 + pointOffset;

		indices[indexOffset + idx * 4] = Triangle(offset + 0, offset + 1, offset + 2);
		indices[indexOffset + idx * 4 + 1] = Triangle(offset + 0, offset + 1, offset + 3);
		indices[indexOffset + idx * 4 + 2] = Triangle(offset + 1, offset + 2, offset + 3);
		indices[indexOffset + idx * 4 + 3] = Triangle(offset + 0, offset + 2, offset + 3);
	}

	__global__ void SetupVerticesForSphereInstances(
		DArray<Vec3f> vertices,
		DArray<Vec3f> sphereVertices,
		DArray<Sphere3D> sphereInstances,
		uint pointOffset)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= sphereInstances.size() * sphereVertices.size()) return;

		uint instanceId = tId / sphereVertices.size();
		uint vertexId = tId % sphereVertices.size();

		Sphere3D sphere = sphereInstances[instanceId];

		Vec3f v = sphereVertices[vertexId];
		vertices[pointOffset + tId] = sphere.center + sphere.radius * v;
	}

	template<typename Triangle>
	__global__ void SetupIndicesForSphereInstances(
		DArray<Triangle> indices,
		DArray<Triangle> sphereIndices,
		DArray<Sphere3D> sphereInstances,
		uint vertexSize,						//vertex size of the instance sphere 
		uint indexOffset)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= sphereInstances.size() * sphereIndices.size()) return;

		uint instanceId = tId / sphereIndices.size();
		uint indexId = tId % sphereIndices.size();

		int vertexOffset = indexOffset + instanceId * vertexSize;
		
		Triangle tIndex = sphereIndices[indexId];
		indices[indexOffset + tId] = Triangle(tIndex[0] + vertexOffset, tIndex[1] + vertexOffset, tIndex[2] + vertexOffset);
	}

	template<typename TDataType>
	bool DiscreteElementsToTriangleSet<TDataType>::apply()
	{
		if (this->outTriangleSet()->isEmpty())
		{
			this->outTriangleSet()->allocate();
		}

		auto inTopo = this->inDiscreteElements()->getDataPtr();

		//printf("====================================================== inside box update\n");
		auto& sphereInstances = inTopo->getSpheres();
		auto& boxes = inTopo->getBoxes();
		auto& tets = inTopo->getTets();
		auto& caps = inTopo->getCaps();
		auto& tris = inTopo->getTris();
		ElementOffset elementOffset = inTopo->calculateElementOffset();

		int numOfSpheres = sphereInstances.size();
		int numOfBoxes = boxes.size();
		int numOfTets = tets.size();

		auto triSet = this->outTriangleSet()->getDataPtr();

		auto& vertices = triSet->getPoints();
		auto& indices = triSet->getTriangles();

		auto& sphereVertices = mStandardSphere.getPoints();
		auto& sphereIndices = mStandardSphere.getTriangles();

		int numOfVertices = 8 * numOfBoxes + 4 * numOfTets + sphereVertices.size() * numOfSpheres;
		int numOfTriangles = 12 * numOfBoxes + 4 * numOfTets + sphereIndices.size() * numOfSpheres;

		vertices.resize(numOfVertices);
		indices.resize(numOfTriangles);

		uint vertexOffset = 0;
		uint indexOffset = 0;

		cuExecute(numOfSpheres * sphereVertices.size(),
			SetupVerticesForSphereInstances,
			vertices,
			sphereVertices,
			sphereInstances,
			vertexOffset);

		cuExecute(numOfSpheres * sphereIndices.size(),
			SetupIndicesForSphereInstances,
			indices,
			sphereIndices,
			sphereInstances,
			sphereVertices.size(),
			indexOffset);

		vertexOffset += numOfSpheres * sphereVertices.size();
		indexOffset += numOfSpheres * sphereIndices.size();

		cuExecute(numOfBoxes,
			SetupCubeInstances,
			vertices,
			indices,
			boxes,
			vertexOffset,
			indexOffset);

		vertexOffset += boxes.size() * 8;
		indexOffset += boxes.size() * 12;

		cuExecute(numOfTets,
			SetupTetInstances,
			vertices,
			indices,
			tets,
			vertexOffset,
			indexOffset);

		this->outTriangleSet()->getDataPtr()->updateEdges();

		return true;
	}

	DEFINE_CLASS(DiscreteElementsToTriangleSet);
}