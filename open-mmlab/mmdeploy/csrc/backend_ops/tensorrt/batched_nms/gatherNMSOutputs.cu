#include "hip/hip_runtime.h"
// Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
// modify from
// https://github.com/NVIDIA/TensorRT/tree/master/plugin/batchedNMSPlugin
#include <vector>

#include "kernel.h"
#include "trt_plugin_helper.hpp"

template <typename T_BBOX, typename T_SCORE, unsigned nthds_per_cta>
__launch_bounds__(nthds_per_cta) __global__
    void gatherNMSOutputs_kernel(const bool shareLocation, const int numImages,
                                 const int numPredsPerClass, const int numClasses, const int topK,
                                 const int keepTopK, const int *indices, const T_SCORE *scores,
                                 const T_BBOX *bboxData, T_BBOX *nmsedDets, int *nmsedLabels,
                                 bool clipBoxes) {
  if (keepTopK > topK) return;
  for (int i = blockIdx.x * nthds_per_cta + threadIdx.x; i < numImages * keepTopK;
       i += gridDim.x * nthds_per_cta) {
    const int imgId = i / keepTopK;
    const int detId = i % keepTopK;
    const int offset = imgId * numClasses * topK;
    const int index = indices[offset + detId];
    const T_SCORE score = scores[offset + detId];
    if (index == -1) {
      nmsedLabels[i] = -1;
      nmsedDets[i * 5] = 0;
      nmsedDets[i * 5 + 1] = 0;
      nmsedDets[i * 5 + 2] = 0;
      nmsedDets[i * 5 + 3] = 0;
      nmsedDets[i * 5 + 4] = 0;
    } else {
      const int bboxOffset =
          imgId * (shareLocation ? numPredsPerClass : (numClasses * numPredsPerClass));
      const int bboxId =
          ((shareLocation ? (index % numPredsPerClass) : index % (numClasses * numPredsPerClass)) +
           bboxOffset) *
          4;
      nmsedLabels[i] = (index % (numClasses * numPredsPerClass)) / numPredsPerClass;  // label
      // clipped bbox xmin
      nmsedDets[i * 5] =
          clipBoxes ? max(min(bboxData[bboxId], T_BBOX(1.)), T_BBOX(0.)) : bboxData[bboxId];
      // clipped bbox ymin
      nmsedDets[i * 5 + 1] =
          clipBoxes ? max(min(bboxData[bboxId + 1], T_BBOX(1.)), T_BBOX(0.)) : bboxData[bboxId + 1];
      // clipped bbox xmax
      nmsedDets[i * 5 + 2] =
          clipBoxes ? max(min(bboxData[bboxId + 2], T_BBOX(1.)), T_BBOX(0.)) : bboxData[bboxId + 2];
      // clipped bbox ymax
      nmsedDets[i * 5 + 3] =
          clipBoxes ? max(min(bboxData[bboxId + 3], T_BBOX(1.)), T_BBOX(0.)) : bboxData[bboxId + 3];
      nmsedDets[i * 5 + 4] = score;
    }
  }
}

template <typename T_BBOX, typename T_SCORE>
pluginStatus_t gatherNMSOutputs_gpu(hipStream_t stream, const bool shareLocation,
                                    const int numImages, const int numPredsPerClass,
                                    const int numClasses, const int topK, const int keepTopK,
                                    const void *indices, const void *scores, const void *bboxData,
                                    void *nmsedDets, void *nmsedLabels, bool clipBoxes) {
  const int BS = 32;
  const int GS = 32;
  gatherNMSOutputs_kernel<T_BBOX, T_SCORE, BS><<<GS, BS, 0, stream>>>(
      shareLocation, numImages, numPredsPerClass, numClasses, topK, keepTopK, (int *)indices,
      (T_SCORE *)scores, (T_BBOX *)bboxData, (T_BBOX *)nmsedDets, (int *)nmsedLabels, clipBoxes);

  CSC(hipGetLastError(), STATUS_FAILURE);
  return STATUS_SUCCESS;
}

// gatherNMSOutputs LAUNCH CONFIG {{{
typedef pluginStatus_t (*nmsOutFunc)(hipStream_t, const bool, const int, const int, const int,
                                     const int, const int, const void *, const void *, const void *,
                                     void *, void *, bool);
struct nmsOutLaunchConfig {
  DataType t_bbox;
  DataType t_score;
  nmsOutFunc function;

  nmsOutLaunchConfig(DataType t_bbox, DataType t_score) : t_bbox(t_bbox), t_score(t_score) {}
  nmsOutLaunchConfig(DataType t_bbox, DataType t_score, nmsOutFunc function)
      : t_bbox(t_bbox), t_score(t_score), function(function) {}
  bool operator==(const nmsOutLaunchConfig &other) {
    return t_bbox == other.t_bbox && t_score == other.t_score;
  }
};

using nvinfer1::DataType;

static std::vector<nmsOutLaunchConfig> nmsOutFuncVec;

bool nmsOutputInit() {
  nmsOutFuncVec.push_back(
      nmsOutLaunchConfig(DataType::kFLOAT, DataType::kFLOAT, gatherNMSOutputs_gpu<float, float>));
  return true;
}

static bool initialized = nmsOutputInit();

//}}}

pluginStatus_t gatherNMSOutputs(hipStream_t stream, const bool shareLocation, const int numImages,
                                const int numPredsPerClass, const int numClasses, const int topK,
                                const int keepTopK, const DataType DT_BBOX, const DataType DT_SCORE,
                                const void *indices, const void *scores, const void *bboxData,
                                void *nmsedDets, void *nmsedLabels, bool clipBoxes) {
  nmsOutLaunchConfig lc = nmsOutLaunchConfig(DT_BBOX, DT_SCORE);
  for (unsigned i = 0; i < nmsOutFuncVec.size(); ++i) {
    if (lc == nmsOutFuncVec[i]) {
      DEBUG_PRINTF("gatherNMSOutputs kernel %d\n", i);
      return nmsOutFuncVec[i].function(stream, shareLocation, numImages, numPredsPerClass,
                                       numClasses, topK, keepTopK, indices, scores, bboxData,
                                       nmsedDets, nmsedLabels, clipBoxes);
    }
  }
  return STATUS_BAD_PARAM;
}
